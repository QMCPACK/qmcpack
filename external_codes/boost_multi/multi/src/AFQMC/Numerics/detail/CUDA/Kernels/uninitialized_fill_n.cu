#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov
//    Lawrence Livermore National Laboratory
////////////////////////////////////////////////////////////////////////////////

#include <cassert>
#include <complex>
#include <type_traits>
/*
#include <thrust/complex.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/uninitialized_fill.h>
*/
#define ENABLE_CUDA 1
#include "AFQMC/Memory/CUDA/cuda_utilities.h"
#include "AFQMC/Numerics/detail/CUDA/Kernels/cuda_settings.h"

namespace kernels
{
template<typename T, typename Size>
__global__ void kernel_uninitialized_fill_n(Size N, T* x, T const a)
{
  Size N0(8 * blockDim.x * blockIdx.x);
  T* x_(x + N0);
  Size N_(min(Size(8 * blockDim.x), N - N0));
  for (Size ip = Size(threadIdx.x); ip < N_; ip += Size(blockDim.x))
  {
    x_[ip] = a;
  }
}

void uninitialized_fill_n(bool* first, int N, bool const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(int* first, int N, int const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(float* first, int N, float const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(double* first, int N, double const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(std::complex<float>* first, int N, std::complex<float> const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(std::complex<double>* first, int N, std::complex<double> const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

/*void uninitialized_fill_n(double2 * first, int N, double2 const value)*/
/*{ */
/*kernel_uninitialized_fill_n<<<1,256>>>(N,first,value);*/
/*qmc_cuda::cuda_check(hipGetLastError());*/
/*qmc_cuda::cuda_check(hipDeviceSynchronize());*/
/*}*/

void uninitialized_fill_n(bool* first, long N, bool const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(int* first, long N, int const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(float* first, long N, float const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(double* first, long N, double const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(std::complex<float>* first, long N, std::complex<float> const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void uninitialized_fill_n(std::complex<double>* first, long N, std::complex<double> const value)
{
  int N_(8 * DEFAULT_BLOCK_SIZE);
  size_t nblk((N + N_ - 1) / N_);
  size_t nthr(DEFAULT_BLOCK_SIZE);
  kernel_uninitialized_fill_n<<<nblk, nthr>>>(N, first, value);
  qmc_cuda::cuda_check(hipGetLastError());
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

/*void uninitialized_fill_n(double2 * first, long N, double2 const value)*/
/*{*/
/*kernel_uninitialized_fill_n<<<1,256>>>(N,first,value);*/
/*qmc_cuda::cuda_check(hipGetLastError());*/
/*qmc_cuda::cuda_check(hipDeviceSynchronize());*/
/*}*/


} // namespace kernels
