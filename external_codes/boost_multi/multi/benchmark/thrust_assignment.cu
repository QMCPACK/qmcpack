#ifdef COMPILATION// -*-indent-tabs-mode:t;c-basic-offset:4;tab-width:4;-*-
/usr/local/cuda-11.0/bin/nvcc -std=c++17 -ftemplate-backtrace-limit=0 $0 -o $0.$X `pkg-config --cflags --libs cudart-11.0 cuda-11.0` -lboost_timer&&$0.$X&&rm $0.$X;exit
#endif
// © Alfredo A. Correa 2020

#include<benchmark/benchmark.h>

//#include<thrust/complex.h>
#include<thrust/device_allocator.h>
#include<thrust/device_vector.h>

#include "../../multi/array.hpp"
#include "../../multi/adaptors/thrust.hpp"

namespace multi = boost::multi;

#if not defined(NDEBUG)
#warning "Benchmark in debug mode?"
#endif

static void 
BM_cpu_vector_double_assignment
(benchmark::State& st){
	std::vector<double> const A(1<<28, 1.);
	std::vector<double>       B(A.size(), 2.);
	for(auto _ : st){
		B = A;
		benchmark::DoNotOptimize(B);
		benchmark::ClobberMemory();
	//	hipDeviceSynchronize();
	}
	std::cout << A.size()*sizeof(A.front())/1e6 << "MB" << std::endl;
	st.SetBytesProcessed(st.iterations()*A.size()*sizeof(A.front()));
	st.SetItemsProcessed(st.iterations()*A.size());
}

static void 
BM_device_cudaMemcpy_double_assignment
(benchmark::State& st){
	thrust::device_vector<double> const A(1<<28, 1.);
	thrust::device_vector<double>       B(A.size(), 2.);
	for(auto _ : st){
		hipMemcpy(raw_pointer_cast(B.data()), raw_pointer_cast(A.data()), A.size()*sizeof(A.front()), hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();
	}
	st.SetBytesProcessed(st.iterations()*A.size()*sizeof(A.front()));
	st.SetItemsProcessed(st.iterations()*A.size());
}

static void 
BM_device_vector_double_assignment
(benchmark::State& st){
	thrust::device_vector<double> const A(1<<28, 1.);
	thrust::device_vector<double>       B(A.size(), 2.);
	for(auto _ : st){
		B = A;
	}
	st.SetBytesProcessed(st.iterations()*A.size()*sizeof(A.front()));
	st.SetItemsProcessed(st.iterations()*A.size());
}

static void BM_device_array_double_assignment(benchmark::State& st){
	using T = double;
	using alloc = thrust::device_allocator<T>; // std::allocator<T>;
	multi::array<T, 1, alloc> const A(1<<28, 1.);
	multi::array<T, 1, alloc>       B(extensions(A), 2.);
	for(auto _ : st){
		B() = A();
	}
	st.SetBytesProcessed(st.iterations()*A.num_elements()*sizeof(*A.base()));
	st.SetItemsProcessed(st.iterations()*A.num_elements());
}

static void BM_cpu_array_2D_double_assignment(benchmark::State& st){
	using T = double;
	using alloc = std::allocator<T>; //thrust::device_allocator<T>; // std::allocator<T>;
	multi::array<T, 2, alloc> const A({1<<14, 1<<14}, 1.);
	multi::array<T, 2, alloc>       B(extensions(A), 2.);
	for(auto _ : st){
		B() = A();
	}
	std::cout << A.num_elements()*sizeof(*A.base())/1e6 << "MB"<<std::endl;
	st.SetBytesProcessed(st.iterations()*A.num_elements()*sizeof(*A.base()));
	st.SetItemsProcessed(st.iterations()*A.num_elements());
}

static void BM_device_array_2D_double_assignment(benchmark::State& st){
	using T = double;
	using alloc = thrust::device_allocator<T>; // std::allocator<T>;
	multi::array<T, 2, alloc> const A({1<<14, 1<<14}, 1.);
	multi::array<T, 2, alloc>       B(extensions(A), 2.);
	for(auto _ : st){
		B() = A();
		hipDeviceSynchronize();
	}
	if( B[10][10] == 2.) throw 0;
	st.SetBytesProcessed(st.iterations()*A.num_elements()*sizeof(*A.base()));
	st.SetItemsProcessed(st.iterations()*A.num_elements());
}

BENCHMARK(BM_cpu_vector_double_assignment);
BENCHMARK(BM_device_vector_double_assignment);
BENCHMARK(BM_device_cudaMemcpy_double_assignment);
BENCHMARK(BM_device_array_double_assignment);
BENCHMARK(BM_cpu_array_2D_double_assignment);
BENCHMARK(BM_device_array_2D_double_assignment);

BENCHMARK_MAIN();

//	using T = thrust::complex<double>;
//	using alloc = thrust::device_allocator<T>; // std::allocator<T>;
//	multi::array<T, 1, alloc> const A(10000, 1.);
//	multi::array<T, 1, alloc>       B(10000);
//	B = A;
//	assert( T{B[10]} == 1. );

//}

