#define BOOST_TEST_MODULE "C++ Unit Tests for Multi CUDA thrust universal copy and assignment"
#include <boost/test/unit_test.hpp>

#include <multi/array.hpp>

#include <multi/adaptors/thrust.hpp>

#include <thrust/complex.h>

#include <boost/mpl/list.hpp>

namespace multi = boost::multi;

// template<>
// inline constexpr bool multi::force_element_trivial_default_construction<std::complex<double>> = false;

// template<>
// inline constexpr bool multi::force_element_trivial_default_construction<thrust::complex<double>> = false;

// template<>
// inline constexpr bool multi::force_element_trivial_default_construction<std::complex<float>> = false;

// template<>
// inline constexpr bool multi::force_element_trivial_default_construction<thrust::complex<float>> = false;

using test_types = boost::mpl::list<
	char, unsigned, int,
	::thrust::complex<double>, std::complex<double>,
	::thrust::complex<float>, std::complex<float>,
	double, float>;

BOOST_AUTO_TEST_CASE(warmup) {
	using T = double;

	auto const n = 8000;

	multi::array<T, 2, thrust::cuda::universal_allocator<T>> src({n, n});
	multi::array<T, 2, thrust::cuda::universal_allocator<T>> dst(extensions(src));

	auto const threshold = 0.30;

	auto const size = src.num_elements() * sizeof(T) / 1e9;

	auto const dummy = std::invoke([&] {
		auto start_time = std::chrono::high_resolution_clock::now();
		hipMemcpy(raw_pointer_cast(dst.data_elements()), raw_pointer_cast(src.data_elements()), src.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - start_time;
		auto                          rate = size / time.count();
		// std::cout<<"memcpy    rate = "<< rate <<" GB/s (ratio = 1)\n";
		return rate;
	});

	auto const memcpy_rate = std::invoke([&] {
		auto start_time = std::chrono::high_resolution_clock::now();
		hipMemcpy(raw_pointer_cast(dst.data_elements()), raw_pointer_cast(src.data_elements()), src.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - start_time;
		auto                          rate = size / time.count();
		// std::cout<<"memcpy    rate = "<< rate <<" GB/s (ratio = 1)\n";
		return rate;
	});

	{  // cctor
		auto tick = std::chrono::high_resolution_clock::now();

		auto dst2 = src;

		std::chrono::duration<double> time  = std::chrono::high_resolution_clock::now() - tick;
		double                        rate  = size / time.count();
		double                        ratio = rate / memcpy_rate;

		// std::cout<<"cctor      rate = "<< rate <<" GB/s (ratio = "<< ratio <<")\n";
		if(ratio >= threshold) {
			std::cout << "x";
		}
	}
	{  // assign
		auto tick = std::chrono::high_resolution_clock::now();

		dst = src;

		std::chrono::duration<double> time  = std::chrono::high_resolution_clock::now() - tick;
		double                        rate  = size / time.count();
		double                        ratio = rate / memcpy_rate;

		// std::cout << "assign     rate = "<< rate <<" GB/s (ratio = "<< ratio <<")\n";
		if(ratio >= threshold) {
			std::cout << "x";
		}
	}
	{  // subarray assign
		auto tick = std::chrono::high_resolution_clock::now();

		dst({0, n - 2}, {0, n - 2}) = src({2, n}, {2, n});

		std::chrono::duration<double> time  = std::chrono::high_resolution_clock::now() - tick;
		double                        rate  = size / time.count();
		double                        ratio = rate / memcpy_rate;
		// std::cout << "subasssign rate = "<< rate <<" GB/s (ratio = "<< ratio << ")\n";
		if(ratio >= threshold) {
			std::cout << "x";
		}
	}
}

BOOST_AUTO_TEST_CASE_TEMPLATE(thrust_universal_speed, T, test_types) {
	std::cout << typeid(T).name() << " ******************************************\n";

	auto const n = 8000;

	multi::array<T, 2, thrust::cuda::universal_allocator<T>> src({n, n});
	multi::array<T, 2, thrust::cuda::universal_allocator<T>> dst(extensions(src));

	auto const threshold = 0.10;

	auto const size = src.num_elements() * sizeof(T) / 1e9;

	auto const dummy = std::invoke([&] {
		auto start_time = std::chrono::high_resolution_clock::now();
		hipMemcpy(raw_pointer_cast(dst.data_elements()), raw_pointer_cast(src.data_elements()), src.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - start_time;
		auto                          rate = size / time.count();
		std::cout << "memcpy    rate = " << rate << " GB/s (warmup)\n";
		return rate;
	});

	auto const memcpy_rate = std::invoke([&] {
		auto start_time = std::chrono::high_resolution_clock::now();
		hipMemcpy(raw_pointer_cast(dst.data_elements()), raw_pointer_cast(src.data_elements()), src.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - start_time;
		auto                          rate = size / time.count();
		std::cout << "memcpy    rate = " << rate << " GB/s (ratio = 1)\n";
		return rate;
	});

	{  // cctor
		auto tick = std::chrono::high_resolution_clock::now();

		auto dst2 = src;

		std::chrono::duration<double> time  = std::chrono::high_resolution_clock::now() - tick;
		double                        rate  = size / time.count();
		double                        ratio = rate / memcpy_rate;

		std::cout << "cctor      rate = " << rate << " GB/s (ratio = " << ratio << ")\n";
		BOOST_TEST(ratio >= threshold);
	}
	{  // assign
		auto tick = std::chrono::high_resolution_clock::now();

		dst = src;

		std::chrono::duration<double> time  = std::chrono::high_resolution_clock::now() - tick;
		double                        rate  = size / time.count();
		double                        ratio = rate / memcpy_rate;

		std::cout << "assign     rate = " << rate << " GB/s (ratio = " << ratio << ")\n";
		BOOST_TEST(ratio >= threshold);
	}
	{  // subarray assign
		auto tick = std::chrono::high_resolution_clock::now();

		dst({0, n - 2}, {0, n - 2}) = src({2, n}, {2, n});

		std::chrono::duration<double> time  = std::chrono::high_resolution_clock::now() - tick;
		double                        rate  = size / time.count();
		double                        ratio = rate / memcpy_rate;
		std::cout << "subasssign rate = " << rate << " GB/s (ratio = " << ratio << ")\n";
		BOOST_TEST(ratio >= threshold);
	}
}
