#define BOOST_TEST_MODULE "C++ Unit Tests for Multi CUDA thrust universal copy and assignment"
#include<boost/test/unit_test.hpp>

#include <multi/array.hpp>
#include <multi/adaptors/thrust.hpp>

#include <thrust/complex.h>

#include <boost/mpl/list.hpp>


namespace multi = boost::multi;
using complex = thrust::complex<double>;

typedef boost::mpl::list<double, complex> test_types;

BOOST_AUTO_TEST_CASE_TEMPLATE(thrust_universal_speed, T, test_types) {

	auto const n = 8000;

	multi::array<T, 2, thrust::cuda::universal_allocator<T>> src({n, n});
	multi::array<T, 2, thrust::cuda::universal_allocator<T>> dst(extensions(src));

	auto const threshold = 0.2;

	auto const size = src.num_elements()*sizeof(T)/1e9;

	auto const dummy = std::invoke([&]{
		auto start_time = std::chrono::high_resolution_clock::now();
		hipMemcpy(raw_pointer_cast(dst.data_elements()), raw_pointer_cast(src.data_elements()), src.num_elements()*sizeof(T), hipMemcpyDeviceToDevice);
		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - start_time;
		auto rate = size/time.count();
		std::cout<<"memcpy    rate = "<< rate <<" GB/s (ratio = 1)\n";
		return rate;
	});

	auto const memcpy_rate = std::invoke([&]{
		auto start_time = std::chrono::high_resolution_clock::now();
		hipMemcpy(raw_pointer_cast(dst.data_elements()), raw_pointer_cast(src.data_elements()), src.num_elements()*sizeof(T), hipMemcpyDeviceToDevice);
		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - start_time;
		auto rate = size/time.count();
		std::cout<<"memcpy    rate = "<< rate <<" GB/s (ratio = 1)\n";
		return rate;
	});

	{ //cctor
		auto tick = std::chrono::high_resolution_clock::now();

		auto dst2 = src;

		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - tick;
		double rate = size/time.count();
		double ratio = rate/memcpy_rate;

		std::cout<<"cctor      rate = "<< rate <<" GB/s (ratio = "<< ratio <<")\n";
		BOOST_TEST(ratio >= threshold);
	}
	{ //assign
		auto tick = std::chrono::high_resolution_clock::now();

		dst = src;

		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - tick;
		double rate = size/time.count();
		double ratio = rate/memcpy_rate;

		std::cout << "assign     rate = "<< rate <<" GB/s (ratio = "<< ratio <<")\n";
		BOOST_TEST(ratio >= threshold);
	}
	{ //subarray assign
		auto tick = std::chrono::high_resolution_clock::now();

		dst({0, n - 2}, {0, n - 2}) = src({2, n}, {2, n});

		std::chrono::duration<double> time = std::chrono::high_resolution_clock::now() - tick;
		double rate = size/time.count();
		double ratio = rate/memcpy_rate;
		std::cout << "subasssign rate = "<< rate <<" GB/s (ratio = "<< ratio << ")\n";
		BOOST_TEST(ratio >= threshold);
	}
}

