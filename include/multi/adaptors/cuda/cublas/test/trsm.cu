// -*-indent-tabs-mode:t;c-basic-offset:4;tab-width:4;autowrap:nil;-*-
// Copyright 2023 Alfredo A. Correa

#define BOOST_TEST_MODULE "C++ Unit Tests for Multi CUBLAS trsm"
#include<boost/test/unit_test.hpp>

#include <multi/adaptors/cuda/cublas.hpp>
#include <multi/adaptors/blas/trsm.hpp>

#include <multi/adaptors/thrust.hpp>

#include<thrust/complex.h>

namespace multi = boost::multi;

BOOST_AUTO_TEST_CASE(unit_trsm_multi_blas_trsm_complex_nonsquare_default_diagonal_hermitized_gemm_check_no_const) {
	namespace blas = multi::blas;
	using complex = thrust::complex<double>; complex const I{0.0, 1.0};  // NOLINT(readability-identifier-length) imag unit
	// NOLINTNEXTLINE(readability-identifier-length) BLAS naming
	multi::array<complex, 2> const A = {
		{ 1.0 + 0.0*I, 3.0 + 1.2*I,  5.0 - 12.0*I},
		{ 0.0 + 0.0*I, 1.0 + 0.0*I,  2.1 +  1.1*I},
		{ 0.0 + 0.0*I, 0.0 + 0.0*I,  1.0 +  0.0*I},
	};
	// NOLINTNEXTLINE(readability-identifier-length) BLAS naming
	multi::array<complex, 2> B = {
		{1.0 + 1.0*I, 2.0 + 1.0*I, 3.0 + 1.0*I},
		{5.0 + 3.0*I, 9.0 + 3.0*I, 1.0 - 1.0*I},
	};

	multi::thrust::cuda::array<complex, 2> const A_gpu = A;
	multi::thrust::cuda::array<complex, 2> B_gpu = B;

	using multi::blas::trsm;
	using multi::blas::filling;
	using multi::blas::hermitized;

	blas::trsm(blas::side::left, blas::filling::upper, blas::diagonal::unit, complex{1.0, 0.0}, A    , blas::H(B));  // B†←A⁻¹.B†, B←B.A⁻¹†, B←(A⁻¹.B†)†
	BOOST_REQUIRE_CLOSE( B[1][0].real() , -43.439999999999998 , 0.001);
	BOOST_REQUIRE_CLOSE( B[1][0].imag() , -13.000000000000002 , 0.001);

	blas::trsm(blas::side::left, blas::filling::upper, blas::diagonal::unit, complex{1.0, 0.0}, A_gpu, blas::H(B_gpu));
	multi::array<complex, 2> B_cpy = B_gpu;

	BOOST_REQUIRE_CLOSE( B_cpy[1][0].real() , -43.439999999999998 , 0.001);
	BOOST_REQUIRE_CLOSE( B_cpy[1][0].imag() , -13.000000000000002 , 0.001);
}

BOOST_AUTO_TEST_CASE(trsm_multi_blas_trsm_complex_nonsquare_default_diagonal_hermitized_gemm_check_no_const) {
	namespace blas = multi::blas;
	using complex = thrust::complex<double>; complex const I{0.0, 1.0};  // NOLINT(readability-identifier-length) imag unit
	// NOLINTNEXTLINE(readability-identifier-length) BLAS naming
	multi::array<complex, 2> const A = {
		{ 1.0 + 4.0*I, 3.0 + 0.0*I,  4.0 - 10.0*I},
		{ 0.0 + 0.0*I, 7.0 - 3.0*I,  1.0 +  0.0*I},
		{ 0.0 + 0.0*I, 0.0 + 0.0*I,  8.0 -  2.0*I},
	};
	// NOLINTNEXTLINE(readability-identifier-length) BLAS naming
	multi::array<complex, 2> B = {
		{1.0 + 1.0*I, 2.0 + 1.0*I, 3.0 + 1.0*I},
		{5.0 + 3.0*I, 9.0 + 3.0*I, 1.0 - 1.0*I},
	};

	multi::thrust::cuda::array<complex, 2> const A_gpu = A;
	multi::thrust::cuda::array<complex, 2> B_gpu = B;

	using multi::blas::trsm;
	using multi::blas::filling;
	using multi::blas::hermitized;

	// B = ConjugateTranspose[Inverse[A] . ConjugateTranspose[B]]
	// ConjugateTranspose[B] = Inverse[A] . ConjugateTranspose[B]
	blas::trsm(blas::side::left, blas::filling::upper, blas::diagonal::non_unit, complex{1.0, 0.0}, A    , blas::H(B));  // B†←A⁻¹.B†, B←B.A⁻¹†, B←(A⁻¹.B†)†
	BOOST_REQUIRE_CLOSE( B[1][0].real(), -0.72562939983295538 , 0.001);
	BOOST_REQUIRE_CLOSE( B[1][0].imag(),  0.046772461520104877, 0.001);

	BOOST_REQUIRE_CLOSE( real(blas::H(B)[0][1]), -0.72562939983295538 , 0.001);
	BOOST_REQUIRE_CLOSE( imag(blas::H(B)[0][1]), -0.046772461520104877, 0.001);

	blas::trsm(blas::side::left, blas::filling::upper, blas::diagonal::non_unit, complex{1.0, 0.0}, A_gpu, blas::H(B_gpu));
	hipDeviceSynchronize();

	multi::array<complex, 2> B_cpy = B_gpu;
	BOOST_REQUIRE_CLOSE( B_cpy[1][0].real() , -0.72562939983295538 , 0.001);
	BOOST_REQUIRE_CLOSE( B_cpy[1][0].imag() ,  0.046772461520104877, 0.001);
}

BOOST_AUTO_TEST_CASE(default_param_unit_trsm_multi_blas_trsm_complex_nonsquare_default_diagonal_hermitized_gemm_check_no_const) {
	namespace blas = multi::blas;
	using complex = thrust::complex<double>; complex const I{0.0, 1.0};  // NOLINT(readability-identifier-length) imag unit
	// NOLINTNEXTLINE(readability-identifier-length) BLAS naming
	multi::array<complex, 2> const A = {
		{ 1.0 + 4.0*I, 3.0 + 0.0*I,  4.0 - 10.0*I},
		{ 0.0 + 0.0*I, 7.0 - 3.0*I,  1.0 +  0.0*I},
		{ 0.0 + 0.0*I, 0.0 + 0.0*I,  8.0 -  2.0*I},
	};
	// NOLINTNEXTLINE(readability-identifier-length) BLAS naming
	multi::array<complex, 2> B = {
		{1.0 + 1.0*I, 2.0 + 1.0*I, 3.0 + 1.0*I},
		{5.0 + 3.0*I, 9.0 + 3.0*I, 1.0 - 1.0*I},
	};

	multi::thrust::cuda::array<complex, 2> const A_gpu = A;
	multi::thrust::cuda::array<complex, 2> B_gpu = B;

	using multi::blas::trsm;
	using multi::blas::filling;
	using multi::blas::hermitized;

	// B = ConjugateTranspose[Inverse[A] . ConjugateTranspose[B]]
	// ConjugateTranspose[B] = Inverse[A] . ConjugateTranspose[B]
	blas::trsm(blas::side::left, blas::filling::upper, complex{1.0, 0.0}, A    , blas::H(B));  // B†←A⁻¹.B†, B←B.A⁻¹†, B←(A⁻¹.B†)†
	BOOST_REQUIRE_CLOSE( B[1][0].real(), -0.72562939983295538 , 0.001);
	BOOST_REQUIRE_CLOSE( B[1][0].imag(),  0.046772461520104877, 0.001);

	BOOST_REQUIRE_CLOSE( real(blas::H(B)[0][1]), -0.72562939983295538 , 0.001);
	BOOST_REQUIRE_CLOSE( imag(blas::H(B)[0][1]), -0.046772461520104877, 0.001);

	blas::trsm(blas::side::left, blas::filling::upper, blas::diagonal::non_unit, complex{1.0, 0.0}, A_gpu, blas::H(B_gpu));
	hipDeviceSynchronize();

	multi::array<complex, 2> B_cpy = B_gpu;
	BOOST_REQUIRE_CLOSE( B_cpy[1][0].real() , -0.72562939983295538 , 0.001);
	BOOST_REQUIRE_CLOSE( B_cpy[1][0].imag() ,  0.046772461520104877, 0.001);
}
