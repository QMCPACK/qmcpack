#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by: Ken Esler, kpesler@gmail.com, University of Illinois at Urbana-Champaign
//                    Jeremy McMinnis, jmcminis@gmail.com, University of Illinois at Urbana-Champaign
//                    Jeongnim Kim, jeongnim.kim@gmail.com, University of Illinois at Urbana-Champaign
//                    Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ken Esler, kpesler@gmail.com, University of Illinois at Urbana-Champaign
//////////////////////////////////////////////////////////////////////////////////////


#include "CudaCoulomb.h"


const int MAX_TEXTURES = 10;
__constant__ float Acuda[16];

void init_Acuda()
{
  static bool initialized(false);
  if (!initialized)
  {
    float A_h[16] = {-1.0 / 6.0, 3.0 / 6.0, -3.0 / 6.0, 1.0 / 6.0, 3.0 / 6.0, -6.0 / 6.0, 0.0 / 6.0, 4.0 / 6.0,
                     -3.0 / 6.0, 3.0 / 6.0, 3.0 / 6.0,  1.0 / 6.0, 1.0 / 6.0, 0.0 / 6.0,  0.0 / 6.0, 0.0 / 6.0};
    hipMemcpyToSymbol(HIP_SYMBOL(Acuda), A_h, 16 * sizeof(float), 0, hipMemcpyHostToDevice);
    initialized = true;
  }
}


texture<float, 1, hipReadModeElementType> myTex;
texture<float, 1, hipReadModeElementType> tex00, tex01, tex02, tex03, tex04, tex05, tex06, tex07, tex08, tex09;
bool textureInUse[MAX_TEXTURES] = {false, false, false, false, false, false, false, false, false, false};


#define arraytexFetch(_u, _texnum, _return) \
  switch (_texnum)                          \
  {                                         \
  case 0:                                   \
    _return = tex1D(tex00, (_u));           \
    break;                                  \
  case 1:                                   \
    _return = tex1D(tex01, (_u));           \
    break;                                  \
  case 2:                                   \
    _return = tex1D(tex02, (_u));           \
    break;                                  \
  case 3:                                   \
    _return = tex1D(tex03, (_u));           \
    break;                                  \
  case 4:                                   \
    _return = tex1D(tex04, (_u));           \
    break;                                  \
  case 5:                                   \
    _return = tex1D(tex05, (_u));           \
    break;                                  \
  case 6:                                   \
    _return = tex1D(tex06, (_u));           \
    break;                                  \
  case 7:                                   \
    _return = tex1D(tex07, (_u));           \
    break;                                  \
  case 8:                                   \
    _return = tex1D(tex08, (_u));           \
    break;                                  \
  case 9:                                   \
    _return = tex1D(tex09, (_u));           \
    break;                                  \
  }

#include <stdio.h>

TextureSpline::TextureSpline()
{
  int iTex = 0;
  while (iTex < MAX_TEXTURES && textureInUse[iTex])
    iTex++;
  if (iTex == MAX_TEXTURES)
  {
    fprintf(stderr,
            "Unable to allocated a texture.  Increase MAX_TEXTURES "
            "in CudaCoulomb.cu.\n");
    abort();
  }
  MyTexture          = iTex;
  textureInUse[iTex] = true;
}

TextureSpline::~TextureSpline() { textureInUse[MyTexture] = false; }


void TextureSpline::set(double data[], int numPoints, double rmin, double rmax)
{
  rMin      = rmin;
  rMax      = rmax;
  NumPoints = numPoints;
  float data_Host[numPoints];
  for (int i = 0; i < numPoints; i++)
    data_Host[i] = data[i];
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipMallocArray(&myArray, &channelDesc, numPoints);
  cudaMemcpyToArrayAsync(myArray, 0, 0, data_Host, numPoints * sizeof(float), hipMemcpyHostToDevice);
  switch (MyTexture)
  {
  case 0:
    tex00.addressMode[0] = hipAddressModeClamp;
    tex00.filterMode     = hipFilterModeLinear;
    tex00.normalized     = false;
    hipBindTextureToArray(tex00, myArray, channelDesc);
    break;
  case 1:
    tex01.addressMode[0] = hipAddressModeClamp;
    tex01.filterMode     = hipFilterModeLinear;
    tex01.normalized     = false;
    hipBindTextureToArray(tex01, myArray, channelDesc);
    break;
  case 2:
    tex02.addressMode[0] = hipAddressModeClamp;
    tex02.filterMode     = hipFilterModeLinear;
    tex02.normalized     = false;
    hipBindTextureToArray(tex02, myArray, channelDesc);
    break;
  case 3:
    tex03.addressMode[0] = hipAddressModeClamp;
    tex03.filterMode     = hipFilterModeLinear;
    tex03.normalized     = false;
    hipBindTextureToArray(tex03, myArray, channelDesc);
    break;
  case 4:
    tex04.addressMode[0] = hipAddressModeClamp;
    tex04.filterMode     = hipFilterModeLinear;
    tex04.normalized     = false;
    hipBindTextureToArray(tex04, myArray, channelDesc);
    break;
  case 5:
    tex05.addressMode[0] = hipAddressModeClamp;
    tex05.filterMode     = hipFilterModeLinear;
    tex05.normalized     = false;
    hipBindTextureToArray(tex05, myArray, channelDesc);
    break;
  case 6:
    tex06.addressMode[0] = hipAddressModeClamp;
    tex06.filterMode     = hipFilterModeLinear;
    tex06.normalized     = false;
    hipBindTextureToArray(tex06, myArray, channelDesc);
    break;
  case 7:
    tex07.addressMode[0] = hipAddressModeClamp;
    tex07.filterMode     = hipFilterModeLinear;
    tex07.normalized     = false;
    hipBindTextureToArray(tex07, myArray, channelDesc);
    break;
  case 8:
    tex08.addressMode[0] = hipAddressModeClamp;
    tex08.filterMode     = hipFilterModeLinear;
    tex08.normalized     = false;
    hipBindTextureToArray(tex08, myArray, channelDesc);
    break;
  case 9:
    tex09.addressMode[0] = hipAddressModeClamp;
    tex09.filterMode     = hipFilterModeLinear;
    tex09.normalized     = false;
    hipBindTextureToArray(tex09, myArray, channelDesc);
    break;
  }
}

__device__ float dist(float dx, float dy, float dz) { return sqrtf(dx * dx + dy * dy + dz * dz); }

__device__ double dist(double dx, double dy, double dz) { return sqrt(dx * dx + dy * dy + dz * dz); }

template<typename T>
__device__ T min_dist(T& x, T& y, T& z, T L[3][3], T Linv[3][3])
{
  T u0 = Linv[0][0] * x + Linv[0][1] * y + Linv[0][2] * z;
  T u1 = Linv[1][0] * x + Linv[1][1] * y + Linv[1][2] * z;
  T u2 = Linv[2][0] * x + Linv[2][1] * y + Linv[2][2] * z;
  u0 -= rintf(u0);
  u1 -= rintf(u1);
  u2 -= rintf(u2);
  x = L[0][0] * u0 + L[0][1] * u1 + L[0][2] * u2;
  y = L[1][0] * u0 + L[1][1] * u1 + L[1][2] * u2;
  z = L[2][0] * u0 + L[2][1] * u1 + L[2][2] * u2;
  // T u0 = Linv[0][0]*x; u0 -= rintf(u0); x = L[0][0]*u0;
  // T u1 = Linv[1][1]*y; u1 -= rintf(u1); y = L[1][1]*u1;
  // T u2 = Linv[2][2]*z; u2 -= rintf(u2); z = L[2][2]*u2;
  //  return sqrtf(x*x + y*y + z*z);
  T d2min = x * x + y * y + z * z;
  for (T i = -1.0f; i <= 1.001; i += 1.0f)
    for (T j = -1.0f; j <= 1.001; j += 1.0f)
      for (T k = -1.0f; k <= 1.001; k += 1.0f)
      {
        T xnew = L[0][0] * (u0 + i) + L[0][1] * (u1 + j) + L[0][2] * (u2 + k);
        T ynew = L[1][0] * (u0 + i) + L[1][1] * (u1 + j) + L[1][2] * (u2 + k);
        T znew = L[2][0] * (u0 + i) + L[2][1] * (u1 + j) + L[2][2] * (u2 + k);
        T d2   = xnew * xnew + ynew * ynew + znew * znew;
        d2min  = min(d2, d2min);
        if (d2 < d2min)
        {
          d2min = d2;
          x     = xnew;
          y     = ynew;
          z     = znew;
        }
      }
  return sqrt(d2min);
}


template<typename T>
__device__ T min_dist2(T& x, T& y, T& z, T L[3][3], T Linv[3][3])
{
  T u0 = Linv[0][0] * x + Linv[0][1] * y + Linv[0][2] * z;
  T u1 = Linv[1][0] * x + Linv[1][1] * y + Linv[1][2] * z;
  T u2 = Linv[2][0] * x + Linv[2][1] * y + Linv[2][2] * z;
  u0 -= rintf(u0);
  u1 -= rintf(u1);
  u2 -= rintf(u2);
  x = L[0][0] * u0 + L[0][1] * u1 + L[0][2] * u2;
  y = L[1][0] * u0 + L[1][1] * u1 + L[1][2] * u2;
  z = L[2][0] * u0 + L[2][1] * u1 + L[2][2] * u2;
  // T u0 = Linv[0][0]*x; u0 -= rintf(u0); x = L[0][0]*u0;
  // T u1 = Linv[1][1]*y; u1 -= rintf(u1); y = L[1][1]*u1;
  // T u2 = Linv[2][2]*z; u2 -= rintf(u2); z = L[2][2]*u2;
  //  return sqrtf(x*x + y*y + z*z);
  T d2min = x * x + y * y + z * z;
  for (T i = -1.0f; i <= 1.001; i += 1.0f)
    for (T j = -1.0f; j <= 1.001; j += 1.0f)
      for (T k = -1.0f; k <= 1.001; k += 1.0f)
      {
        T xnew = L[0][0] * (u0 + i) + L[0][1] * (u1 + j) + L[0][2] * (u2 + k);
        T ynew = L[1][0] * (u0 + i) + L[1][1] * (u1 + j) + L[1][2] * (u2 + k);
        T znew = L[2][0] * (u0 + i) + L[2][1] * (u1 + j) + L[2][2] * (u2 + k);
        T d2   = xnew * xnew + ynew * ynew + znew * znew;
        d2min  = min(d2, d2min);
        if (d2 < d2min)
        {
          d2min = d2;
          x     = xnew;
          y     = ynew;
          z     = znew;
        }
      }
  return d2min;
}


__device__ float recipSqrt(float x) { return rsqrtf(x); }
__device__ double recipSqrt(double x) { return rsqrt(x); }


template<typename TR, typename T, int BS>
__global__ void coulomb_AA_PBC_kernel(TR** R,
                                      int N,
                                      T rMax,
                                      int Ntex,
                                      int textureNum,
                                      T* lattice,
                                      T* latticeInv,
                                      T* sum)
{
  int tid = threadIdx.x;
  __shared__ TR* myR;
  if (tid == 0)
    myR = R[blockIdx.x];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9)
  {
    L[0][tid]    = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  __syncthreads();
  T nrm = (T)(Ntex - 1) / rMax;
  __shared__ T r1[BS][3], r2[BS][3];
  int NB  = N / BS + ((N % BS) ? 1 : 0);
  T mysum = (T)0.0;
  // Do diagonal blocks first
  for (int b = 0; b < NB; b++)
  {
    for (int i = 0; i < 3; i++)
      if ((3 * b + i) * BS + tid < 3 * N)
        r1[0][i * BS + tid] = myR[(3 * b + i) * BS + tid];
    int ptcl1 = b * BS + tid;
    if (ptcl1 < N)
    {
      int end = (b + 1) * BS < N ? BS : N - b * BS;
      for (int p2 = 0; p2 < end; p2++)
      {
        int ptcl2 = b * BS + p2;
        T dx, dy, dz;
        dx     = r1[p2][0] - r1[tid][0];
        dy     = r1[p2][1] - r1[tid][1];
        dz     = r1[p2][2] - r1[tid][2];
        T dist = min_dist(dx, dy, dz, L, Linv);
        if (ptcl1 != ptcl2)
        {
          float tval;
          arraytexFetch(nrm * dist + 0.5, textureNum, tval);
          mysum += tval / dist;
        }
        //	  mysum += dist;
      }
    }
  }
  // Avoid double-counting on the diagonal blocks
  mysum *= 0.5;
  // Now do off-diagonal blocks
  for (int b1 = 0; b1 < NB; b1++)
  {
    for (int i = 0; i < 3; i++)
      if ((3 * b1 + i) * BS + tid < 3 * N)
        r1[0][i * BS + tid] = myR[(3 * b1 + i) * BS + tid];
    int ptcl1 = b1 * BS + tid;
    if (ptcl1 < N)
    {
      for (int b2 = b1 + 1; b2 < NB; b2++)
      {
        for (int i = 0; i < 3; i++)
          if ((3 * b2 + i) * BS + tid < 3 * N)
            r2[0][i * BS + tid] = myR[(3 * b2 + i) * BS + tid];
        int end = ((b2 + 1) * BS < N) ? BS : (N - b2 * BS);
        for (int j = 0; j < end; j++)
        {
          T dx, dy, dz;
          dx     = r2[j][0] - r1[tid][0];
          dy     = r2[j][1] - r1[tid][1];
          dz     = r2[j][2] - r1[tid][2];
          T dist = min_dist(dx, dy, dz, L, Linv);
          float tval;
          arraytexFetch(nrm * dist + 0.5, textureNum, tval);
          mysum += tval / dist;
          //	  mysum += tex1D(shortTex[textureNum], nrm*dist+0.5)/dist;
        }
      }
    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] = shared_sum[0];
}

template<typename T, int BS>
__global__ void coulomb_AA_kernel(T** R, int N, T* sum)
{
  int tid = threadIdx.x;
  __shared__ T* myR;
  if (tid == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  __shared__ T r1[BS][3], r2[BS][3];
  int NB  = (N + BS - 1) / BS;
  T mysum = (T)0.0;
  // Do diagonal blocks first
  for (int b = 0; b < NB; b++)
  {
    for (int i = 0; i < 3; i++)
      if ((3 * b + i) * BS + tid < 3 * N)
        r1[0][i * BS + tid] = myR[(3 * b + i) * BS + tid];
    int ptcl1 = b * BS + tid;
    if (ptcl1 < N)
    {
      int end = (b + 1) * BS < N ? BS : N - b * BS;
      for (int p2 = 0; p2 < end; p2++)
      {
        int ptcl2 = b * BS + p2;
        T dx, dy, dz;
        dx        = r1[p2][0] - r1[tid][0];
        dy        = r1[p2][1] - r1[tid][1];
        dz        = r1[p2][2] - r1[tid][2];
        T distInv = recipSqrt(dx * dx + dy * dy + dz * dz);
        if (ptcl1 != ptcl2)
          mysum += distInv;
        //	  mysum += dist;
      }
    }
  }
  // Avoid double-counting on the diagonal blocks
  mysum *= 0.5;
  // Now do off-diagonal blocks
  for (int b1 = 0; b1 < NB; b1++)
  {
    for (int i = 0; i < 3; i++)
      if ((3 * b1 + i) * BS + tid < 3 * N)
        r1[0][i * BS + tid] = myR[(3 * b1 + i) * BS + tid];
    int ptcl1 = b1 * BS + tid;
    if (ptcl1 < N)
    {
      for (int b2 = b1 + 1; b2 < NB; b2++)
      {
        for (int i = 0; i < 3; i++)
          if ((3 * b2 + i) * BS + tid < 3 * N)
            r2[0][i * BS + tid] = myR[(3 * b2 + i) * BS + tid];
        int end = ((b2 + 1) * BS < N) ? BS : (N - b2 * BS);
        for (int j = 0; j < end; j++)
        {
          T dx, dy, dz;
          dx        = r2[j][0] - r1[tid][0];
          dy        = r2[j][1] - r1[tid][1];
          dz        = r2[j][2] - r1[tid][2];
          T distInv = recipSqrt(dx * dx + dy * dy + dz * dz);
          mysum += distInv;
        }
      }
    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] = shared_sum[0];
}


void CoulombAA_SR_Sum(float* R[],
                      int N,
                      float rMax,
                      int Ntex,
                      int textureNum,
                      float lattice[],
                      float latticeInv[],
                      float sum[],
                      int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AA_PBC_kernel<float, float, BS>
      <<<dimGrid, dimBlock>>>(R, N, rMax, Ntex, textureNum, lattice, latticeInv, sum);
}


void CoulombAA_SR_Sum(float* R[],
                      int N,
                      double rMax,
                      int Ntex,
                      int textureNum,
                      double lattice[],
                      double latticeInv[],
                      double sum[],
                      int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AA_PBC_kernel<float, double, BS>
      <<<dimGrid, dimBlock>>>(R, N, rMax, Ntex, textureNum, lattice, latticeInv, sum);
}


void CoulombAA_SR_Sum(double* R[],
                      int N,
                      double rMax,
                      int Ntex,
                      int textureNum,
                      double lattice[],
                      double latticeInv[],
                      double sum[],
                      int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AA_PBC_kernel<double, double, BS>
      <<<dimGrid, dimBlock>>>(R, N, rMax, Ntex, textureNum, lattice, latticeInv, sum);
}


void CoulombAA_Sum(float* R[], int N, float sum[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AA_kernel<float, BS><<<dimGrid, dimBlock>>>(R, N, sum);
}


void CoulombAA_Sum(double* R[], int N, double sum[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AA_kernel<double, BS><<<dimGrid, dimBlock>>>(R, N, sum);
}


template<typename T, int BS>
__global__ void MPC_SR_kernel(T** R, int N, T* lattice, T* latticeInv, T* sum)
{
  int tid = threadIdx.x;
  __shared__ T* myR;
  if (tid == 0)
    myR = R[blockIdx.x];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9)
  {
    L[0][tid]    = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  __syncthreads();
  __shared__ T r1[BS][3], r2[BS][3];
  int NB  = (N + BS - 1) / BS;
  T mysum = (T)0.0;
  // Do diagonal blocks first
  for (int b = 0; b < NB; b++)
  {
    for (int i = 0; i < 3; i++)
      if ((3 * b + i) * BS + tid < 3 * N)
        r1[0][i * BS + tid] = myR[(3 * b + i) * BS + tid];
    __syncthreads();
    int ptcl1 = b * BS + tid;
    if (ptcl1 < N)
    {
      int end = (b + 1) * BS < N ? BS : N - b * BS;
      for (int p2 = 0; p2 < end; p2++)
      {
        int ptcl2 = b * BS + p2;
        T dx, dy, dz;
        dx        = r1[p2][0] - r1[tid][0];
        dy        = r1[p2][1] - r1[tid][1];
        dz        = r1[p2][2] - r1[tid][2];
        T distinv = recipSqrt(min_dist2(dx, dy, dz, L, Linv));
        if (ptcl1 != ptcl2)
          mysum += distinv;
      }
    }
  }
  // Avoid double-counting on the diagonal blocks
  mysum *= 0.5;
  // Now do off-diagonal blocks
  for (int b1 = 0; b1 < NB; b1++)
  {
    for (int i = 0; i < 3; i++)
      if ((3 * b1 + i) * BS + tid < 3 * N)
        r1[0][i * BS + tid] = myR[(3 * b1 + i) * BS + tid];
    __syncthreads();
    int ptcl1 = b1 * BS + tid;
    if (ptcl1 < N)
    {
      for (int b2 = b1 + 1; b2 < NB; b2++)
      {
        for (int i = 0; i < 3; i++)
          if ((3 * b2 + i) * BS + tid < 3 * N)
            r2[0][i * BS + tid] = myR[(3 * b2 + i) * BS + tid];
        int end = ((b2 + 1) * BS < N) ? BS : (N - b2 * BS);
        for (int j = 0; j < end; j++)
        {
          T dx, dy, dz;
          dx        = r2[j][0] - r1[tid][0];
          dy        = r2[j][1] - r1[tid][1];
          dz        = r2[j][2] - r1[tid][2];
          T distinv = recipSqrt(min_dist2(dx, dy, dz, L, Linv));
          mysum += distinv;
        }
      }
    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] = shared_sum[0];
}


void MPC_SR_Sum(float* R[], int N, float lattice[], float latticeInv[], float sum[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  MPC_SR_kernel<float, BS><<<dimGrid, dimBlock>>>(R, N, lattice, latticeInv, sum);
}


void MPC_SR_Sum(double* R[], int N, double lattice[], double latticeInv[], double sum[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  MPC_SR_kernel<double, BS><<<dimGrid, dimBlock>>>(R, N, lattice, latticeInv, sum);
}

template<typename T>
struct Three
{};
template<>
struct Three<float>
{
  typedef float3 type;
};
template<>
struct Three<double>
{
  typedef double3 type;
};


template<typename T, int BS>
__global__ void MPC_LR_kernel(T** R,
                              int N,
                              T* coefs,
                              typename Three<T>::type gridInv,
                              uint3 dim,
                              uint3 strides,
                              T* latticeInv,
                              T* sum)
{
  int tid = threadIdx.x;
  __shared__ T r[BS][3], u[BS][3], Linv[3][3];
  __shared__ int index[BS][3];
  __shared__ T* myR;
  if (tid < 9)
    Linv[0][tid] = latticeInv[tid];
  if (tid == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  int numBlocks = (N + BS - 1) / BS;
  T myval       = T();
  for (int block = 0; block < numBlocks; block++)
  {
    for (int i = 0; i < 3; i++)
    {
      int off = (3 * block + i) * BS + tid;
      if (off < 3 * N)
        r[0][i * BS + tid] = myR[off];
    }
    __syncthreads();
    u[tid][0] = (Linv[0][0] * r[tid][0] + Linv[0][1] * r[tid][1] + Linv[0][2] * r[tid][2]);
    u[tid][1] = (Linv[1][0] * r[tid][0] + Linv[1][1] * r[tid][1] + Linv[1][2] * r[tid][2]);
    u[tid][2] = (Linv[2][0] * r[tid][0] + Linv[2][1] * r[tid][1] + Linv[2][2] * r[tid][2]);
    u[tid][0] -= floor(u[tid][0]);
    u[tid][1] -= floor(u[tid][1]);
    u[tid][2] -= floor(u[tid][2]);
    // We don't need r anymore, so we can now reuse to store t.
    T s, sf;
    s             = u[tid][0] * gridInv.x;
    sf            = floor(s);
    index[tid][0] = min(max(0, (int)sf), dim.x - 1);
    r[tid][0]     = s - sf;
    s             = u[tid][1] * gridInv.y;
    sf            = floor(s);
    index[tid][1] = min(max(0, (int)sf), dim.y - 1);
    r[tid][1]     = s - sf;
    s             = u[tid][2] * gridInv.z;
    sf            = floor(s);
    index[tid][2] = min(max(0, (int)sf), dim.z - 1);
    r[tid][2]     = s - sf;
    int end       = min(BS, N - block * BS);
    // This loop assumes BS=32
    for (int i = 0; i < end; i++)
    {
      __shared__ T a[4][3];
      if (tid < 12)
      {
        int j   = tid >> 2;
        int k   = tid & 3;
        T t     = r[i][j];
        a[k][j] = (Acuda[4 * k + 0] * t * t * t + Acuda[4 * k + 1] * t * t + Acuda[4 * k + 2] * t + Acuda[4 * k + 3]);
      }
      __syncthreads();
      // There are 64 elements to sum.  With BS=32, we use 2 passes
      // First 32 coefs
      int ix  = tid >> 4;
      int iy  = (tid >> 2) & 3;
      int iz  = (tid & 3);
      T abc   = a[ix][0] * a[iy][1] * a[iz][2];
      int off = ((index[i][0] + ix) * strides.x + (index[i][1] + iy) * strides.y + (index[i][2] + iz));
      myval += abc * coefs[off];
      // Second 32 coefs
      ix += 2;
      abc = a[ix][0] * a[iy][1] * a[iz][2];
      off = ((index[i][0] + ix) * strides.x + (index[i][1] + iy) * strides.y + (index[i][2] + iz));
      myval += abc * coefs[off];
    }
  }
  __syncthreads();
  // reuse u for reduction
  u[0][tid] = myval;
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      u[0][tid] += u[0][tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] = u[0][0];
}


void MPC_LR_Sum(float* R[], int N, UBspline_3d_s_cuda* spline, float latticeInv[], float sum[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  MPC_LR_kernel<float, BS>
      <<<dimGrid, dimBlock>>>(R, N, spline->coefs, spline->gridInv, spline->dim, spline->stride, latticeInv, sum);
}

void MPC_LR_Sum(double* R[], int N, UBspline_3d_d_cuda* spline, double latticeInv[], double sum[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  MPC_LR_kernel<double, BS>
      <<<dimGrid, dimBlock>>>(R, N, spline->coefs, spline->gridInv, spline->dim, spline->stride, latticeInv, sum);
}


template<typename TR, typename T, int BS>
__global__ void coulomb_AB_PBC_kernel(TR** R,
                                      int Nelec,
                                      TR* I,
                                      int Ifirst,
                                      int Ilast,
                                      T rMax,
                                      int Ntex,
                                      int textureNum,
                                      T* lattice,
                                      T* latticeInv,
                                      T* sum)
{
  int tid = threadIdx.x;
  __shared__ TR* myR;
  int Nion = Ilast - Ifirst + 1;
  if (tid == 0)
    myR = R[blockIdx.x];
  __shared__ T L[3][3], Linv[3][3];
  if (tid < 9)
  {
    L[0][tid]    = lattice[tid];
    Linv[0][tid] = latticeInv[tid];
  }
  __syncthreads();
  T nrm = (T)(Ntex - 1) / rMax;
  __shared__ T r[BS][3], i[BS][3];
  int NeBlocks = Nelec / BS + ((Nelec % BS) ? 1 : 0);
  int NiBlocks = Nion / BS + ((Nion % BS) ? 1 : 0);
  T mysum      = (T)0.0;
  // Now do off-diagonal blocks
  for (int iBlock = 0; iBlock < NiBlocks; iBlock++)
  {
    for (int j = 0; j < 3; j++)
      if ((3 * iBlock + j) * BS + tid < 3 * Nion)
        i[0][j * BS + tid] = I[3 * Ifirst + (3 * iBlock + j) * BS + tid];
    __syncthreads();
    int ion = iBlock * BS + tid;
    for (int eBlock = 0; eBlock < NeBlocks; eBlock++)
    {
      for (int j = 0; j < 3; j++)
        if ((3 * eBlock + j) * BS + tid < 3 * Nelec)
          r[0][j * BS + tid] = myR[(3 * eBlock + j) * BS + tid];
      __syncthreads();
      int end = ((eBlock + 1) * BS < Nelec) ? BS : (Nelec - eBlock * BS);
      if (ion < Nion)
      {
        for (int j = 0; j < end; j++)
        {
          T dx, dy, dz;
          dx     = r[j][0] - i[tid][0];
          dy     = r[j][1] - i[tid][1];
          dz     = r[j][2] - i[tid][2];
          T dist = min_dist(dx, dy, dz, L, Linv);
          float tval;
          arraytexFetch(nrm * dist + 0.5, textureNum, tval);
          mysum += tval / dist;
        }
      }
      __syncthreads();
    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] = shared_sum[0];
}


void CoulombAB_SR_Sum(float* R[],
                      int Nelec,
                      float I[],
                      int Ifirst,
                      int Ilast,
                      float rMax,
                      int Ntex,
                      int textureNum,
                      float lattice[],
                      float latticeInv[],
                      float sum[],
                      int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AB_PBC_kernel<float, float, BS>
      <<<dimGrid, dimBlock>>>(R, Nelec, I, Ifirst, Ilast, rMax, Ntex, textureNum, lattice, latticeInv, sum);
}


void CoulombAB_SR_Sum(float* R[],
                      int Nelec,
                      float I[],
                      int Ifirst,
                      int Ilast,
                      double rMax,
                      int Ntex,
                      int textureNum,
                      double lattice[],
                      double latticeInv[],
                      double sum[],
                      int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AB_PBC_kernel<float, double, BS>
      <<<dimGrid, dimBlock>>>(R, Nelec, I, Ifirst, Ilast, rMax, Ntex, textureNum, lattice, latticeInv, sum);
}


void CoulombAB_SR_Sum(double* R[],
                      int Nelec,
                      double I[],
                      int Ifirst,
                      int Ilast,
                      double rMax,
                      int Ntex,
                      int textureNum,
                      double lattice[],
                      double latticeInv[],
                      double sum[],
                      int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AB_PBC_kernel<double, double, BS>
      <<<dimGrid, dimBlock>>>(R, Nelec, I, Ifirst, Ilast, rMax, Ntex, textureNum, lattice, latticeInv, sum);
}


template<typename T, int BS>
__global__ void local_ecp_kernel(T** R,
                                 int Nelec,
                                 T* I,
                                 int Ifirst,
                                 int Ilast,
                                 T rMax,
                                 int Ntex,
                                 int textureNum,
                                 T* sum)
{
  int tid = threadIdx.x;
  __shared__ T* myR;
  int Nion = Ilast - Ifirst + 1;
  if (tid == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  T nrm = (T)(Ntex - 1) / rMax;
  __shared__ T r[BS][3], i[BS][3];
  int NeBlocks = Nelec / BS + ((Nelec % BS) ? 1 : 0);
  int NiBlocks = Nion / BS + ((Nion % BS) ? 1 : 0);
  T mysum      = (T)0.0;
  // Now do off-diagonal blocks
  for (int iBlock = 0; iBlock < NiBlocks; iBlock++)
  {
    for (int j = 0; j < 3; j++)
      if ((3 * iBlock + j) * BS + tid < 3 * Nion)
        i[0][j * BS + tid] = I[3 * Ifirst + (3 * iBlock + j) * BS + tid];
    __syncthreads();
    int ion = iBlock * BS + tid;
    for (int eBlock = 0; eBlock < NeBlocks; eBlock++)
    {
      for (int j = 0; j < 3; j++)
        if ((3 * eBlock + j) * BS + tid < 3 * Nelec)
          r[0][j * BS + tid] = myR[(3 * eBlock + j) * BS + tid];
      __syncthreads();
      int end = ((eBlock + 1) * BS < Nelec) ? BS : (Nelec - eBlock * BS);
      if (ion < Nion)
      {
        for (int j = 0; j < end; j++)
        {
          T dx, dy, dz;
          dx  = r[j][0] - i[tid][0];
          dy  = r[j][1] - i[tid][1];
          dz  = r[j][2] - i[tid][2];
          T d = dist(dx, dy, dz);
          float tval;
          arraytexFetch(nrm * d + 0.5, textureNum, tval);
          mysum += tval / d;
        }
      }
      __syncthreads();
    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] = shared_sum[0];
}


void local_ecp_sum(float* R[],
                   int Nelec,
                   float I[],
                   int Ifirst,
                   int Ilast,
                   float rMax,
                   int Ntex,
                   int textureNum,
                   float sum[],
                   int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  local_ecp_kernel<float, BS><<<dimGrid, dimBlock>>>(R, Nelec, I, Ifirst, Ilast, rMax, Ntex, textureNum, sum);
}


void local_ecp_sum(double* R[],
                   int Nelec,
                   double I[],
                   int Ifirst,
                   int Ilast,
                   double rMax,
                   int Ntex,
                   int textureNum,
                   double sum[],
                   int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  local_ecp_kernel<double, BS><<<dimGrid, dimBlock>>>(R, Nelec, I, Ifirst, Ilast, rMax, Ntex, textureNum, sum);
}


template<typename T, int BS>
__global__ void coulomb_AB_kernel(T** R, int Nelec, T* I, T* Zion, int Nion, T* sum)
{
  int tid = threadIdx.x;
  __shared__ T* myR;
  if (tid == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  __shared__ T r[BS][3], i[BS][3], z[BS];
  int NeBlocks = Nelec / BS + ((Nelec % BS) ? 1 : 0);
  int NiBlocks = Nion / BS + ((Nion % BS) ? 1 : 0);
  T mysum      = (T)0.0;
  // Now do off-diagonal blocks
  for (int iBlock = 0; iBlock < NiBlocks; iBlock++)
  {
    for (int j = 0; j < 3; j++)
      if ((3 * iBlock + j) * BS + tid < 3 * Nion)
        i[0][j * BS + tid] = I[(3 * iBlock + j) * BS + tid];
    if (tid < Nion)
      z[tid] = Zion[tid];
    __syncthreads();
    int ion = iBlock * BS + tid;
    for (int eBlock = 0; eBlock < NeBlocks; eBlock++)
    {
      for (int j = 0; j < 3; j++)
        if ((3 * eBlock + j) * BS + tid < 3 * Nelec)
          r[0][j * BS + tid] = myR[(3 * eBlock + j) * BS + tid];
      __syncthreads();
      int end = ((eBlock + 1) * BS < Nelec) ? BS : (Nelec - eBlock * BS);
      if (ion < Nion)
      {
        for (int j = 0; j < end; j++)
        {
          T dx, dy, dz;
          dx        = r[j][0] - i[tid][0];
          dy        = r[j][1] - i[tid][1];
          dz        = r[j][2] - i[tid][2];
          T distInv = recipSqrt(dx * dx + dy * dy + dz * dz);
          mysum -= z[tid] * distInv;
        }
      }
      __syncthreads();
    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] = shared_sum[0];
}


void CoulombAB_Sum(float* R[], int Nelec, float I[], float Zion[], int Nion, float sum[], int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AB_kernel<float, BS><<<dimGrid, dimBlock>>>(R, Nelec, I, Zion, Nion, sum);
}


void CoulombAB_Sum(double* R[], int Nelec, double I[], double Zion[], int Nion, double sum[], int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  coulomb_AB_kernel<double, BS><<<dimGrid, dimBlock>>>(R, Nelec, I, Zion, Nion, sum);
}


template<typename T, int BS>
__global__ void eval_rhok_kernel(T** R, int numr, T* kpoints, int numk, T** rhok)
{
  int tid = threadIdx.x;
  __shared__ T r[BS][3], k[BS][3], *myR, *myrhok;
  if (tid == 0)
  {
    myR    = R[blockIdx.x];
    myrhok = rhok[blockIdx.x];
  }
  __syncthreads();
  int NrBlock = numr / BS + ((numr % BS) ? 1 : 0);
  int NkBlock = numk / BS + ((numk % BS) ? 1 : 0);
  __shared__ T rhok_re[BS], rhok_im[BS], rhok_s[2 * BS];
  for (int kBlock = 0; kBlock < NkBlock; kBlock++)
  {
    for (int i = 0; i < 3; i++)
      if ((i + 3 * kBlock) * BS + tid < 3 * numk)
        k[0][BS * i + tid] = kpoints[(i + 3 * kBlock) * BS + tid];
    rhok_re[tid] = rhok_im[tid] = 0.0f;
    for (int rBlock = 0; rBlock < NrBlock; rBlock++)
    {
      for (int i = 0; i < 3; i++)
        if ((i + 3 * rBlock) * BS + tid < 3 * numr)
          r[0][BS * i + tid] = myR[(i + 3 * rBlock) * BS + tid];
      int end = ((rBlock + 1) * BS < numr) ? BS : (numr - rBlock * BS);
      for (int j = 0; j < end; j++)
      {
        T phase = (k[tid][0] * r[j][0] + k[tid][1] * r[j][1] + k[tid][2] * r[j][2]);
        T s, c;
        sincos(phase, &s, &c);
        rhok_im[tid] += s;
        rhok_re[tid] += c;
      }
    }
    // Write rhok to global memory
    rhok_s[2 * tid + 0] = rhok_re[tid];
    rhok_s[2 * tid + 1] = rhok_im[tid];
    __syncthreads();
    if (2 * (kBlock * BS) + tid < 2 * numk)
      myrhok[2 * (kBlock * BS) + tid] = rhok_s[tid];
    if (2 * (kBlock * BS) + tid + BS < 2 * numk)
      myrhok[2 * (kBlock * BS) + tid + BS] = rhok_s[tid + BS];
  }
}


template<typename TR, typename T, int BS>
__global__ void eval_rhok_kernel(TR** R, int first, int last, T* kpoints, int numk, T** rhok)
{
  int tid  = threadIdx.x;
  int numr = last - first + 1;
  __shared__ TR* myR;
  __shared__ T r[BS][3], k[BS][3], *myrhok;
  if (tid == 0)
  {
    myR    = R[blockIdx.x];
    myrhok = rhok[blockIdx.x];
  }
  __syncthreads();
  int NrBlock = numr / BS + ((numr % BS) ? 1 : 0);
  int NkBlock = numk / BS + ((numk % BS) ? 1 : 0);
  __shared__ T rhok_re[BS], rhok_im[BS], rhok_s[2 * BS];
  for (int kBlock = 0; kBlock < NkBlock; kBlock++)
  {
    for (int i = 0; i < 3; i++)
      if ((i + 3 * kBlock) * BS + tid < 3 * numk)
        k[0][BS * i + tid] = kpoints[(i + 3 * kBlock) * BS + tid];
    rhok_re[tid] = rhok_im[tid] = 0.0f;
    for (int rBlock = 0; rBlock < NrBlock; rBlock++)
    {
      for (int i = 0; i < 3; i++)
        if ((i + 3 * rBlock) * BS + tid < 3 * numr)
          r[0][BS * i + tid] = myR[3 * first + (i + 3 * rBlock) * BS + tid];
      int end = ((rBlock + 1) * BS < numr) ? BS : (numr - rBlock * BS);
      for (int j = 0; j < end; j++)
      {
        T phase = (k[tid][0] * r[j][0] + k[tid][1] * r[j][1] + k[tid][2] * r[j][2]);
        T s, c;
        sincos(phase, &s, &c);
        rhok_im[tid] += s;
        rhok_re[tid] += c;
      }
    }
    // Write rhok to global memory
    rhok_s[2 * tid + 0] = rhok_re[tid];
    rhok_s[2 * tid + 1] = rhok_im[tid];
    __syncthreads();
    if (2 * (kBlock * BS) + tid < 2 * numk)
      myrhok[2 * (kBlock * BS) + tid] = rhok_s[tid];
    if (2 * (kBlock * BS) + tid + BS < 2 * numk)
      myrhok[2 * (kBlock * BS) + tid + BS] = rhok_s[tid + BS];
  }
}


void eval_rhok_cuda(float* R[], int numr, float kpoints[], int numk, float* rhok[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  eval_rhok_kernel<float, BS><<<dimGrid, dimBlock>>>(R, numr, kpoints, numk, rhok);
}

void eval_rhok_cuda(double* R[], int numr, double kpoints[], int numk, double* rhok[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  eval_rhok_kernel<double, BS><<<dimGrid, dimBlock>>>(R, numr, kpoints, numk, rhok);
}


void eval_rhok_cuda(float* R[], int first, int last, float kpoints[], int numk, float* rhok[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  eval_rhok_kernel<float, float, BS><<<dimGrid, dimBlock>>>(R, first, last, kpoints, numk, rhok);
}

void eval_rhok_cuda(float* R[], int first, int last, double kpoints[], int numk, double* rhok[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  eval_rhok_kernel<float, double, BS><<<dimGrid, dimBlock>>>(R, first, last, kpoints, numk, rhok);
}

void eval_rhok_cuda(double* R[], int first, int last, double kpoints[], int numk, double* rhok[], int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  eval_rhok_kernel<double, double, BS><<<dimGrid, dimBlock>>>(R, first, last, kpoints, numk, rhok);
}


template<typename T, int BS>
__global__ void vk_sum_kernel(T** rhok, T* vk, int numk, T* sum)
{
  int tid = threadIdx.x;
  __shared__ T* myrhok;
  if (tid == 0)
    myrhok = rhok[blockIdx.x];
  __syncthreads();
  // Used to do coalesced global loads
  __shared__ T rhok_s[2 * BS];
  int NB  = numk / BS + ((numk % BS) ? 1 : 0);
  T mysum = 0.0f;
  for (int b = 0; b < NB; b++)
  {
    if (2 * b * BS + tid < 2 * numk)
      rhok_s[tid] = myrhok[2 * b * BS + tid];
    if ((2 * b + 1) * BS + tid < 2 * numk)
      rhok_s[BS + tid] = myrhok[(2 * b + 1) * BS + tid];
    __syncthreads();
    if (b * BS + tid < numk)
      mysum +=
          vk[b * BS + tid] * (rhok_s[2 * tid + 0] * rhok_s[2 * tid + 0] + rhok_s[2 * tid + 1] * rhok_s[2 * tid + 1]);
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  // Not sure if this 0.25 factor is correct.
  if (tid == 0)
    sum[blockIdx.x] += 0.25 * shared_sum[0];
}


template<typename T, int BS>
__global__ void vk_sum_kernel2(T** rhok1, T** rhok2, T* vk, int numk, T* sum)
{
  int tid = threadIdx.x;
  __shared__ T *myrhok1, *myrhok2;
  if (tid == 0)
  {
    myrhok1 = rhok1[blockIdx.x];
    myrhok2 = rhok2[blockIdx.x];
  }
  __syncthreads();
  // Used to do coalesced global loads
  __shared__ T rhok_s1[2 * BS], rhok_s2[2 * BS];
  int NB  = numk / BS + ((numk % BS) ? 1 : 0);
  T mysum = 0.0f;
  for (int b = 0; b < NB; b++)
  {
    if (2 * b * BS + tid < 2 * numk)
    {
      rhok_s1[tid] = myrhok1[2 * b * BS + tid];
      rhok_s2[tid] = myrhok2[2 * b * BS + tid];
    }
    if ((2 * b + 1) * BS + tid < 2 * numk)
    {
      rhok_s1[BS + tid] = myrhok1[(2 * b + 1) * BS + tid];
      rhok_s2[BS + tid] = myrhok2[(2 * b + 1) * BS + tid];
    }
    __syncthreads();
    if (b * BS + tid < numk)
      mysum += vk[b * BS + tid] *
          (rhok_s1[2 * tid + 0] * rhok_s2[2 * tid + 0] + rhok_s1[2 * tid + 1] * rhok_s2[2 * tid + 1]);
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  T factor = (myrhok1 == myrhok2) ? 0.5f : 1.0f;
  if (tid == 0)
    sum[blockIdx.x] += factor * shared_sum[0];
}


void eval_vk_sum_cuda(float* rhok[], float vk[], int numk, float sum[], int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  vk_sum_kernel<float, BS><<<dimGrid, dimBlock>>>(rhok, vk, numk, sum);
}

void eval_vk_sum_cuda(double* rhok[], double vk[], int numk, double sum[], int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  vk_sum_kernel<double, BS><<<dimGrid, dimBlock>>>(rhok, vk, numk, sum);
}


void eval_vk_sum_cuda(float* rhok1[], float* rhok2[], float vk[], int numk, float sum[], int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  vk_sum_kernel2<float, BS><<<dimGrid, dimBlock>>>(rhok1, rhok2, vk, numk, sum);
}


void eval_vk_sum_cuda(double* rhok1[], double* rhok2[], double vk[], int numk, double sum[], int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  vk_sum_kernel2<double, BS><<<dimGrid, dimBlock>>>(rhok1, rhok2, vk, numk, sum);
}


template<typename T, int BS>
__global__ void vk_sum_kernel2(T** rhok1, T* rhok2, T* vk, int numk, T* sum)
{
  int tid = threadIdx.x;
  __shared__ T* myrhok1;
  if (tid == 0)
    myrhok1 = rhok1[blockIdx.x];
  __syncthreads();
  // Used to do coalesced global loads
  __shared__ T rhok_s1[2 * BS], rhok_s2[2 * BS];
  int NB  = numk / BS + ((numk % BS) ? 1 : 0);
  T mysum = 0.0f;
  for (int b = 0; b < NB; b++)
  {
    if (2 * b * BS + tid < 2 * numk)
    {
      rhok_s1[tid] = myrhok1[2 * b * BS + tid];
      rhok_s2[tid] = rhok2[2 * b * BS + tid];
    }
    if ((2 * b + 1) * BS + tid < 2 * numk)
    {
      rhok_s1[BS + tid] = myrhok1[(2 * b + 1) * BS + tid];
      rhok_s2[BS + tid] = rhok2[(2 * b + 1) * BS + tid];
    }
    __syncthreads();
    if (b * BS + tid < numk)
      mysum += vk[b * BS + tid] *
          (rhok_s1[2 * tid + 0] * rhok_s2[2 * tid + 0] + rhok_s1[2 * tid + 1] * rhok_s2[2 * tid + 1]);
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] += shared_sum[0];
}

void eval_vk_sum_cuda(float* rhok1[], float rhok2[], float vk[], int numk, float sum[], int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  vk_sum_kernel2<float, BS><<<dimGrid, dimBlock>>>(rhok1, rhok2, vk, numk, sum);
}

void eval_vk_sum_cuda(double* rhok1[], double rhok2[], double vk[], int numk, double sum[], int numWalkers)
{
  const int BS = 64;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  vk_sum_kernel2<double, BS><<<dimGrid, dimBlock>>>(rhok1, rhok2, vk, numk, sum);
}


#ifdef CUDA_COULOMB_TEST


__global__ void test_texture_kernel(float x[], float vals[], int Ntex, int Nvals)
{
  float nrm = (float)(Ntex - 1) / (float)Ntex;
  for (int i = 0; i < Nvals; i++)
    vals[i] = tex1D(myTex, nrm * x[i] + 0.5);
}

#include <stdio.h>

void TestTexture()
{
  int Ntex    = 2000;
  int Npoints = 31415;
  hipArray* myArray;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipMallocArray(&myArray, &channelDesc, Ntex);
  float data[Ntex];
  for (int i = 0; i < Ntex; i++)
  {
    double x = (double)i / (double)(Ntex - 1) * 2.0 * M_PI;
    data[i]  = (float)sin(x);
  }
  cudaMemcpyToArrayAsync(myArray, 0, 0, data, Ntex * sizeof(float), hipMemcpyHostToDevice);
  myTex.addressMode[0] = hipAddressModeClamp;
  myTex.filterMode     = hipFilterModeLinear;
  myTex.normalized     = false;
  hipBindTextureToArray(myTex, myArray, channelDesc);
  float *x_d, *vals_d;
  hipMalloc((void**)&x_d, Npoints * sizeof(float));
  hipMalloc((void**)&vals_d, Npoints * sizeof(float));
  float x_host[Npoints];
  for (int i = 0; i < Npoints; i++)
    x_host[i] = (double)i / (double)(Npoints - 1) * (double)Ntex;
  hipMemcpyAsync(x_d, x_host, Npoints * sizeof(float), hipMemcpyHostToDevice);
  dim3 dimBlock(1);
  dim3 dimGrid(1);
  test_texture_kernel<<<dimGrid, dimBlock>>>(x_d, vals_d, Ntex, Npoints);
  float vals_host[Npoints];
  hipMemcpy(vals_host, vals_d, Npoints * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < Npoints; i++)
    fprintf(stderr, "%18.10f %18.10f\n", sin(2.0 * M_PI * x_host[i] / (double)Ntex), vals_host[i]);
}


main() { TestTexture(); }
#endif
