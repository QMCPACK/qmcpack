#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2020 QMCPACK developers.
//
// File developed by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//////////////////////////////////////////////////////////////////////////////////////


#include "QMCWaveFunctions/Fermion/matrix_update_helper.hpp"

namespace qmcplusplus
{
/** interface to cuBLAS_inhouse calls for different data types S/C/D/Z
 */
namespace CUDA
{
template<typename T, int COLBS>
__global__ void copyAinvRow_saveGL_kernel(const int rowchanged,
                                          const int n,
                                          const T* const Ainv[],
                                          const int lda,
                                          T* const temp[],
                                          T* const rcopy[],
                                          const T* const dphi_in[],
                                          const T* const d2phi_in[],
                                          T* const dphi_out[],
                                          T* const d2phi_out[])
{
  const int iw                      = blockIdx.x;
  const T* __restrict__ Ainv_iw     = Ainv[iw];
  T* __restrict__ temp_iw           = temp[iw];
  T* __restrict__ rcopy_iw          = rcopy[iw];
  const T* __restrict__ dphi_in_iw  = dphi_in[iw];
  const T* __restrict__ d2phi_in_iw = d2phi_in[iw];
  T* __restrict__ dphi_out_iw       = dphi_out[iw];
  T* __restrict__ d2phi_out_iw      = d2phi_out[iw];

  const int tid = threadIdx.x;
  if (tid == 0)
    temp_iw[rowchanged] -= T(1);

  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  for (int ib = 0; ib < num_col_blocks; ib++)
  {
    const int col_id = ib * COLBS + threadIdx.x;
    if (col_id < n)
    {
      rcopy_iw[col_id] = Ainv_iw[rowchanged * lda + col_id];

      // the following copying data on the device is not part of SM-1
      // it is intended to copy dphiV and d2phiV from temporary to final without a separate kernel.
      dphi_out_iw[col_id * 3]     = dphi_in_iw[col_id * 3];
      dphi_out_iw[col_id * 3 + 1] = dphi_in_iw[col_id * 3 + 1];
      dphi_out_iw[col_id * 3 + 2] = dphi_in_iw[col_id * 3 + 2];
      d2phi_out_iw[col_id]        = d2phi_in_iw[col_id];
    }
  }
}

hipError_t copyAinvRow_saveGL_cuda(hipStream_t& hstream,
                                    const int rowchanged,
                                    const int n,
                                    const float* const Ainv[],
                                    const int lda,
                                    float* const temp[],
                                    float* const rcopy[],
                                    const float* const dphi_in[],
                                    const float* const d2phi_in[],
                                    float* const dphi_out[],
                                    float* const d2phi_out[],
                                    const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  copyAinvRow_saveGL_kernel<float, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(rowchanged, n, Ainv, lda, temp, rcopy,
                                                                             dphi_in, d2phi_in, dphi_out, d2phi_out);
  return hipPeekAtLastError();
}

hipError_t copyAinvRow_saveGL_cuda(hipStream_t& hstream,
                                    const int rowchanged,
                                    const int n,
                                    const double* const Ainv[],
                                    const int lda,
                                    double* const temp[],
                                    double* const rcopy[],
                                    const double* const dphi_in[],
                                    const double* const d2phi_in[],
                                    double* const dphi_out[],
                                    double* const d2phi_out[],
                                    const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  copyAinvRow_saveGL_kernel<double, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(rowchanged, n, Ainv, lda, temp, rcopy,
                                                                              dphi_in, d2phi_in, dphi_out, d2phi_out);
  return hipPeekAtLastError();
}

template<typename T, int COLBS, int DIM = 3>
__global__ void calcGradients_kernel(const int n,
                                     const T* const Ainvrow[],
                                     const T* const dpsiMrow[],
                                     T* const grads_now)
{
  const int iw                    = blockIdx.x;
  const T* __restrict__ invRow    = Ainvrow[iw];
  const T* __restrict__ dpsiM_row = dpsiMrow[iw];

  __shared__ T sum[DIM][COLBS];
  const int tid = threadIdx.x;
  for (int idim = 0; idim < DIM; idim++)
    sum[idim][tid] = T(0);

  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  for (int ib = 0; ib < num_col_blocks; ib++)
  {
    const int col_id = ib * COLBS + tid;
    for (int idim = 0; idim < DIM; idim++)
      if (col_id < n)
        sum[idim][tid] += invRow[col_id] * dpsiM_row[col_id * DIM + idim];
  }

  for (int iend = COLBS / 2; iend > 0; iend /= 2)
  {
    __syncthreads();
    for (int idim = 0; idim < DIM; idim++)
      if (tid < iend)
        sum[idim][tid] += sum[idim][tid + iend];
  }

  if (tid == 0)
    for (int idim = 0; idim < DIM; idim++)
      grads_now[iw * DIM + idim] = sum[idim][0];
}

hipError_t calcGradients_cuda(hipStream_t& hstream,
                               const int n,
                               const float* const Ainvrow[],
                               const float* const dpsiMrow[],
                               float* const grads_now,
                               const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  calcGradients_kernel<float, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, Ainvrow, dpsiMrow, grads_now);
  return hipPeekAtLastError();
}

hipError_t calcGradients_cuda(hipStream_t& hstream,
                               const int n,
                               const double* const Ainvrow[],
                               const double* const dpsiMrow[],
                               double* const grads_now,
                               const int batch_count)
{
  if (batch_count == 0)
    return hipSuccess;

  const int COLBS = 64;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_count);
  calcGradients_kernel<double, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, Ainvrow, dpsiMrow, grads_now);
  return hipPeekAtLastError();
}

} // namespace CUDA
} // namespace qmcplusplus
