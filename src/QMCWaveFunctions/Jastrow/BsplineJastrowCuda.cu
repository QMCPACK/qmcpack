#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by: Ken Esler, kpesler@gmail.com, University of Illinois at Urbana-Champaign
//                    Luke Shulenburger, lshulen@sandia.gov, Sandia National Laboratories
//                    Jeremy McMinnis, jmcminis@gmail.com, University of Illinois at Urbana-Champaign
//                    Jeongnim Kim, jeongnim.kim@gmail.com, University of Illinois at Urbana-Champaign
//                    Paul R. C. Kent, kentpr@ornl.gov, Oak Ridge National Laboratory
//                    Ye Luo, yeluo@anl.gov, Argonne National Laboratory
//
// File created by: Ken Esler, kpesler@gmail.com, University of Illinois at Urbana-Champaign
//////////////////////////////////////////////////////////////////////////////////////


#define MAX_SPLINES 100
#include <stdio.h>
#include "BsplineJastrowCuda.h"
#include "../../CUDA/gpu_misc.h"

bool AisInitialized = false;

__constant__ float AcudaSpline[48];
__constant__ double AcudaSpline_double[48];

inline __device__ float recipSqrt(float x) { return rsqrtf(x); }
inline __device__ double recipSqrt(double x) { return rsqrt(x); }

inline __device__ float dist(float dx, float dy, float dz) { return sqrtf(dx * dx + dy * dy + dz * dz); }

inline __device__ double dist(double dx, double dy, double dz) { return sqrt(dx * dx + dy * dy + dz * dz); }


void cuda_spline_init()
{
  // clang-format off
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
                    3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
                    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
                    1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
                    0.0,     -0.5,      1.0,    -0.5,
                    0.0,      1.5,     -2.0,     0.0,
                    0.0,     -1.5,      1.0,     0.5,
                    0.0,      0.5,      0.0,     0.0,
                    0.0,      0.0,     -1.0,     1.0,
                    0.0,      0.0,      3.0,    -2.0,
                    0.0,      0.0,     -3.0,     1.0,
                    0.0,      0.0,      1.0,     0.0
                  };
  hipMemcpyToSymbol(HIP_SYMBOL(AcudaSpline), A_h, 48*sizeof(float), 0,
                     hipMemcpyHostToDevice);
  double A_d[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
                     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
                     -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
                     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
                     0.0,     -0.5,      1.0,    -0.5,
                     0.0,      1.5,     -2.0,     0.0,
                     0.0,     -1.5,      1.0,     0.5,
                     0.0,      0.5,      0.0,     0.0,
                     0.0,      0.0,     -1.0,     1.0,
                     0.0,      0.0,      3.0,    -2.0,
                     0.0,      0.0,     -3.0,     1.0,
                     0.0,      0.0,      1.0,     0.0
                   };
  hipMemcpyToSymbol(HIP_SYMBOL(AcudaSpline_double), A_d, 48*sizeof(double), 0,
                     hipMemcpyHostToDevice);
  AisInitialized = true;
  // clang-format on
}


template<typename T>
__device__ __forceinline__ T eval_1d_spline(const T dist, const T rmax, const T drInv, const T A[4][4], T coefs[])
{
  if (dist >= rmax)
    return (T)0.0;
  T s = dist * drInv;
  T sf;
  T t       = modff(s, (float*)&sf);
  int index = (int)sf;
  T t2      = t * t;
  T t3      = t * t2;
  T coefs0  = coefs[index];
  T coefs1  = coefs[index + 1];
  T coefs2  = coefs[index + 2];
  T coefs3  = coefs[index + 3];
  T val0    = A[0][0] * t3 + A[0][1] * t2 + A[0][2] * t + A[0][3];
  T val1    = A[1][0] * t3 + A[1][1] * t2 + A[1][2] * t + A[1][3];
  T val2    = A[2][0] * t3 + A[2][1] * t2 + A[2][2] * t + A[2][3];
  T val3    = A[3][0] * t3 + A[3][1] * t2 + A[3][2] * t + A[3][3];
  return (coefs0 * val0 + coefs1 * val1 + coefs2 * val2 + coefs3 * val3);
}


template<typename T>
__device__ inline void eval_1d_spline_vgl(T dist, T rmax, T drInv, T A[12][4], T coefs[], T& u, T& du, T& d2u)
{
  if (dist >= rmax)
  {
    u = du = d2u = (T)0.0;
    return;
  }
  T s       = dist * drInv;
  T sf      = floorf(s);
  int index = (int)sf;
  T t       = s - sf;
  T t2      = t * t;
  T t3      = t * t2;
  u         = (coefs[index + 0] * (A[0][0] * t3 + A[0][1] * t2 + A[0][2] * t + A[0][3]) +
       coefs[index + 1] * (A[1][0] * t3 + A[1][1] * t2 + A[1][2] * t + A[1][3]) +
       coefs[index + 2] * (A[2][0] * t3 + A[2][1] * t2 + A[2][2] * t + A[2][3]) +
       coefs[index + 3] * (A[3][0] * t3 + A[3][1] * t2 + A[3][2] * t + A[3][3]));
  du        = drInv *
      (coefs[index + 0] * (A[4][0] * t3 + A[4][1] * t2 + A[4][2] * t + A[4][3]) +
       coefs[index + 1] * (A[5][0] * t3 + A[5][1] * t2 + A[5][2] * t + A[5][3]) +
       coefs[index + 2] * (A[6][0] * t3 + A[6][1] * t2 + A[6][2] * t + A[6][3]) +
       coefs[index + 3] * (A[7][0] * t3 + A[7][1] * t2 + A[7][2] * t + A[7][3]));
  d2u = drInv * drInv *
      (coefs[index + 0] * (A[8][0] * t3 + A[8][1] * t2 + A[8][2] * t + A[8][3]) +
       coefs[index + 1] * (A[9][0] * t3 + A[9][1] * t2 + A[9][2] * t + A[9][3]) +
       coefs[index + 2] * (A[10][0] * t3 + A[10][1] * t2 + A[10][2] * t + A[10][3]) +
       coefs[index + 3] * (A[11][0] * t3 + A[11][1] * t2 + A[11][2] * t + A[11][3]));
}


#define MAX_COEFS 32
template<typename T, int BS>
__global__ void two_body_sum_kernel(T** R,
                                    int e1_first,
                                    int e1_last,
                                    int e2_first,
                                    int e2_last,
                                    T* spline_coefs,
                                    int numCoefs,
                                    T rMax,
                                    T* sum)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T* myR;
  if (tid == 0)
    myR = R[blockIdx.x];
  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T A[4][4];
  if (tid < 16)
    A[tid >> 2][tid & 3] = AcudaSpline[tid];
  __syncthreads();
  int N1  = e1_last - e1_first + 1;
  int N2  = e2_last - e2_first + 1;
  int NB1 = N1 / BS + ((N1 % BS) ? 1 : 0);
  int NB2 = N2 / BS + ((N2 % BS) ? 1 : 0);
  T mysum = (T)0.0;
  for (int b1 = 0; b1 < NB1; b1++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
      if ((3 * b1 + i) * BS + tid < 3 * N1)
        r1[0][i * BS + tid] = myR[3 * e1_first + (3 * b1 + i) * BS + tid];
    __syncthreads();
    int ptcl1 = e1_first + b1 * BS + tid;
    for (int b2 = 0; b2 < NB2; b2++)
    {
      // Load block of positions from global memory
      for (int i = 0; i < 3; i++)
        if ((3 * b2 + i) * BS + tid < 3 * N2)
          r2[0][i * BS + tid] = myR[3 * e2_first + (3 * b2 + i) * BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2 + 1) * BS < N2 ? BS : N2 - b2 * BS;
      for (int j = 0; j < end; j++)
      {
        int ptcl2 = e2_first + b2 * BS + j;
        T dx, dy, dz;
        dx  = r2[j][0] - r1[tid][0];
        dy  = r2[j][1] - r1[tid][1];
        dz  = r2[j][2] - r1[tid][2];
        T d = dist(dx, dy, dz);
        if (ptcl1 != ptcl2 && (ptcl1 < (N1 + e1_first)) && (ptcl2 < (N2 + e2_first)))
          mysum += eval_1d_spline(d, rMax, drInv, A, coefs);
      }
      __syncthreads();
    }
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  T factor = (e1_first == e2_first) ? 0.5 : 1.0;
  if (tid == 0)
    sum[blockIdx.x] += factor * shared_sum[0];
}

void two_body_sum(float* R[],
                  int e1_first,
                  int e1_last,
                  int e2_first,
                  int e2_last,
                  float spline_coefs[],
                  int numCoefs,
                  float rMax,
                  float sum[],
                  int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 128;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_sum_kernel<float, BS>
      <<<dimGrid, dimBlock>>>(R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, rMax, sum);
}


void two_body_sum(double* R[],
                  int e1_first,
                  int e1_last,
                  int e2_first,
                  int e2_last,
                  double spline_coefs[],
                  int numCoefs,
                  double rMax,
                  double sum[],
                  int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 128;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_sum_kernel<double, BS>
      <<<dimGrid, dimBlock>>>(R, e1_first, e1_last, e2_first, e2_last, spline_coefs, numCoefs, rMax, sum);
}


template<typename T, int BS>
__global__ void two_body_ratio_kernel(T** R,
                                      int first,
                                      int last,
                                      T* Rnew,
                                      int inew,
                                      int offset,
                                      T* spline_coefs,
                                      int numCoefs,
                                      T rMax,
                                      T* sum)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T* myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3)
  {
    myRnew[tid] = Rnew[3 * (blockIdx.x + offset) + tid];
    myRold[tid] = myR[3 * inew + tid];
  }
  __syncthreads();
  __shared__ T coefs[MAX_COEFS];
  __shared__ T r1[BS][3];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T A[4][4];
  if (tid < 16)
    A[(tid >> 2)][tid & 3] = AcudaSpline[tid];
  __syncthreads();
  int N  = last - first + 1;
  int NB = N / BS + ((N % BS) ? 1 : 0);
  __shared__ T shared_sum[BS];
  shared_sum[tid] = (T)0.0;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
    {
      int n = i * BS + tid;
      if ((3 * b + i) * BS + tid < 3 * N)
        r1[0][n] = myR[3 * first + (3 * b + i) * BS + tid];
    }
    __syncthreads();
    int ptcl1 = first + b * BS + tid;
    T dx, dy, dz;
    dx      = myRnew[0] - r1[tid][0];
    dy      = myRnew[1] - r1[tid][1];
    dz      = myRnew[2] - r1[tid][2];
    T d     = dist(dx, dy, dz);
    T delta = eval_1d_spline(d, rMax, drInv, A, coefs);
    dx      = myRold[0] - r1[tid][0];
    dy      = myRold[1] - r1[tid][1];
    dz      = myRold[2] - r1[tid][2];
    d       = dist(dx, dy, dz);
    delta -= eval_1d_spline(d, rMax, drInv, A, coefs);
    if (ptcl1 != inew && (ptcl1 < (N + first)))
      shared_sum[tid] += delta;
    __syncthreads();
  }
  __syncthreads();
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] += shared_sum[0];
}


void two_body_ratio(float* R[],
                    int first,
                    int last,
                    float Rnew[],
                    int inew,
                    int offset,
                    float spline_coefs[],
                    int numCoefs,
                    float rMax,
                    float sum[],
                    int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 128;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_ratio_kernel<float, BS>
      <<<dimGrid, dimBlock>>>(R, first, last, Rnew, inew, offset, spline_coefs, numCoefs, rMax, sum);
}


void two_body_ratio(double* R[],
                    int first,
                    int last,
                    double Rnew[],
                    int inew,
                    int offset,
                    double spline_coefs[],
                    int numCoefs,
                    double rMax,
                    double sum[],
                    int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  dim3 dimBlock(128);
  dim3 dimGrid(numWalkers);
  two_body_ratio_kernel<double, 128>
      <<<dimGrid, dimBlock>>>(R, first, last, Rnew, inew, offset, spline_coefs, numCoefs, rMax, sum);
}


template<typename T, int BS>
__global__ void two_body_ratio_grad_kernel(T** R,
                                           int first,
                                           int last,
                                           T* Rnew,
                                           int inew,
                                           int offset,
                                           T* spline_coefs,
                                           int numCoefs,
                                           T rMax,
                                           bool zero,
                                           T* ratio_grad)
{
  int tid = threadIdx.x;
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  __shared__ T* myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3)
  {
    myRnew[tid] = Rnew[3 * (blockIdx.x + offset) + tid];
    myRold[tid] = myR[3 * inew + tid];
  }
  __syncthreads();
  __shared__ T coefs[MAX_COEFS];
  __shared__ T r1[BS][3];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __syncthreads();
  __shared__ T A[12][4];
  if (tid < 16)
  {
    A[0 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 0];
    A[4 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 16];
    A[8 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 32];
  }
  __syncthreads();
  int N  = last - first + 1;
  int NB = N / BS + ((N % BS) ? 1 : 0);
  __shared__ T shared_sum[BS];
  __shared__ T shared_grad[BS][3];
  shared_sum[tid]     = (T)0.0;
  shared_grad[tid][0] = shared_grad[tid][1] = shared_grad[tid][2] = 0.0f;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
    {
      int n = i * BS + tid;
      if ((3 * b + i) * BS + tid < 3 * N)
        r1[0][n] = myR[3 * first + (3 * b + i) * BS + tid];
    }
    __syncthreads();
    int ptcl1 = first + b * BS + tid;
    T dx, dy, dz, u, du, d2u, delta, d;
    dx    = myRold[0] - r1[tid][0];
    dy    = myRold[1] - r1[tid][1];
    dz    = myRold[2] - r1[tid][2];
    d     = dist(dx, dy, dz);
    delta = -eval_1d_spline(d, rMax, drInv, A, coefs);
    dx    = myRnew[0] - r1[tid][0];
    dy    = myRnew[1] - r1[tid][1];
    dz    = myRnew[2] - r1[tid][2];
    d     = dist(dx, dy, dz);
    eval_1d_spline_vgl(d, rMax, drInv, A, coefs, u, du, d2u);
    delta += u;
    if (ptcl1 != inew && (ptcl1 < (N + first)))
    {
      du /= d;
      shared_sum[tid]     += delta;
      shared_grad[tid][0] += du * dx;
      shared_grad[tid][1] += du * dy;
      shared_grad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  __syncthreads();
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
    {
      shared_sum[tid]     += shared_sum[tid + s];
      shared_grad[tid][0] += shared_grad[tid + s][0];
      shared_grad[tid][1] += shared_grad[tid + s][1];
      shared_grad[tid][2] += shared_grad[tid + s][2];
    }
    __syncthreads();
  }
  if (tid == 0)
  {
    if (zero)
    {
      ratio_grad[4 * blockIdx.x + 0] = shared_sum[0];
      ratio_grad[4 * blockIdx.x + 1] = shared_grad[0][0];
      ratio_grad[4 * blockIdx.x + 2] = shared_grad[0][1];
      ratio_grad[4 * blockIdx.x + 3] = shared_grad[0][2];
    }
    else
    {
      ratio_grad[4 * blockIdx.x + 0] += shared_sum[0];
      ratio_grad[4 * blockIdx.x + 1] += shared_grad[0][0];
      ratio_grad[4 * blockIdx.x + 2] += shared_grad[0][1];
      ratio_grad[4 * blockIdx.x + 3] += shared_grad[0][2];
    }
  }
}


void two_body_ratio_grad(float* R[],
                         int first,
                         int last,
                         float Rnew[],
                         int inew,
                         int offset,
                         float spline_coefs[],
                         int numCoefs,
                         float rMax,
                         bool zero,
                         float ratio_grad[],
                         int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_ratio_grad_kernel<float, BS>
      <<<dimGrid, dimBlock>>>(R, first, last, Rnew, inew, offset, spline_coefs, numCoefs, rMax, zero, ratio_grad);
}


void two_body_ratio_grad(double* R[],
                         int first,
                         int last,
                         double Rnew[],
                         int inew,
                         int offset,
                         double spline_coefs[],
                         int numCoefs,
                         double rMax,
                         bool zero,
                         double ratio_grad[],
                         int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_ratio_grad_kernel<double, BS>
      <<<dimGrid, dimBlock>>>(R, first, last, Rnew, inew, offset, spline_coefs, numCoefs, rMax, zero, ratio_grad);
}


template<int BS>
__global__ void two_body_NLratio_kernel(NLjobGPU<float>* jobs,
                                        int first,
                                        int last,
                                        float** spline_coefs,
                                        int* numCoefs,
                                        float* rMaxList)
{
  const int MAX_RATIOS = 18;
  int tid              = threadIdx.x;
  __shared__ NLjobGPU<float> myJob;
  __shared__ float myRnew[MAX_RATIOS][3], myRold[3];
  __shared__ float* myCoefs;
  __shared__ int myNumCoefs;
  __shared__ float rMax;
  if (tid == 0)
  {
    myJob      = jobs[blockIdx.x];
    myCoefs    = spline_coefs[blockIdx.x];
    myNumCoefs = numCoefs[blockIdx.x];
    rMax       = rMaxList[blockIdx.x];
  }
  __syncthreads();
  if (tid < 3)
    myRold[tid] = myJob.R[3 * myJob.Elec + tid];
  for (int i = 0; i < 3; i++)
    if (i * BS + tid < 3 * myJob.NumQuadPoints)
      myRnew[0][i * BS + tid] = myJob.QuadPoints[i * BS + tid];
  float dr    = rMax / (float)(myNumCoefs - 3);
  float drInv = 1.0 / dr;
  __shared__ float coefs[MAX_COEFS];
  //  __shared__ float r1[BS][3];
  if (tid < myNumCoefs)
    coefs[tid] = myCoefs[tid];
  __shared__ float A[4][4];
  if (tid < 16)
    A[(tid >> 2)][tid & 3] = AcudaSpline[tid];
  __syncthreads();
  int N  = last - first + 1;
  int NB = N / BS + ((N % BS) ? 1 : 0);
  __shared__ float shared_sum[MAX_RATIOS][BS + 1];
  for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (float)0.0;
  for (int b = 0; b < NB * BS; b += BS)
  {
    float3 r1_r = ((float3*)(myJob.R))[first + b + tid];
    int ptcl1   = first + b + tid;
    if (ptcl1 != myJob.Elec && (ptcl1 < (N + first)))
    {
      float dx, dy, dz;
      dx         = myRold[0] - r1_r.x;
      dy         = myRold[1] - r1_r.y;
      dz         = myRold[2] - r1_r.z;
      float d    = dist(dx, dy, dz);
      float uOld = eval_1d_spline(d, rMax, drInv, A, coefs);
      for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
      {
        dx = myRnew[iq][0] - r1_r.x;
        dy = myRnew[iq][1] - r1_r.y;
        dz = myRnew[iq][2] - r1_r.z;
        d  = dist(dx, dy, dz);
        shared_sum[iq][tid] += eval_1d_spline(d, rMax, drInv, A, coefs) - uOld;
      }
    }
  }
  __syncthreads();
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
        shared_sum[iq][tid] += shared_sum[iq][tid + s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= expf(-shared_sum[tid][0]); // note this is single-precision!
}


template<int BS>
__global__ void two_body_NLratio_kernel(NLjobGPU<double>* jobs,
                                        int first,
                                        int last,
                                        double** spline_coefs,
                                        int* numCoefs,
                                        double* rMaxList)
{
  const int MAX_RATIOS = 18;
  int tid              = threadIdx.x;
  __shared__ NLjobGPU<double> myJob;
  __shared__ double myRnew[MAX_RATIOS][3], myRold[3];
  __shared__ double* myCoefs;
  __shared__ int myNumCoefs;
  __shared__ double rMax;
  if (tid == 0)
  {
    myJob      = jobs[blockIdx.x];
    myCoefs    = spline_coefs[blockIdx.x];
    myNumCoefs = numCoefs[blockIdx.x];
    rMax       = rMaxList[blockIdx.x];
  }
  __syncthreads();
  if (tid < 3)
    myRold[tid] = myJob.R[3 * myJob.Elec + tid];
  for (int i = 0; i < 3; i++)
    if (i * BS + tid < 3 * myJob.NumQuadPoints)
      myRnew[0][i * BS + tid] = myJob.QuadPoints[i * BS + tid];
  __syncthreads();
  double dr    = rMax / (double)(myNumCoefs - 3);
  double drInv = 1.0 / dr;
  __shared__ double coefs[MAX_COEFS];
  __shared__ double r1[BS][3];
  if (tid < myNumCoefs)
    coefs[tid] = myCoefs[tid];
  __syncthreads();
  __shared__ double A[4][4];
  if (tid < 16)
    A[(tid >> 2)][tid & 3] = AcudaSpline[tid];
  __syncthreads();
  int N  = last - first + 1;
  int NB = N / BS + ((N % BS) ? 1 : 0);
  __shared__ double shared_sum[MAX_RATIOS][BS + 1];
  for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (double)0.0;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
    {
      int n = i * BS + tid;
      if ((3 * b + i) * BS + tid < 3 * N)
        r1[0][n] = myJob.R[3 * first + (3 * b + i) * BS + tid];
    }
    __syncthreads();
    int ptcl1 = first + b * BS + tid;
    double dx, dy, dz;
    dx          = myRold[0] - r1[tid][0];
    dy          = myRold[1] - r1[tid][1];
    dz          = myRold[2] - r1[tid][2];
    double d    = dist(dx, dy, dz);
    double uOld = eval_1d_spline(d, rMax, drInv, A, coefs);
    for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
    {
      dx = myRnew[iq][0] - r1[tid][0];
      dy = myRnew[iq][1] - r1[tid][1];
      dz = myRnew[iq][2] - r1[tid][2];
      d  = dist(dx, dy, dz);
      if (ptcl1 != myJob.Elec && (ptcl1 < (N + first)))
        shared_sum[iq][tid] += eval_1d_spline(d, rMax, drInv, A, coefs) - uOld;
    }
    __syncthreads();
  }
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
        shared_sum[iq][tid] += shared_sum[iq][tid + s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= exp(-shared_sum[tid][0]);
}


void two_body_NLratios(NLjobGPU<float> jobs[],
                       int first,
                       int last,
                       float* spline_coefs[],
                       int numCoefs[],
                       float rMax[],
                       int numjobs)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  while (numjobs > 65535)
  {
    dim3 dimGrid(65535);
    two_body_NLratio_kernel<BS><<<dimGrid, dimBlock>>>(jobs, first, last, spline_coefs, numCoefs, rMax);
    jobs += 65535;
    numjobs -= 65535;
  }
  dim3 dimGrid(numjobs);
  two_body_NLratio_kernel<BS><<<dimGrid, dimBlock>>>(jobs, first, last, spline_coefs, numCoefs, rMax);
}


void two_body_NLratios(NLjobGPU<double> jobs[],
                       int first,
                       int last,
                       double* spline_coefs[],
                       int numCoefs[],
                       double rMax[],
                       int numjobs)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  while (numjobs > 65535)
  {
    dim3 dimGrid(65535);
    two_body_NLratio_kernel<BS><<<dimGrid, dimBlock>>>(jobs, first, last, spline_coefs, numCoefs, rMax);
    jobs += 65535;
    numjobs -= 65535;
  }
  dim3 dimGrid(numjobs);
  two_body_NLratio_kernel<BS><<<dimGrid, dimBlock>>>(jobs, first, last, spline_coefs, numCoefs, rMax);
}


#define MAX_COEFS 32

template<typename T, int BS>
__global__ void two_body_grad_lapl_kernel(T** R,
                                          int e1_first,
                                          int e1_last,
                                          int e2_first,
                                          int e2_last,
                                          T* spline_coefs,
                                          int numCoefs,
                                          T rMax,
                                          T* gradLapl,
                                          int row_stride)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T* myR;
  if (tid == 0)
    myR = R[blockIdx.x];
  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T A[12][4];
  if (tid < 16)
  {
    A[0 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 0];
    A[4 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 16];
    A[8 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 32];
  }
  __syncthreads();
  int N1  = e1_last - e1_first + 1;
  int N2  = e2_last - e2_first + 1;
  int NB1 = N1 / BS + ((N1 % BS) ? 1 : 0);
  int NB2 = N2 / BS + ((N2 % BS) ? 1 : 0);
  __shared__ T sGradLapl[BS][4];
  for (int b1 = 0; b1 < NB1; b1++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
      if ((3 * b1 + i) * BS + tid < 3 * N1)
        r1[0][i * BS + tid] = myR[3 * e1_first + (3 * b1 + i) * BS + tid];
    __syncthreads();
    int ptcl1         = e1_first + b1 * BS + tid;
    int offset        = blockIdx.x * row_stride + 4 * b1 * BS + 4 * e1_first;
    sGradLapl[tid][0] = sGradLapl[tid][1] = sGradLapl[tid][2] = sGradLapl[tid][3] = (T)0.0;
    for (int b2 = 0; b2 < NB2; b2++)
    {
      // Load block of positions from global memory
      for (int i = 0; i < 3; i++)
        if ((3 * b2 + i) * BS + tid < 3 * N2)
          r2[0][i * BS + tid] = myR[3 * e2_first + (3 * b2 + i) * BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2 + 1) * BS < N2 ? BS : N2 - b2 * BS;
      for (int j = 0; j < end; j++)
      {
        int ptcl2 = e2_first + b2 * BS + j;
        T dx, dy, dz, u, du, d2u;
        dx  = r2[j][0] - r1[tid][0];
        dy  = r2[j][1] - r1[tid][1];
        dz  = r2[j][2] - r1[tid][2];
        T d = dist(dx, dy, dz);
        eval_1d_spline_vgl(d, rMax, drInv, A, coefs, u, du, d2u);
        if (ptcl1 != ptcl2 && (ptcl1 < (N1 + e1_first)) && (ptcl2 < (N2 + e2_first)))
        {
          du /= d;
          sGradLapl[tid][0] += du * dx;
          sGradLapl[tid][1] += du * dy;
          sGradLapl[tid][2] += du * dz;
          sGradLapl[tid][3] -= d2u + 2.0 * du;
        }
      }
      __syncthreads();
    }
    for (int i = 0; i < 4; i++)
      if ((4 * b1 + i) * BS + tid < 4 * N1)
        gradLapl[offset + i * BS + tid] += sGradLapl[0][i * BS + tid];
    __syncthreads();
  }
}


void two_body_grad_lapl(float* R[],
                        int e1_first,
                        int e1_last,
                        int e2_first,
                        int e2_last,
                        float spline_coefs[],
                        int numCoefs,
                        float rMax,
                        float gradLapl[],
                        int row_stride,
                        int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_grad_lapl_kernel<float, BS><<<dimGrid, dimBlock>>>(R,
                                                              e1_first,
                                                              e1_last,
                                                              e2_first,
                                                              e2_last,
                                                              spline_coefs,
                                                              numCoefs,
                                                              rMax,
                                                              gradLapl,
                                                              row_stride);
}


void two_body_grad_lapl(double* R[],
                        int e1_first,
                        int e1_last,
                        int e2_first,
                        int e2_last,
                        double spline_coefs[],
                        int numCoefs,
                        double rMax,
                        double gradLapl[],
                        int row_stride,
                        int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_grad_lapl_kernel<double, BS><<<dimGrid, dimBlock>>>(R,
                                                               e1_first,
                                                               e1_last,
                                                               e2_first,
                                                               e2_last,
                                                               spline_coefs,
                                                               numCoefs,
                                                               rMax,
                                                               gradLapl,
                                                               row_stride);
}


template<typename T, int BS>
__global__ void two_body_grad_kernel(T** R,
                                     int first,
                                     int last,
                                     int iat,
                                     T* spline_coefs,
                                     int numCoefs,
                                     T rMax,
                                     bool zeroOut,
                                     T* grad)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T *myR, r2[3];
  if (tid == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3)
    r2[tid] = myR[3 * iat + tid];
  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3];
  __shared__ T A[12][4];
  if (tid < 16)
  {
    A[0 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 0];
    A[4 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 16];
    A[8 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 32];
  }
  __syncthreads();
  int N  = last - first + 1;
  int NB = N / BS + ((N % BS) ? 1 : 0);
  __shared__ T sGrad[BS][3];
  sGrad[tid][0] = sGrad[tid][1] = sGrad[tid][2] = (T)0.0;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
      if ((3 * b + i) * BS + tid < 3 * N)
        r1[0][i * BS + tid] = myR[3 * first + (3 * b + i) * BS + tid];
    __syncthreads();
    int ptcl1 = first + b * BS + tid;
    T dx, dy, dz, u, du, d2u;
    dx  = r2[0] - r1[tid][0];
    dy  = r2[1] - r1[tid][1];
    dz  = r2[2] - r1[tid][2];
    T d = dist(dx, dy, dz);
    eval_1d_spline_vgl(d, rMax, drInv, A, coefs, u, du, d2u);
    if (ptcl1 != iat && ptcl1 < (N + first))
    {
      du /= d;
      sGrad[tid][0] += du * dx;
      sGrad[tid][1] += du * dy;
      sGrad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  // Do reduction across threads in block
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      sGrad[tid][0] += sGrad[tid + s][0];
      sGrad[tid][1] += sGrad[tid + s][1];
      sGrad[tid][2] += sGrad[tid + s][2];
    }
    __syncthreads();
  }
  if (tid < 3)
  {
    if (zeroOut)
      grad[3 * blockIdx.x + tid] = sGrad[0][tid];
    else
      grad[3 * blockIdx.x + tid] += sGrad[0][tid];
  }
}


void two_body_gradient(float* R[],
                       int first,
                       int last,
                       int iat,
                       float spline_coefs[],
                       int numCoefs,
                       float rMax,
                       bool zeroOut,
                       float grad[],
                       int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_grad_kernel<float, BS>
      <<<dimGrid, dimBlock, 0, gpu::kernelStream>>>(R, first, last, iat, spline_coefs, numCoefs, rMax, zeroOut, grad);
}


void two_body_gradient(double* R[],
                       int first,
                       int last,
                       int iat,
                       double spline_coefs[],
                       int numCoefs,
                       double rMax,
                       bool zeroOut,
                       double grad[],
                       int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_grad_kernel<double, BS>
      <<<dimGrid, dimBlock, 0, gpu::kernelStream>>>(R, first, last, iat, spline_coefs, numCoefs, rMax, zeroOut, grad);
}


template<typename T, int BS, unsigned COMPLEX>
__global__ void two_body_derivs_kernel(T** R,
                                       T** gradLogPsi,
                                       int e1_first,
                                       int e1_last,
                                       int e2_first,
                                       int e2_last,
                                       int numCoefs,
                                       T rMax,
                                       T** derivs)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0f / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T *myR, *myGrad, *myDerivs;
  if (tid == 0)
  {
    myR      = R[blockIdx.x];
    myGrad   = gradLogPsi[blockIdx.x];
    myDerivs = derivs[blockIdx.x];
  }
  __shared__ T sderivs[MAX_COEFS][2];
  // __shared__ T coefs[MAX_COEFS];
  // if (tid < numCoefs)
  //   coefs[tid] = spline_coefs[tid];
  __shared__ T r1[BS][3], r2[BS][3];
  __shared__ T A[12][4];
  if (tid < 16)
  {
    A[0 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 0];
    A[4 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 16];
    A[8 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 32];
  }
  __syncthreads();
  sderivs[tid][0] = T();
  sderivs[tid][1] = T();
  int N1          = e1_last - e1_first + 1;
  int N2          = e2_last - e2_first + 1;
  int NB1         = N1 / BS + ((N1 % BS) ? 1 : 0);
  int NB2         = N2 / BS + ((N2 % BS) ? 1 : 0);
  __shared__ T sGrad[BS][3];
  for (int b1 = 0; b1 < NB1; b1++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
      if ((3 * b1 + i) * BS + tid < 3 * N1)
      {
        int outoff       = i * BS + tid;
        int inoff        = outoff + 3 * e1_first + 3 * b1 * BS;
        r1[0][outoff]    = myR[inoff]; //[3*e1_first + (3*b1+i)*BS + tid];
        sGrad[0][outoff] = myGrad[inoff * COMPLEX];
      }
    __syncthreads();
    int ptcl1 = e1_first + b1 * BS + tid;
    for (int b2 = 0; b2 < NB2; b2++)
    {
      // Load block of positions from global memory
      for (int i = 0; i < 3; i++)
        if ((3 * b2 + i) * BS + tid < 3 * N2)
          r2[0][i * BS + tid] = myR[3 * e2_first + (3 * b2 + i) * BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (b2 + 1) * BS < N2 ? BS : N2 - b2 * BS;
      for (int j = 0; j < end; j++)
      {
        int ptcl2 = e2_first + b2 * BS + j;
        T dx, dy, dz;
        dx        = r2[j][0] - r1[tid][0];
        dy        = r2[j][1] - r1[tid][1];
        dz        = r2[j][2] - r1[tid][2];
        T d       = dist(dx, dy, dz);
        T dInv    = 1.0f / d;
        T s       = d * drInv;
        T sf      = floorf(s);
        int index = (int)sf;
        T t       = s - sf;
        T t2      = t * t;
        T t3      = t * t2;
        T v0, v1, v2, v3;
        // sderivs[index+0][0] += (A[0][0]*t3 + A[0][1]*t2 + A[0][2]*t + A[0][3]);
        // sderivs[index+1][0] += (A[1][0]*t3 + A[1][1]*t2 + A[1][2]*t + A[1][3]);
        // sderivs[index+2][0] += (A[2][0]*t3 + A[2][1]*t2 + A[2][2]*t + A[2][3]);
        // sderivs[index+3][0] += (A[3][0]*t3 + A[3][1]*t2 + A[3][2]*t + A[3][3]);
        v0 = (A[0][0] * t3 + A[0][1] * t2 + A[0][2] * t + A[0][3]);
        v1 = (A[1][0] * t3 + A[1][1] * t2 + A[1][2] * t + A[1][3]);
        v2 = (A[2][0] * t3 + A[2][1] * t2 + A[2][2] * t + A[2][3]);
        v3 = (A[3][0] * t3 + A[3][1] * t2 + A[3][2] * t + A[3][3]);
        for (int id = 0; id < BS; id++)
          if (tid == id && ptcl1 != ptcl2 && ptcl1 <= e1_last && (d < rMax))
          {
            sderivs[index + 0][0] += v0;
            sderivs[index + 1][0] += v1;
            sderivs[index + 2][0] += v2;
            sderivs[index + 3][0] += v3;
          }
        T prefact = (dx * sGrad[tid][0] + dy * sGrad[tid][1] + dz * sGrad[tid][2]) * dInv;
        T du0     = drInv * (A[4][0] * t3 + A[4][1] * t2 + A[4][2] * t + A[4][3]);
        T du1     = drInv * (A[5][0] * t3 + A[5][1] * t2 + A[5][2] * t + A[5][3]);
        T du2     = drInv * (A[6][0] * t3 + A[6][1] * t2 + A[6][2] * t + A[6][3]);
        T du3     = drInv * (A[7][0] * t3 + A[7][1] * t2 + A[7][2] * t + A[7][3]);
        // This is the dot (gradu, grad_log_psi) term.
        v0 = 2.0f * prefact * du0;
        v1 = 2.0f * prefact * du1;
        v2 = 2.0f * prefact * du2;
        v3 = 2.0f * prefact * du3;
        // This is the lapl u term
        v0 -= drInv * drInv * (A[8][0] * t3 + A[8][1] * t2 + A[8][2] * t + A[8][3]) + 2.0f * du0 * dInv;
        v1 -= drInv * drInv * (A[9][0] * t3 + A[9][1] * t2 + A[9][2] * t + A[9][3]) + 2.0f * du1 * dInv;
        v2 -= drInv * drInv * (A[10][0] * t3 + A[10][1] * t2 + A[10][2] * t + A[10][3]) + 2.0f * du2 * dInv;
        v3 -= drInv * drInv * (A[11][0] * t3 + A[11][1] * t2 + A[11][2] * t + A[11][3]) + 2.0f * du3 * dInv;
        for (int id = 0; id < BS; id++)
          if (tid == id && ptcl1 != ptcl2 && ptcl1 <= e1_last && (d < rMax))
          {
            sderivs[index + 0][1] += v0;
            sderivs[index + 1][1] += v1;
            sderivs[index + 2][1] += v2;
            sderivs[index + 3][1] += v3;
          }
      }
      __syncthreads();
    }
  }
  //  if (e1_first == e2_first)
  sderivs[tid][0] *= 0.5f;
  sderivs[tid][1] *= 0.5f;
  if (tid < 2 * numCoefs)
    myDerivs[tid] = -sderivs[0][tid];
  if (tid + BS < 2 * numCoefs)
    myDerivs[tid + BS] = sderivs[0][tid + BS];
}


void two_body_derivs(float* R[],
                     float* gradLogPsi[],
                     int e1_first,
                     int e1_last,
                     int e2_first,
                     int e2_last,
                     int numCoefs,
                     float rMax,
                     float* derivs[],
                     int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_derivs_kernel<float, BS, 1>
      <<<dimGrid, dimBlock>>>(R, gradLogPsi, e1_first, e1_last, e2_first, e2_last, numCoefs, rMax, derivs);
}

void two_body_derivs(double* R[],
                     double* gradLogPsi[],
                     int e1_first,
                     int e1_last,
                     int e2_first,
                     int e2_last,
                     int numCoefs,
                     double rMax,
                     double* derivs[],
                     int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  two_body_derivs_kernel<double, BS, 1>
      <<<dimGrid, dimBlock>>>(R, gradLogPsi, e1_first, e1_last, e2_first, e2_last, numCoefs, rMax, derivs);
}


// Ye: use offset to recycle the old routines
// block size can be further optimized.
#ifdef QMC_COMPLEX
void two_body_derivs(float* R[],
                     std::complex<float>* gradLogPsi[],
                     int e1_first,
                     int e1_last,
                     int e2_first,
                     int e2_last,
                     int numCoefs,
                     float rMax,
                     float* derivs[],
                     int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_derivs_kernel<float, BS, 2>
      <<<dimGrid, dimBlock>>>(R, (float**)gradLogPsi, e1_first, e1_last, e2_first, e2_last, numCoefs, rMax, derivs);
}

void two_body_derivs(double* R[],
                     std::complex<double>* gradLogPsi[],
                     int e1_first,
                     int e1_last,
                     int e2_first,
                     int e2_last,
                     int numCoefs,
                     double rMax,
                     double* derivs[],
                     int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  two_body_derivs_kernel<double, BS, 2>
      <<<dimGrid, dimBlock>>>(R, (double**)gradLogPsi, e1_first, e1_last, e2_first, e2_last, numCoefs, rMax, derivs);
}
#endif


////////////////////////////////////////////////////////////////
//                      One-body routines                     //
////////////////////////////////////////////////////////////////

template<typename T, int BS>
__global__ void one_body_sum_kernel(T* C,
                                    T** R,
                                    int cfirst,
                                    int clast,
                                    int efirst,
                                    int elast,
                                    T* spline_coefs,
                                    int numCoefs,
                                    T rMax,
                                    T* sum)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T* myR;
  if (tid == 0)
    myR = R[blockIdx.x];
  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T rc[BS][3], re[BS][3];
  __shared__ T A[4][4];
  if (tid < 16)
    A[tid >> 2][tid & 3] = AcudaSpline[tid];
  __syncthreads();
  int Nc  = clast - cfirst + 1;
  int Ne  = elast - efirst + 1;
  int NBc = Nc / BS + ((Nc % BS) ? 1 : 0);
  int NBe = Ne / BS + ((Ne % BS) ? 1 : 0);
  T mysum = (T)0.0;
  for (int bc = 0; bc < NBc; bc++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
      if ((3 * bc + i) * BS + tid < 3 * Nc)
        rc[0][i * BS + tid] = C[3 * cfirst + (3 * bc + i) * BS + tid];
    __syncthreads();
    int ptcl1 = cfirst + bc * BS + tid;
    for (int be = 0; be < NBe; be++)
    {
      // Load block of positions from global memory
      for (int i = 0; i < 3; i++)
        if ((3 * be + i) * BS + tid < 3 * Ne)
          re[0][i * BS + tid] = myR[3 * efirst + (3 * be + i) * BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = (be + 1) * BS < Ne ? BS : Ne - be * BS;
      for (int j = 0; j < end; j++)
      {
        int ptcl2 = efirst + be * BS + j;
        T dx, dy, dz;
        dx  = re[j][0] - rc[tid][0];
        dy  = re[j][1] - rc[tid][1];
        dz  = re[j][2] - rc[tid][2];
        T d = dist(dx, dy, dz);
        if ((ptcl1 < (Nc + cfirst)) && (ptcl2 < (Ne + efirst)))
          mysum += eval_1d_spline(d, rMax, drInv, A, coefs);
      }
    }
    __syncthreads();
  }
  __shared__ T shared_sum[BS];
  shared_sum[tid] = mysum;
  __syncthreads();
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] += shared_sum[0];
}

void one_body_sum(float C[],
                  float* R[],
                  int cfirst,
                  int clast,
                  int efirst,
                  int elast,
                  float spline_coefs[],
                  int numCoefs,
                  float rMax,
                  float sum[],
                  int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_sum_kernel<float, BS>
      <<<dimGrid, dimBlock>>>(C, R, cfirst, clast, efirst, elast, spline_coefs, numCoefs, rMax, sum);
}


void one_body_sum(double C[],
                  double* R[],
                  int cfirst,
                  int clast,
                  int efirst,
                  int elast,
                  double spline_coefs[],
                  int numCoefs,
                  double rMax,
                  double sum[],
                  int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 128;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_sum_kernel<double, BS>
      <<<dimGrid, dimBlock>>>(C, R, cfirst, clast, efirst, elast, spline_coefs, numCoefs, rMax, sum);
}


template<typename T, int BS>
__global__ void one_body_ratio_kernel(T* C,
                                      T** R,
                                      int cfirst,
                                      int clast,
                                      T* Rnew,
                                      int inew,
                                      T* spline_coefs,
                                      int numCoefs,
                                      int nw,
                                      T rMax,
                                      T* sum)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T* myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0)
    myR = R[blockIdx.x % nw];
  __syncthreads();
  if (tid < 3)
  {
    myRnew[tid] = Rnew[3 * blockIdx.x + tid];
    myRold[tid] = myR[3 * (inew + blockIdx.x / nw) + tid];
  }
  __syncthreads();
  __shared__ T coefs[MAX_COEFS];
  __shared__ T c[BS][3];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T A[4][4];
  if (tid < 16)
    A[(tid >> 2)][tid & 3] = AcudaSpline[tid];
  __syncthreads();
  int Nc = clast - cfirst + 1;
  int NB = Nc / BS + ((Nc % BS) ? 1 : 0);
  __shared__ T shared_sum[BS];
  shared_sum[tid] = (T)0.0;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
    {
      int n = i * BS + tid;
      if ((3 * b + i) * BS + tid < 3 * Nc)
        c[0][n] = C[3 * cfirst + (3 * b + i) * BS + tid];
    }
    __syncthreads();
    int ptcl1 = cfirst + b * BS + tid;
    T dx, dy, dz;
    dx      = myRnew[0] - c[tid][0];
    dy      = myRnew[1] - c[tid][1];
    dz      = myRnew[2] - c[tid][2];
    T d     = dist(dx, dy, dz);
    T delta = eval_1d_spline(d, rMax, drInv, A, coefs);
    dx      = myRold[0] - c[tid][0];
    dy      = myRold[1] - c[tid][1];
    dz      = myRold[2] - c[tid][2];
    d       = dist(dx, dy, dz);
    delta -= eval_1d_spline(d, rMax, drInv, A, coefs);
    if (ptcl1 < (Nc + cfirst))
      shared_sum[tid] += delta;
    __syncthreads();
  }
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      shared_sum[tid] += shared_sum[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    sum[blockIdx.x] += shared_sum[0];
}


void one_body_ratio(float C[],
                    float* R[],
                    int first,
                    int last,
                    float Rnew[],
                    int inew,
                    float spline_coefs[],
                    int numCoefs,
                    int nw,
                    float rMax,
                    float sum[],
                    int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_ratio_kernel<float, BS>
      <<<dimGrid, dimBlock>>>(C, R, first, last, Rnew, inew, spline_coefs, numCoefs, nw, rMax, sum);
}


void one_body_ratio(double C[],
                    double* R[],
                    int first,
                    int last,
                    double Rnew[],
                    int inew,
                    double spline_coefs[],
                    int numCoefs,
                    int nw,
                    double rMax,
                    double sum[],
                    int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  dim3 dimBlock(128);
  dim3 dimGrid(numWalkers);
  one_body_ratio_kernel<double, 128>
      <<<dimGrid, dimBlock>>>(C, R, first, last, Rnew, inew, spline_coefs, numCoefs, nw, rMax, sum);
}


template<typename T, int BS>
__global__ void one_body_ratio_grad_kernel(T* C,
                                           T** R,
                                           int cfirst,
                                           int clast,
                                           T* Rnew,
                                           int inew,
                                           T* spline_coefs,
                                           int numCoefs,
                                           int nw,
                                           T rMax,
                                           bool zero,
                                           T* ratio_grad)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T* myR;
  __shared__ T myRnew[3], myRold[3];
  if (tid == 0)
    myR = R[blockIdx.x % nw];
  __syncthreads();
  if (tid < 3)
  {
    myRnew[tid] = Rnew[3 * blockIdx.x + tid];
    myRold[tid] = myR[3 * (inew + blockIdx.x / nw) + tid];
  }
  __syncthreads();
  __shared__ T coefs[MAX_COEFS];
  __shared__ T c[BS][3];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T A[12][4];
  if (tid < 16)
  {
    A[0 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 0];
    A[4 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 16];
    A[8 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 32];
  }
  __syncthreads();
  int Nc = clast - cfirst + 1;
  int NB = Nc / BS + ((Nc % BS) ? 1 : 0);
  __shared__ T shared_sum[BS];
  __shared__ T shared_grad[BS][3];
  shared_sum[tid]     = (T)0.0;
  shared_grad[tid][0] = shared_grad[tid][1] = shared_grad[tid][2] = 0.0f;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
    {
      int n = i * BS + tid;
      if ((3 * b + i) * BS + tid < 3 * Nc)
        c[0][n] = C[3 * cfirst + (3 * b + i) * BS + tid];
    }
    __syncthreads();
    int ptcl1 = cfirst + b * BS + tid;
    T dx, dy, dz, d, delta, u, du, d2u;
    dx    = myRold[0] - c[tid][0];
    dy    = myRold[1] - c[tid][1];
    dz    = myRold[2] - c[tid][2];
    d     = dist(dx, dy, dz);
    delta = -eval_1d_spline(d, rMax, drInv, A, coefs);
    dx    = myRnew[0] - c[tid][0];
    dy    = myRnew[1] - c[tid][1];
    dz    = myRnew[2] - c[tid][2];
    d     = dist(dx, dy, dz);
    eval_1d_spline_vgl(d, rMax, drInv, A, coefs, u, du, d2u);
    delta += u;
    if (ptcl1 < (Nc + cfirst))
    {
      du /= d;
      shared_sum[tid]     += delta;
      shared_grad[tid][0] += du * dx;
      shared_grad[tid][1] += du * dy;
      shared_grad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
    {
      shared_sum[tid]     += shared_sum[tid + s];
      shared_grad[tid][0] += shared_grad[tid + s][0];
      shared_grad[tid][1] += shared_grad[tid + s][1];
      shared_grad[tid][2] += shared_grad[tid + s][2];
    }
    __syncthreads();
  }
  if (tid == 0)
  {
    if (zero)
    {
      ratio_grad[4 * blockIdx.x + 0] = shared_sum[0];
      ratio_grad[4 * blockIdx.x + 1] = shared_grad[0][0];
      ratio_grad[4 * blockIdx.x + 2] = shared_grad[0][1];
      ratio_grad[4 * blockIdx.x + 3] = shared_grad[0][2];
    }
    else
    {
      ratio_grad[4 * blockIdx.x + 0] += shared_sum[0];
      ratio_grad[4 * blockIdx.x + 1] += shared_grad[0][0];
      ratio_grad[4 * blockIdx.x + 2] += shared_grad[0][1];
      ratio_grad[4 * blockIdx.x + 3] += shared_grad[0][2];
    }
  }
}


void one_body_ratio_grad(float C[],
                         float* R[],
                         int first,
                         int last,
                         float Rnew[],
                         int inew,
                         float spline_coefs[],
                         int numCoefs,
                         int nw,
                         float rMax,
                         bool zero,
                         float ratio_grad[],
                         int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_ratio_grad_kernel<float, BS>
      <<<dimGrid, dimBlock>>>(C, R, first, last, Rnew, inew, spline_coefs, numCoefs, nw, rMax, zero, ratio_grad);
}


void one_body_ratio_grad(double C[],
                         double* R[],
                         int first,
                         int last,
                         double Rnew[],
                         int inew,
                         double spline_coefs[],
                         int numCoefs,
                         int nw,
                         double rMax,
                         bool zero,
                         double ratio_grad[],
                         int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_ratio_grad_kernel<double, BS>
      <<<dimGrid, dimBlock>>>(C, R, first, last, Rnew, inew, spline_coefs, numCoefs, nw, rMax, zero, ratio_grad);
}


template<typename T, int BS>
__global__ void one_body_grad_lapl_kernel(T* C,
                                          T** R,
                                          int cfirst,
                                          int clast,
                                          int efirst,
                                          int elast,
                                          T* spline_coefs,
                                          int numCoefs,
                                          T rMax,
                                          T* gradLapl,
                                          int row_stride)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  //  rMax *= 0.99999f;
  int tid = threadIdx.x;
  __shared__ T* myR;
  if (tid == 0)
    myR = R[blockIdx.x];
  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T r[BS][3], c[BS][3];
  __syncthreads();
  __shared__ T A[12][4];
  if (tid < 16)
  {
    A[0 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 0];
    A[4 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 16];
    A[8 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 32];
  }
  __syncthreads();
  int Nc  = clast - cfirst + 1;
  int Ne  = elast - efirst + 1;
  int NBc = Nc / BS + ((Nc % BS) ? 1 : 0);
  int NBe = Ne / BS + ((Ne % BS) ? 1 : 0);
  __shared__ T sGradLapl[BS][4];
  for (int be = 0; be < NBe; be++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
      if ((3 * be + i) * BS + tid < 3 * Ne)
        r[0][i * BS + tid] = myR[3 * efirst + (3 * be + i) * BS + tid];
    __syncthreads();
    int eptcl         = efirst + be * BS + tid;
    int offset        = blockIdx.x * row_stride + 4 * be * BS + 4 * efirst;
    sGradLapl[tid][0] = sGradLapl[tid][1] = sGradLapl[tid][2] = sGradLapl[tid][3] = (T)0.0;
    for (int bc = 0; bc < NBc; bc++)
    {
      // Load block of positions from global memory
      for (int i = 0; i < 3; i++)
        if ((3 * bc + i) * BS + tid < 3 * Nc)
          c[0][i * BS + tid] = C[3 * cfirst + (3 * bc + i) * BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = ((bc + 1) * BS < Nc) ? BS : Nc - bc * BS;
      for (int j = 0; j < end; j++)
      {
        int cptcl = cfirst + bc * BS + j;
        T dx, dy, dz, u, du, d2u;
        dx  = r[tid][0] - c[j][0];
        dy  = r[tid][1] - c[j][1];
        dz  = r[tid][2] - c[j][2];
        T d = dist(dx, dy, dz);
        eval_1d_spline_vgl(d, rMax, drInv, A, coefs, u, du, d2u);
        //u = du = d2u = 0.0f;
        if (cptcl < (Nc + cfirst) && (eptcl < (Ne + efirst)))
        {
          du /= d;
          sGradLapl[tid][0] -= du * dx;
          sGradLapl[tid][1] -= du * dy;
          sGradLapl[tid][2] -= du * dz;
          sGradLapl[tid][3] -= d2u + 2.0 * du;
        }
      }
      __syncthreads();
    }
    __syncthreads();
    for (int i = 0; i < 4; i++)
      if ((4 * be + i) * BS + tid < 4 * Ne)
        gradLapl[offset + i * BS + tid] += sGradLapl[0][i * BS + tid];
    __syncthreads();
  }
}


void one_body_grad_lapl(float C[],
                        float* R[],
                        int e1_first,
                        int e1_last,
                        int e2_first,
                        int e2_last,
                        float spline_coefs[],
                        int numCoefs,
                        float rMax,
                        float gradLapl[],
                        int row_stride,
                        int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_grad_lapl_kernel<float, BS><<<dimGrid, dimBlock>>>(C,
                                                              R,
                                                              e1_first,
                                                              e1_last,
                                                              e2_first,
                                                              e2_last,
                                                              spline_coefs,
                                                              numCoefs,
                                                              rMax,
                                                              gradLapl,
                                                              row_stride);
}


void one_body_grad_lapl(double C[],
                        double* R[],
                        int e1_first,
                        int e1_last,
                        int e2_first,
                        int e2_last,
                        double spline_coefs[],
                        int numCoefs,
                        double rMax,
                        double gradLapl[],
                        int row_stride,
                        int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_grad_lapl_kernel<double, BS><<<dimGrid, dimBlock>>>(C,
                                                               R,
                                                               e1_first,
                                                               e1_last,
                                                               e2_first,
                                                               e2_last,
                                                               spline_coefs,
                                                               numCoefs,
                                                               rMax,
                                                               gradLapl,
                                                               row_stride);
}


template<int BS>
__global__ void one_body_NLratio_kernel(NLjobGPU<float>* jobs,
                                        float* C,
                                        int first,
                                        int last,
                                        float* spline_coefs,
                                        int numCoefs,
                                        float rMax)
{
  const int MAX_RATIOS = 18;
  int tid              = threadIdx.x;
  __shared__ NLjobGPU<float> myJob;
  __shared__ float myRnew[MAX_RATIOS][3], myRold[3];
  if (tid == 0)
    myJob = jobs[blockIdx.x];
  __syncthreads();
  if (tid < 3)
    myRold[tid] = myJob.R[3 * myJob.Elec + tid];
  for (int i = 0; i < 3; i++)
    if (i * BS + tid < 3 * myJob.NumQuadPoints)
      myRnew[0][i * BS + tid] = myJob.QuadPoints[i * BS + tid];
  __syncthreads();
  float dr    = rMax / (float)(numCoefs - 3);
  float drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  __shared__ float coefs[MAX_COEFS];
  __shared__ float c[BS][3];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __syncthreads();
  __shared__ float A[4][4];
  if (tid < 16)
    A[(tid >> 2)][tid & 3] = AcudaSpline[tid];
  __syncthreads();
  int N  = last - first + 1;
  int NB = N / BS + ((N % BS) ? 1 : 0);
  __shared__ float shared_sum[MAX_RATIOS][BS + 1];
  for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (float)0.0;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
    {
      int n = i * BS + tid;
      if ((3 * b + i) * BS + tid < 3 * N)
        c[0][n] = C[3 * first + (3 * b + i) * BS + tid];
    }
    __syncthreads();
    int ptcl1 = first + b * BS + tid;
    float dx, dy, dz;
    dx         = myRold[0] - c[tid][0];
    dy         = myRold[1] - c[tid][1];
    dz         = myRold[2] - c[tid][2];
    float d    = dist(dx, dy, dz);
    float uOld = eval_1d_spline(d, rMax, drInv, A, coefs);
    for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
    {
      dx = myRnew[iq][0] - c[tid][0];
      dy = myRnew[iq][1] - c[tid][1];
      dz = myRnew[iq][2] - c[tid][2];
      d  = dist(dx, dy, dz);
      if (ptcl1 < (N + first))
        shared_sum[iq][tid] += eval_1d_spline(d, rMax, drInv, A, coefs) - uOld;
    }
    __syncthreads();
  }
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
        shared_sum[iq][tid] += shared_sum[iq][tid + s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= exp(-shared_sum[tid][0]);
}


template<int BS>
__global__ void one_body_NLratio_kernel(NLjobGPU<double>* jobs,
                                        double* C,
                                        int first,
                                        int last,
                                        double* spline_coefs,
                                        int numCoefs,
                                        double rMax)
{
  const int MAX_RATIOS = 18;
  int tid              = threadIdx.x;
  __shared__ NLjobGPU<double> myJob;
  __shared__ double myRnew[MAX_RATIOS][3], myRold[3];
  if (tid == 0)
    myJob = jobs[blockIdx.x];
  __syncthreads();
  if (tid < 3)
    myRold[tid] = myJob.R[3 * myJob.Elec + tid];
  for (int i = 0; i < 3; i++)
    if (i * BS + tid < 3 * myJob.NumQuadPoints)
      myRnew[0][i * BS + tid] = myJob.QuadPoints[i * BS + tid];
  __syncthreads();
  double dr    = rMax / (double)(numCoefs - 3);
  double drInv = 1.0 / dr;
  __shared__ double coefs[MAX_COEFS];
  __shared__ double c[BS][3];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __syncthreads();
  __shared__ double A[4][4];
  if (tid < 16)
    A[(tid >> 2)][tid & 3] = AcudaSpline[tid];
  __syncthreads();
  int N  = last - first + 1;
  int NB = N / BS + ((N % BS) ? 1 : 0);
  __shared__ double shared_sum[MAX_RATIOS][BS + 1];
  for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
    shared_sum[iq][tid] = (double)0.0;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
    {
      int n = i * BS + tid;
      if ((3 * b + i) * BS + tid < 3 * N)
        c[0][n] = C[3 * first + (3 * b + i) * BS + tid];
    }
    __syncthreads();
    int ptcl1 = first + b * BS + tid;
    double dx, dy, dz;
    dx          = myRold[0] - c[tid][0];
    dy          = myRold[1] - c[tid][1];
    dz          = myRold[2] - c[tid][2];
    double d    = dist(dx, dy, dz);
    double uOld = eval_1d_spline(d, rMax, drInv, A, coefs);
    for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
    {
      dx = myRnew[iq][0] - c[tid][0];
      dy = myRnew[iq][1] - c[tid][1];
      dz = myRnew[iq][2] - c[tid][2];
      d  = dist(dx, dy, dz);
      if (ptcl1 < (N + first))
        shared_sum[iq][tid] += eval_1d_spline(d, rMax, drInv, A, coefs) - uOld;
    }
    __syncthreads();
  }
  for (int s = (BS >> 1); s > 0; s >>= 1)
  {
    if (tid < s)
      for (int iq = 0; iq < myJob.NumQuadPoints; iq++)
        shared_sum[iq][tid] += shared_sum[iq][tid + s];
    __syncthreads();
  }
  if (tid < myJob.NumQuadPoints)
    myJob.Ratios[tid] *= exp(-shared_sum[tid][0]);
}


void one_body_NLratios(NLjobGPU<float> jobs[],
                       float C[],
                       int first,
                       int last,
                       float spline_coefs[],
                       int numCoefs,
                       float rMax,
                       int numjobs)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  while (numjobs > 65535)
  {
    dim3 dimGrid(65535);
    one_body_NLratio_kernel<BS><<<dimGrid, dimBlock>>>(jobs, C, first, last, spline_coefs, numCoefs, rMax);
    numjobs -= 65535;
    jobs += 65535;
  }
  dim3 dimGrid(numjobs);
  one_body_NLratio_kernel<BS><<<dimGrid, dimBlock>>>(jobs, C, first, last, spline_coefs, numCoefs, rMax);
}


void one_body_NLratios(NLjobGPU<double> jobs[],
                       double C[],
                       int first,
                       int last,
                       double spline_coefs[],
                       int numCoefs,
                       double rMax,
                       int numjobs)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  int blockx = numjobs % 65535;
  int blocky = numjobs / 65535 + 1;
  dim3 dimGrid(blockx, blocky);
  one_body_NLratio_kernel<BS><<<dimGrid, dimBlock>>>(jobs, C, first, last, spline_coefs, numCoefs, rMax);
}


template<typename T, int BS>
__global__ void one_body_grad_kernel(T** R,
                                     int iat,
                                     T* C,
                                     int first,
                                     int last,
                                     T* spline_coefs,
                                     int numCoefs,
                                     T rMax,
                                     bool zeroOut,
                                     T* grad)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T *myR, r[3];
  if (tid == 0)
    myR = R[blockIdx.x];
  __syncthreads();
  if (tid < 3)
    r[tid] = myR[3 * iat + tid];
  __shared__ T coefs[MAX_COEFS];
  if (tid < numCoefs)
    coefs[tid] = spline_coefs[tid];
  __shared__ T c[BS][3];
  __shared__ T A[12][4];
  if (tid < 16)
  {
    A[0 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 0];
    A[4 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 16];
    A[8 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 32];
  }
  __syncthreads();
  int N  = last - first + 1;
  int NB = N / BS + ((N % BS) ? 1 : 0);
  __shared__ T sGrad[BS][3];
  sGrad[tid][0] = sGrad[tid][1] = sGrad[tid][2] = (T)0.0;
  for (int b = 0; b < NB; b++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
      if ((3 * b + i) * BS + tid < 3 * N)
        c[0][i * BS + tid] = C[3 * first + (3 * b + i) * BS + tid];
    __syncthreads();
    int ptcl1 = first + b * BS + tid;
    T dx, dy, dz, u, du, d2u;
    dx  = r[0] - c[tid][0];
    dy  = r[1] - c[tid][1];
    dz  = r[2] - c[tid][2];
    T d = dist(dx, dy, dz);
    eval_1d_spline_vgl(d, rMax, drInv, A, coefs, u, du, d2u);
    if (ptcl1 < (N + first))
    {
      du /= d;
      sGrad[tid][0] += du * dx;
      sGrad[tid][1] += du * dy;
      sGrad[tid][2] += du * dz;
    }
    __syncthreads();
  }
  // Do reduction across threads in block
  for (int s = BS >> 1; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      sGrad[tid][0] += sGrad[tid + s][0];
      sGrad[tid][1] += sGrad[tid + s][1];
      sGrad[tid][2] += sGrad[tid + s][2];
    }
    __syncthreads();
  }
  if (tid < 3)
  {
    if (zeroOut)
      grad[3 * blockIdx.x + tid] = sGrad[0][tid];
    else
      grad[3 * blockIdx.x + tid] += sGrad[0][tid];
  }
}


void one_body_gradient(float* Rlist[],
                       int iat,
                       float C[],
                       int first,
                       int last,
                       float spline_coefs[],
                       int num_coefs,
                       float rMax,
                       bool zeroSum,
                       float grad[],
                       int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_grad_kernel<float, BS><<<dimGrid, dimBlock, 0, gpu::kernelStream>>>(Rlist,
                                                                               iat,
                                                                               C,
                                                                               first,
                                                                               last,
                                                                               spline_coefs,
                                                                               num_coefs,
                                                                               rMax,
                                                                               zeroSum,
                                                                               grad);
}

void one_body_gradient(double* Rlist[],
                       int iat,
                       double C[],
                       int first,
                       int last,
                       double spline_coefs[],
                       int num_coefs,
                       double rMax,
                       bool zeroSum,
                       double grad[],
                       int numWalkers)
{
  if (!AisInitialized)
    cuda_spline_init();
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_grad_kernel<double, BS><<<dimGrid, dimBlock, 0, gpu::kernelStream>>>(Rlist,
                                                                                iat,
                                                                                C,
                                                                                first,
                                                                                last,
                                                                                spline_coefs,
                                                                                num_coefs,
                                                                                rMax,
                                                                                zeroSum,
                                                                                grad);
}


template<typename T, int BS, unsigned COMPLEX>
__global__ void one_body_derivs_kernel(T* C,
                                       T** R,
                                       T** gradLogPsi,
                                       int cfirst,
                                       int clast,
                                       int efirst,
                                       int elast,
                                       int numCoefs,
                                       T rMax,
                                       T** derivs)
{
  T dr    = rMax / (T)(numCoefs - 3);
  T drInv = 1.0 / dr;
  __syncthreads();
  // Safety for rounding error
  rMax *= 0.999999f;
  int tid = threadIdx.x;
  __shared__ T *myR, *myGrad, *myDerivs;
  if (tid == 0)
  {
    myR      = R[blockIdx.x];
    myGrad   = gradLogPsi[blockIdx.x];
    myDerivs = derivs[blockIdx.x];
  }
  __shared__ T sderivs[MAX_COEFS][2];
  __shared__ T r[BS][3], c[BS][3];
  __shared__ T A[12][4];
  if (tid < 16)
  {
    A[0 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 0];
    A[4 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 16];
    A[8 + (tid >> 2)][tid & 3] = AcudaSpline[tid + 32];
  }
  __syncthreads();
  sderivs[tid][0] = T();
  sderivs[tid][1] = T();
  int Nc          = clast - cfirst + 1;
  int Ne          = elast - efirst + 1;
  int NBc         = (Nc + BS - 1) / BS;
  int NBe         = (Ne + BS - 1) / BS;
  __shared__ T sGrad[BS][3];
  for (int be = 0; be < NBe; be++)
  {
    // Load block of positions from global memory
    for (int i = 0; i < 3; i++)
      if ((3 * be + i) * BS + tid < 3 * Ne)
      {
        int outoff       = i * BS + tid;
        int inoff        = outoff + 3 * efirst + 3 * be * BS;
        r[0][outoff]     = myR[inoff];
        sGrad[0][outoff] = myGrad[inoff * COMPLEX];
      }
    __syncthreads();
    int eptcl = efirst + be * BS + tid;
    for (int bc = 0; bc < NBc; bc++)
    {
      // Load block of positions from global memory
      for (int i = 0; i < 3; i++)
        if ((3 * bc + i) * BS + tid < 3 * Nc)
          c[0][i * BS + tid] = C[3 * cfirst + (3 * bc + i) * BS + tid];
      __syncthreads();
      // Now, loop over particles
      int end = min(BS, Nc - bc * BS);
      for (int j = 0; j < end; j++)
      {
        T dx, dy, dz;
        dx        = c[j][0] - r[tid][0];
        dy        = c[j][1] - r[tid][1];
        dz        = c[j][2] - r[tid][2];
        T d       = dist(dx, dy, dz);
        T dInv    = 1.0f / d;
        T s       = d * drInv;
        T sf      = floorf(s);
        int index = (int)sf;
        T t       = s - sf;
        T t2      = t * t;
        T t3      = t * t2;
        T v0      = (A[0][0] * t3 + A[0][1] * t2 + A[0][2] * t + A[0][3]);
        T v1      = (A[1][0] * t3 + A[1][1] * t2 + A[1][2] * t + A[1][3]);
        T v2      = (A[2][0] * t3 + A[2][1] * t2 + A[2][2] * t + A[2][3]);
        T v3      = (A[3][0] * t3 + A[3][1] * t2 + A[3][2] * t + A[3][3]);
        for (int id = 0; id < BS; id++)
          if (tid == id && eptcl <= elast && (d < rMax))
          {
            sderivs[index + 0][0] += v0;
            sderivs[index + 1][0] += v1;
            sderivs[index + 2][0] += v2;
            sderivs[index + 3][0] += v3;
          }
        T prefact = (dx * sGrad[tid][0] + dy * sGrad[tid][1] + dz * sGrad[tid][2]) * dInv;
        T du0     = drInv * (A[4][0] * t3 + A[4][1] * t2 + A[4][2] * t + A[4][3]);
        T du1     = drInv * (A[5][0] * t3 + A[5][1] * t2 + A[5][2] * t + A[5][3]);
        T du2     = drInv * (A[6][0] * t3 + A[6][1] * t2 + A[6][2] * t + A[6][3]);
        T du3     = drInv * (A[7][0] * t3 + A[7][1] * t2 + A[7][2] * t + A[7][3]);
        // This is the dot (gradu, grad_log_psi) term.
        v0 = 2.0f * prefact * du0;
        v1 = 2.0f * prefact * du1;
        v2 = 2.0f * prefact * du2;
        v3 = 2.0f * prefact * du3;
        // This is the lapl u term
        v0 -= drInv * drInv * (A[8][0] * t3 + A[8][1] * t2 + A[8][2] * t + A[8][3]) + 2.0f * du0 * dInv;
        v1 -= drInv * drInv * (A[9][0] * t3 + A[9][1] * t2 + A[9][2] * t + A[9][3]) + 2.0f * du1 * dInv;
        v2 -= drInv * drInv * (A[10][0] * t3 + A[10][1] * t2 + A[10][2] * t + A[10][3]) + 2.0f * du2 * dInv;
        v3 -= drInv * drInv * (A[11][0] * t3 + A[11][1] * t2 + A[11][2] * t + A[11][3]) + 2.0f * du3 * dInv;
        for (int id = 0; id < BS; id++)
          if (tid == id && eptcl <= elast && (d < rMax))
          {
            sderivs[index + 0][1] += v0;
            sderivs[index + 1][1] += v1;
            sderivs[index + 2][1] += v2;
            sderivs[index + 3][1] += v3;
          }
      }
      __syncthreads();
    }
  }
  sderivs[tid][1] *= 0.5f;
  if (tid < 2 * numCoefs)
    myDerivs[tid] = -sderivs[0][tid];
  if (tid + BS < 2 * numCoefs)
    myDerivs[tid + BS] = -sderivs[0][tid + BS];
}


void one_body_derivs(float C[],
                     float* R[],
                     float* gradLogPsi[],
                     int cfirst,
                     int clast,
                     int efirst,
                     int elast,
                     int numCoefs,
                     float rMax,
                     float* derivs[],
                     int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_derivs_kernel<float, BS, 1>
      <<<dimGrid, dimBlock>>>(C, R, gradLogPsi, cfirst, clast, efirst, elast, numCoefs, rMax, derivs);
}


void one_body_derivs(double C[],
                     double* R[],
                     double* gradLogPsi[],
                     int cfirst,
                     int clast,
                     int efirst,
                     int elast,
                     int numCoefs,
                     double rMax,
                     double* derivs[],
                     int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  one_body_derivs_kernel<double, BS, 1>
      <<<dimGrid, dimBlock>>>(C, R, gradLogPsi, cfirst, clast, efirst, elast, numCoefs, rMax, derivs);
}

// Ye: use offset to recycle the old routines
// block size can be further optimized.
#ifdef QMC_COMPLEX
void one_body_derivs(float C[],
                     float* R[],
                     std::complex<float>* gradLogPsi[],
                     int cfirst,
                     int clast,
                     int efirst,
                     int elast,
                     int numCoefs,
                     float rMax,
                     float* derivs[],
                     int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  one_body_derivs_kernel<float, BS, 2>
      <<<dimGrid, dimBlock>>>(C, R, (float**)gradLogPsi, cfirst, clast, efirst, elast, numCoefs, rMax, derivs);
}


void one_body_derivs(double C[],
                     double* R[],
                     std::complex<double>* gradLogPsi[],
                     int cfirst,
                     int clast,
                     int efirst,
                     int elast,
                     int numCoefs,
                     double rMax,
                     double* derivs[],
                     int numWalkers)
{
  const int BS = 32;
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);

  one_body_derivs_kernel<double, BS, 2>
      <<<dimGrid, dimBlock>>>(C, R, (double**)gradLogPsi, cfirst, clast, efirst, elast, numCoefs, rMax, derivs);
}

#endif

void test()
{
  dim3 dimBlock(32);
  dim3 dimGrid(1000);
  float* R[1000];
  float spline_coefs[10];
  float dr = 0.1;
  float sum[1000];
  two_body_sum_kernel<float, 32><<<dimGrid, dimBlock>>>(R, 0, 100, 0, 100, spline_coefs, 10, dr, sum);
}
