#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory 
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov 
//    Lawrence Livermore National Laboratory 
////////////////////////////////////////////////////////////////////////////////

#include<cassert>
#include <complex>
#include<hip/hip_runtime.h>
#include <thrust/complex.h>
#include<hip/hip_runtime.h>
#include "AFQMC/Numerics/detail/CUDA/Kernels/cuda_settings.h"
#define ENABLE_CUDA 1
#include "AFQMC/Memory/CUDA/cuda_utilities.h"
#if __CUDA_ARCH__ < 600
#include "AFQMC/Numerics/detail/CUDA/Kernels/myAtomicAdd.cu"
#endif

namespace kernels 
{

// Tab [nbatch][nwalk][nocc][nocc][nchol]
template<typename T, typename T2>
__global__ void kernel_batched_dot_wabn_wban(int nbatch, int nwalk, int nocc, int nchol,
                    thrust::complex<T2> const* alpha, thrust::complex<T2> const* Tab, 
                    thrust::complex<T>* y, int incy)
{
    int batch = blockIdx.x;
    if( batch >= nbatch ) return;
    if( blockIdx.y >= nwalk*nocc*nocc ) return;
    __shared__ thrust::complex<T> cache[ DOT_BLOCK_SIZE ];
    int nocc2 = nocc*nocc;
    int w = blockIdx.y/(nocc2);
    int a = (blockIdx.y%(nocc2))/nocc;
    int b = (blockIdx.y%(nocc2))%nocc;
    int i = threadIdx.x;
    thrust::complex<T> alp = static_cast<thrust::complex<T>>(alpha[batch]);
    thrust::complex<T2> const* A_(Tab + 2*batch*nwalk*nocc2*nchol + ((w*nocc+a)*nocc + b)*nchol);
    thrust::complex<T2> const* B_(Tab + (2*batch+1)*nwalk*nocc2*nchol + ((w*nocc+b)*nocc + a)*nchol);
    cache[ threadIdx.x ] = thrust::complex<T>(0.0);
    while( i < nchol ) {
        cache[ threadIdx.x ] += static_cast<thrust::complex<T>>(A_[ i ] * B_[ i ]);
        i += blockDim.x;
    }
    __syncthreads(); // required because later on the current thread is accessing
                     // data written by another thread    
    i = DOT_BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
    //if( threadIdx.x == 0 ) *(y+w*incy) = (*(y+w*incy)) + alp * cache[ 0 ];
    if( threadIdx.x == 0 ) {
        T re = (alp * cache[ 0 ]).real();
        T im = (alp * cache[ 0 ]).imag();
        T* re_ = reinterpret_cast<T*>(y+w*incy);
#if __CUDA_ARCH__ < 600
        myAtomicAdd(re_,re); 
        myAtomicAdd(re_+1,im); 
#else
        atomicAdd(re_,re); 
        atomicAdd(re_+1,im); 
#endif
    }
}

template<typename T, typename T2>
__global__ void kernel_batched_dot_wanb_wbna(int nbatch, int nwalk, int nocc, int nchol,
                    thrust::complex<T2> const* alpha, thrust::complex<T2> const* Tab,
                    thrust::complex<T>* y, int incy)
{
    int batch = blockIdx.x;
    if( batch >= nbatch ) return;
    if( blockIdx.y >= nwalk*nocc*nocc ) return;
    __shared__ thrust::complex<T> cache[ DOT_BLOCK_SIZE ];
    int nocc2 = nocc*nocc;
    int w = blockIdx.y/(nocc2);
    int a = (blockIdx.y%(nocc2))/nocc;
    int b = (blockIdx.y%(nocc2))%nocc;
    int i = threadIdx.x;
    thrust::complex<T> alp = static_cast<thrust::complex<T>>(alpha[batch]);
    thrust::complex<T2> const* A_(Tab + 2*batch*nwalk*nocc2*nchol + ((w*nocc+a)*nocc)*nchol+b);
    thrust::complex<T2> const* B_(Tab + (2*batch+1)*nwalk*nocc2*nchol + ((w*nocc+b)*nocc)*nchol+a);
    cache[ threadIdx.x ] = thrust::complex<T>(0.0);
    while( i < nchol ) {
        cache[ threadIdx.x ] += static_cast<thrust::complex<T>>(A_[ i*nocc ] * B_[ i*nocc ]);
        i += blockDim.x;
    }
    __syncthreads(); // required because later on the current thread is accessing
                     // data written by another thread    
    i = DOT_BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
    //if( threadIdx.x == 0 ) *(y+w*incy) = (*(y+w*incy)) + alp * cache[ 0 ];
    if( threadIdx.x == 0 ) {
        T re = (alp * cache[ 0 ]).real();
        T im = (alp * cache[ 0 ]).imag();
        T* re_ = reinterpret_cast<T*>(y+w*incy);
#if __CUDA_ARCH__ < 600
        myAtomicAdd(re_,re);
        myAtomicAdd(re_+1,im);
#else
        atomicAdd(re_,re);
        atomicAdd(re_+1,im);
#endif
    }
}

void batched_dot_wabn_wban( int nbatch, int nwalk, int nocc, int nchol, 
                    std::complex<double> const* alpha, std::complex<double> const* Tab, 
                    std::complex<double>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(nbatch,n_,1);
  kernel_batched_dot_wabn_wban<<<grid_dim,DOT_BLOCK_SIZE>>>(nbatch,nwalk,nocc,nchol,
                                   reinterpret_cast<thrust::complex<double> const*>(alpha),
                                   reinterpret_cast<thrust::complex<double> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"batched_dot_wabn_wban");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"batched_dot_wabn_wban");
}

void batched_dot_wabn_wban( int nbatch, int nwalk, int nocc, int nchol, 
                    std::complex<float> const* alpha, std::complex<float> const* Tab,
                    std::complex<float>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(nbatch,n_,1);
  kernel_batched_dot_wabn_wban<<<grid_dim,DOT_BLOCK_SIZE>>>(nbatch,nwalk,nocc,nchol,
                                   reinterpret_cast<thrust::complex<float> const*>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<float> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"batched_dot_wabn_wban");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"batched_dot_wabn_wban");
}

void batched_dot_wabn_wban( int nbatch, int nwalk, int nocc, int nchol, 
                    std::complex<float> const* alpha, std::complex<float> const* Tab,
                    std::complex<double>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(nbatch,n_,1);
  kernel_batched_dot_wabn_wban<<<grid_dim,DOT_BLOCK_SIZE>>>(nbatch,nwalk,nocc,nchol,
                                   reinterpret_cast<thrust::complex<float> const*>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"batched_dot_wabn_wban");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"batched_dot_wabn_wban");
}

// anb/bna
void batched_dot_wanb_wbna( int nbatch, int nwalk, int nocc, int nchol,
                    std::complex<double> const* alpha, std::complex<double> const* Tab,
                    std::complex<double>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(nbatch,n_,1);
  kernel_batched_dot_wanb_wbna<<<grid_dim,DOT_BLOCK_SIZE>>>(nbatch,nwalk,nocc,nchol,
                                   reinterpret_cast<thrust::complex<double> const*>(alpha),
                                   reinterpret_cast<thrust::complex<double> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"batched_dot_wanb_wbna");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"batched_dot_wanb_wbna");
}

void batched_dot_wanb_wbna( int nbatch, int nwalk, int nocc, int nchol,
                    std::complex<float> const* alpha, std::complex<float> const* Tab,
                    std::complex<float>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(nbatch,n_,1);
  kernel_batched_dot_wanb_wbna<<<grid_dim,DOT_BLOCK_SIZE>>>(nbatch,nwalk,nocc,nchol,
                                   reinterpret_cast<thrust::complex<float> const*>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<float> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"batched_dot_wanb_wbna");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"batched_dot_wanb_wbna");
}

void batched_dot_wanb_wbna( int nbatch, int nwalk, int nocc, int nchol,
                    std::complex<float> const* alpha, std::complex<float> const* Tab,
                    std::complex<double>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(nbatch,n_,1);
  kernel_batched_dot_wanb_wbna<<<grid_dim,DOT_BLOCK_SIZE>>>(nbatch,nwalk,nocc,nchol,
                                   reinterpret_cast<thrust::complex<float> const*>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"batched_dot_wanb_wbna");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"batched_dot_wanb_wbna");
}


}
