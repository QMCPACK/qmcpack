#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov
//    Lawrence Livermore National Laboratory
////////////////////////////////////////////////////////////////////////////////

#include <cassert>
#include <complex>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include "AFQMC/Numerics/detail/CUDA/Kernels/cuda_settings.h"
#define ENABLE_CUDA 1
#include "AFQMC/Memory/CUDA/cuda_utilities.h"

namespace kernels
{
// very sloppy, needs improvement!!!!
// A[nocc_tot][nmo_tot][nwalk]
// B[Q][K][nocc_max][nmo_max][nwalk]
template<typename T, typename T2>
__global__ void kernel_KaKjw_to_QKajw(int nwalk,
                                      int nkpts,
                                      int npol,
                                      int nmo_max,
                                      int nmo_tot,
                                      int nocc_max,
                                      int* nmo,
                                      int* nmo0,
                                      int* nocc,
                                      int* nocc0,
                                      int* QKtok2,
                                      T const* A,
                                      T2* B)
{
  int Q   = blockIdx.x;
  int K   = blockIdx.y;
  int pol = blockIdx.z;
  if (Q >= nkpts || K >= nkpts || pol > npol)
    return;
  int QK  = QKtok2[Q * nkpts + K];
  int na0 = nocc0[K];
  int nj0 = nmo0[QK];
  int na  = nocc[K];
  int nj  = nmo[QK];

  T const* A_(A + (na0 * npol * nmo_tot + nj0) * nwalk);
  T2* B_(B + ((Q * nkpts + K) * nocc_max) * npol * nmo_max * nwalk);

  if (threadIdx.x >= nj)
    return;
  if (threadIdx.y >= nwalk)
    return;

  for (int a = 0, a0 = pol * nmo_max * nwalk, a1 = pol * nmo_tot * nwalk; a < na;
       a++, a0 += npol * nmo_max * nwalk, a1 += npol * nmo_tot * nwalk)
    for (int j = threadIdx.x; j < nj; j += blockDim.x)
      for (int n = threadIdx.y; n < nwalk; n += blockDim.y)
        B_[a0 + j * nwalk + n] = static_cast<T2>(A_[a1 + j * nwalk + n]);
}

template<typename T, typename T2>
__global__ void kernel_KaKjw_to_QKajw(int nwalk,
                                      int nkpts,
                                      int npol,
                                      int nmo_max,
                                      int nmo_tot,
                                      int nocc_max,
                                      int* nmo,
                                      int* nmo0,
                                      int* nocc,
                                      int* nocc0,
                                      int* QKtok2,
                                      thrust::complex<T> const* A,
                                      thrust::complex<T2>* B)
{
  int Q   = blockIdx.x;
  int K   = blockIdx.y;
  int pol = blockIdx.z;
  if (Q >= nkpts || K >= nkpts || pol > npol)
    return;
  int QK  = QKtok2[Q * nkpts + K];
  int na0 = nocc0[K];
  int nj0 = nmo0[QK];
  int na  = nocc[K];
  int nj  = nmo[QK];

  thrust::complex<T> const* A_(A + (na0 * npol * nmo_tot + nj0) * nwalk);
  thrust::complex<T2>* B_(B + ((Q * nkpts + K) * nocc_max) * npol * nmo_max * nwalk);

  if (threadIdx.x >= nj)
    return;
  if (threadIdx.y >= nwalk)
    return;

  for (int a = 0, a0 = pol * nmo_max * nwalk, a1 = pol * nmo_tot * nwalk; a < na;
       a++, a0 += npol * nmo_max * nwalk, a1 += npol * nmo_tot * nwalk)
  {
    for (int j = threadIdx.x; j < nj; j += blockDim.x)
      for (int n = threadIdx.y; n < nwalk; n += blockDim.y)
        B_[a0 + j * nwalk + n] = static_cast<thrust::complex<T2>>(A_[a1 + j * nwalk + n]);
  }
}

void KaKjw_to_QKajw(int nwalk,
                    int nkpts,
                    int npol,
                    int nmo_max,
                    int nmo_tot,
                    int nocc_max,
                    int* nmo,
                    int* nmo0,
                    int* nocc,
                    int* nocc0,
                    int* QKtok2,
                    double const* A,
                    double* B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk, 32);
  dim3 block_dim(xblock_dim, yblock_dim, 1);
  dim3 grid_dim(nkpts, nkpts, npol);
  kernel_KaKjw_to_QKajw<<<grid_dim, block_dim>>>(nwalk, nkpts, npol, nmo_max, nmo_tot, nocc_max, nmo, nmo0, nocc, nocc0,
                                                 QKtok2, A, B);
  qmc_cuda::cuda_check(hipGetLastError(), "KaKjw_to_QKajw");
  qmc_cuda::cuda_check(hipDeviceSynchronize(), "KaKjw_to_QKajw");
}

void KaKjw_to_QKajw(int nwalk,
                    int nkpts,
                    int npol,
                    int nmo_max,
                    int nmo_tot,
                    int nocc_max,
                    int* nmo,
                    int* nmo0,
                    int* nocc,
                    int* nocc0,
                    int* QKtok2,
                    float const* A,
                    float* B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk, 32);
  dim3 block_dim(xblock_dim, yblock_dim, 1);
  dim3 grid_dim(nkpts, nkpts, npol);
  kernel_KaKjw_to_QKajw<<<grid_dim, block_dim>>>(nwalk, nkpts, npol, nmo_max, nmo_tot, nocc_max, nmo, nmo0, nocc, nocc0,
                                                 QKtok2, A, B);
  qmc_cuda::cuda_check(hipGetLastError(), "KaKjw_to_QKajw");
  qmc_cuda::cuda_check(hipDeviceSynchronize(), "KaKjw_to_QKajw");
}

void KaKjw_to_QKajw(int nwalk,
                    int nkpts,
                    int npol,
                    int nmo_max,
                    int nmo_tot,
                    int nocc_max,
                    int* nmo,
                    int* nmo0,
                    int* nocc,
                    int* nocc0,
                    int* QKtok2,
                    double const* A,
                    float* B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk, 32);
  dim3 block_dim(xblock_dim, yblock_dim, 1);
  dim3 grid_dim(nkpts, nkpts, npol);
  kernel_KaKjw_to_QKajw<<<grid_dim, block_dim>>>(nwalk, nkpts, npol, nmo_max, nmo_tot, nocc_max, nmo, nmo0, nocc, nocc0,
                                                 QKtok2, A, B);
  qmc_cuda::cuda_check(hipGetLastError(), "KaKjw_to_QKajw");
  qmc_cuda::cuda_check(hipDeviceSynchronize(), "KaKjw_to_QKajw");
}

void KaKjw_to_QKajw(int nwalk,
                    int nkpts,
                    int npol,
                    int nmo_max,
                    int nmo_tot,
                    int nocc_max,
                    int* nmo,
                    int* nmo0,
                    int* nocc,
                    int* nocc0,
                    int* QKtok2,
                    std::complex<float> const* A,
                    std::complex<float>* B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk, 32);
  dim3 block_dim(xblock_dim, yblock_dim, 1);
  dim3 grid_dim(nkpts, nkpts, npol);
  kernel_KaKjw_to_QKajw<<<grid_dim, block_dim>>>(nwalk, nkpts, npol, nmo_max, nmo_tot, nocc_max, nmo, nmo0, nocc, nocc0,
                                                 QKtok2, reinterpret_cast<thrust::complex<float> const*>(A),
                                                 reinterpret_cast<thrust::complex<float>*>(B));
  qmc_cuda::cuda_check(hipGetLastError(), "KaKjw_to_QKajw");
  qmc_cuda::cuda_check(hipDeviceSynchronize(), "KaKjw_to_QKajw");
}

void KaKjw_to_QKajw(int nwalk,
                    int nkpts,
                    int npol,
                    int nmo_max,
                    int nmo_tot,
                    int nocc_max,
                    int* nmo,
                    int* nmo0,
                    int* nocc,
                    int* nocc0,
                    int* QKtok2,
                    std::complex<double> const* A,
                    std::complex<double>* B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk, 32);
  dim3 block_dim(xblock_dim, yblock_dim, 1);
  dim3 grid_dim(nkpts, nkpts, npol);
  kernel_KaKjw_to_QKajw<<<grid_dim, block_dim>>>(nwalk, nkpts, npol, nmo_max, nmo_tot, nocc_max, nmo, nmo0, nocc, nocc0,
                                                 QKtok2, reinterpret_cast<thrust::complex<double> const*>(A),
                                                 reinterpret_cast<thrust::complex<double>*>(B));
  qmc_cuda::cuda_check(hipGetLastError(), "KaKjw_to_QKajw");
  qmc_cuda::cuda_check(hipDeviceSynchronize(), "KaKjw_to_QKajw");
}

void KaKjw_to_QKajw(int nwalk,
                    int nkpts,
                    int npol,
                    int nmo_max,
                    int nmo_tot,
                    int nocc_max,
                    int* nmo,
                    int* nmo0,
                    int* nocc,
                    int* nocc0,
                    int* QKtok2,
                    std::complex<double> const* A,
                    std::complex<float>* B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk, 32);
  dim3 block_dim(xblock_dim, yblock_dim, 1);
  dim3 grid_dim(nkpts, nkpts, npol);
  kernel_KaKjw_to_QKajw<<<grid_dim, block_dim>>>(nwalk, nkpts, npol, nmo_max, nmo_tot, nocc_max, nmo, nmo0, nocc, nocc0,
                                                 QKtok2, reinterpret_cast<thrust::complex<double> const*>(A),
                                                 reinterpret_cast<thrust::complex<float>*>(B));
  qmc_cuda::cuda_check(hipGetLastError(), "KaKjw_to_QKajw");
  qmc_cuda::cuda_check(hipDeviceSynchronize(), "KaKjw_to_QKajw");
}

} // namespace kernels
