#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory 
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov 
//    Lawrence Livermore National Laboratory 
////////////////////////////////////////////////////////////////////////////////

#include<cassert>
#include <complex>
#include<hip/hip_runtime.h>
#include <thrust/complex.h>
#include<hip/hip_runtime.h>
#include "AFQMC/Numerics/detail/CUDA/Kernels/cuda_settings.h"
#define ENABLE_CUDA 1
#include "AFQMC/Memory/CUDA/cuda_utilities.h"
#if __CUDA_ARCH__ < 600
#include "AFQMC/Numerics/detail/CUDA/Kernels/myAtomicAdd.cu"
#endif

namespace kernels 
{

// Tab nwalk][nocc][nocc][nchol]
template<typename T, typename T2>
__global__ void kernel_dot_wabn(int nwalk, int nocc, int nchol,
                    thrust::complex<T2> const alpha, thrust::complex<T2> const* Tab, 
                    thrust::complex<T>* y, int incy)
{
    if( blockIdx.x >= nwalk*nocc*nocc ) return;
    __shared__ thrust::complex<T> cache[ DOT_BLOCK_SIZE ];
    int nocc2 = nocc*nocc;
    int w = blockIdx.x/(nocc2);
    int a = (blockIdx.x%(nocc2))/nocc;
    int b = (blockIdx.x%(nocc2))%nocc;
    int i = threadIdx.x;
    thrust::complex<T> alp = static_cast<thrust::complex<T>>(alpha);
    thrust::complex<T2> const* A_(Tab + ((w*nocc+a)*nocc + b)*nchol);
    thrust::complex<T2> const* B_(Tab + ((w*nocc+b)*nocc + a)*nchol);
    cache[ threadIdx.x ] = thrust::complex<T>(0.0);
    while( i < nchol ) {
        cache[ threadIdx.x ] += static_cast<thrust::complex<T>>(A_[ i ] * B_[ i ]);
        i += blockDim.x;
    }
    __syncthreads(); // required because later on the current thread is accessing
                     // data written by another thread    
    i = DOT_BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
    if( threadIdx.x == 0 ) {
        T re = (alp * cache[ 0 ]).real();
        T im = (alp * cache[ 0 ]).imag();
        T* re_ = reinterpret_cast<T*>(y+w*incy);
#if __CUDA_ARCH__ < 600
        myAtomicAdd(re_,re); 
        myAtomicAdd(re_+1,im); 
#else
        atomicAdd(re_,re); 
        atomicAdd(re_+1,im); 
#endif
    }
}

template<typename T, typename T2>
__global__ void kernel_dot_wanb(int nt, int nwalk, int nocc, int nchol, 
                    thrust::complex<T2> const alpha, thrust::complex<T2> const* Tab,
                    thrust::complex<T>* y, int incy)
{
    if( blockIdx.x >= nwalk ) return;

    int a = blockIdx.y*blockDim.x + threadIdx.x;
    int nb = blockIdx.z*blockDim.y*nt + threadIdx.y;

    __shared__ thrust::complex<T> cache[ 1024 ];

    int nid = blockDim.x*blockDim.y*blockDim.z;
    int id = (threadIdx.x * blockDim.y + threadIdx.y) *blockDim.z + threadIdx.z;

    cache[ id ] = thrust::complex<T>(0.0);
    thrust::complex<T> alp = static_cast<thrust::complex<T>>(alpha);
    thrust::complex<T2> const* A_(Tab + blockIdx.x*nocc*nocc*nchol);
    for(int n=0; n<nt; n++, nb+=blockDim.y) {
      if(a >= nocc || nb >= nocc*nchol) break;
      int i = nb/nocc;
      int b = nb%nocc;
      cache[id] += static_cast<thrust::complex<T>>( A_[(a*nchol + i)*nocc + b] *
                    A_[(b*nchol + i)*nocc + a]);
    }

    __syncthreads(); // required because later on the current thread is accessing
                     // data written by another thread    
    int i = nid / 2;
    while( i > 0 ) {
        if( id < i ) cache[ id ] += cache[ id + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
    if( id == 0 ) {
        T re = (alp * cache[ 0 ]).real();
        T im = (alp * cache[ 0 ]).imag();
        T* re_ = reinterpret_cast<T*>(y+blockIdx.x*incy);
#if __CUDA_ARCH__ < 600
        myAtomicAdd(re_,re);
        myAtomicAdd(re_+1,im);
#else
        atomicAdd(re_,re);
        atomicAdd(re_+1,im);
#endif
    }
}

template<typename T, typename T2>
__global__ void kernel_dot_wanb2(int nwalk, int nocc, int nchol, 
                    thrust::complex<T2> const alpha, thrust::complex<T2> const* Tab,
                    thrust::complex<T>* y, int incy)
{
    if( blockIdx.x >= nwalk*nocc*nocc ) return;
    __shared__ thrust::complex<T> cache[ DOT_BLOCK_SIZE ];
    int nocc2 = nocc*nocc;
    int w = blockIdx.x/(nocc2);
    int a = (blockIdx.x%(nocc2))/nocc;
    int b = (blockIdx.x%(nocc2))%nocc;
    int i = threadIdx.x;
    thrust::complex<T> alp = static_cast<thrust::complex<T>>(alpha);
    thrust::complex<T2> const* A_(Tab + ((w*nocc+a)*nocc)*nchol+b);
    thrust::complex<T2> const* B_(Tab + ((w*nocc+b)*nocc)*nchol+a);
    cache[ threadIdx.x ] = thrust::complex<T>(0.0);
    while( i < nchol ) {
        cache[ threadIdx.x ] += static_cast<thrust::complex<T>>(A_[ i*nocc ] * B_[ i*nocc ]);
        i += blockDim.x;
    }
    __syncthreads(); // required because later on the current thread is accessing
                     // data written by another thread    
    i = DOT_BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
    if( threadIdx.x == 0 ) {
        T re = (alp * cache[ 0 ]).real();
        T im = (alp * cache[ 0 ]).imag();
        T* re_ = reinterpret_cast<T*>(y+w*incy);
#if __CUDA_ARCH__ < 600
        myAtomicAdd(re_,re);
        myAtomicAdd(re_+1,im);
#else
        atomicAdd(re_,re);
        atomicAdd(re_+1,im);
#endif
    }
}

void dot_wabn( int nwalk, int nocc, int nchol, 
               std::complex<double> const alpha, std::complex<double> const* Tab, 
               std::complex<double>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(n_,1,1);
  kernel_dot_wabn<<<grid_dim,DOT_BLOCK_SIZE>>>(nwalk,nocc,nchol,
                                   static_cast<thrust::complex<double> const>(alpha),
                                   reinterpret_cast<thrust::complex<double> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wabn");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wabn");
}

void dot_wabn( int nwalk, int nocc, int nchol,                         
               std::complex<float> const alpha, std::complex<float> const* Tab,      
               std::complex<float>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(n_,1,1);
  kernel_dot_wabn<<<grid_dim,DOT_BLOCK_SIZE>>>(nwalk,nocc,nchol,
                                   static_cast<thrust::complex<float> const>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<float> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wabn");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wabn");
}

void dot_wabn( int nwalk, int nocc, int nchol,                         
               std::complex<float> const alpha, std::complex<float> const* Tab,      
               std::complex<double>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(n_,1,1);
  kernel_dot_wabn<<<grid_dim,DOT_BLOCK_SIZE>>>(nwalk,nocc,nchol,
                                   static_cast<thrust::complex<float> const>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wabn");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wabn");
}

// v2
void dot_wanb( int nwalk, int nocc, int nchol,
               std::complex<double> const alpha, std::complex<double> const* Tab,
               std::complex<double>* y, int incy)
{ 
  int a_(8); 
  int nf(8);
  int b_ = 1024/a_;
  int na = (nocc-1)/a_+1;
  int nb = (nocc*nchol-1)/(b_*nf)+1;
  dim3 grid_dim(nwalk,na,nb);
  dim3 block_dim(a_,b_,1);
  kernel_dot_wanb<<<grid_dim,block_dim>>>(nf,nwalk,nocc,nchol,
                                   static_cast<thrust::complex<double> const>(alpha),
                                   reinterpret_cast<thrust::complex<double> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wanb");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wanb");
}

void dot_wanb( int nwalk, int nocc, int nchol,
               std::complex<float> const alpha, std::complex<float> const* Tab,
               std::complex<float>* y, int incy)
{
  int a_(8);
  int nf(8);
  int b_ = 1024/a_;
  int na = (nocc-1)/a_+1;
  int nb = (nocc*nchol-1)/(b_*nf)+1;
  dim3 grid_dim(nwalk,na,nb);
  dim3 block_dim(a_,b_,1);
  kernel_dot_wanb<<<grid_dim,block_dim>>>(nf,nwalk,nocc,nchol,
                                   static_cast<thrust::complex<float> const>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<float> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wanb");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wanb");
}

void dot_wanb( int nwalk, int nocc, int nchol,
               std::complex<float> const alpha, std::complex<float> const* Tab,
               std::complex<double>* y, int incy)
{
  int a_(8);
  int nf(8);
  int b_ = 1024/a_;
  int na = (nocc-1)/a_+1;
  int nb = (nocc*nchol-1)/(b_*nf)+1;
  dim3 grid_dim(nwalk,na,nb);
  dim3 block_dim(a_,b_,1);
  kernel_dot_wanb<<<grid_dim,block_dim>>>(nf,nwalk,nocc,nchol,
                                   static_cast<thrust::complex<float> const>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wanb");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wanb");
}

/*
// v2
void dot_wanb( int nwalk, int nocc, int nchol, 
               std::complex<double> const alpha, std::complex<double> const* Tab,
               std::complex<double>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(n_,1,1);
  kernel_dot_wanb2<<<grid_dim,DOT_BLOCK_SIZE>>>(nwalk,nocc,nchol,
                                   static_cast<thrust::complex<double> const>(alpha),
                                   reinterpret_cast<thrust::complex<double> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wanb");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wanb");
}

void dot_wanb( int nwalk, int nocc, int nchol, 
               std::complex<float> const alpha, std::complex<float> const* Tab,
               std::complex<float>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(n_,1,1);
  kernel_dot_wanb2<<<grid_dim,DOT_BLOCK_SIZE>>>(nwalk,nocc,nchol,
                                   static_cast<thrust::complex<float> const>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<float> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wanb");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wanb");
}

void dot_wanb( int nwalk, int nocc, int nchol, 
               std::complex<float> const alpha, std::complex<float> const* Tab,
               std::complex<double>* y, int incy)
{
  int n_=nwalk*nocc*nocc;
  dim3 grid_dim(n_,1,1);
  kernel_dot_wanb2<<<grid_dim,DOT_BLOCK_SIZE>>>(nwalk,nocc,nchol,
                                   static_cast<thrust::complex<float> const>(alpha),
                                   reinterpret_cast<thrust::complex<float> const*>(Tab),
                                   reinterpret_cast<thrust::complex<double> *>(y),incy);
  qmc_cuda::cuda_check(hipGetLastError(),"dot_wanb");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"dot_wanb");
}
*/

}
