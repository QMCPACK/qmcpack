#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory 
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov 
//    Lawrence Livermore National Laboratory 
////////////////////////////////////////////////////////////////////////////////

#include<cassert>
#include <complex>
#include<hip/hip_runtime.h>
#include <thrust/complex.h>
#include<hip/hip_runtime.h>
#include "AFQMC/Numerics/detail/CUDA/Kernels/cuda_settings.h"
#define ENABLE_CUDA 1
#include "AFQMC/Memory/CUDA/cuda_utilities.h"

namespace kernels
{

// very sloppy, needs improvement!!!!
// A[nocc_tot][nmo_tot][nwalk]
// B[Ka][Kj][nwalk][nocc_max][nmo_max]
template<typename T, typename T2>
__global__ void kernel_KaKjw_to_KKwaj( int nwalk, int nkpts, int nmo_max, int nmo_tot, 
                                       int nocc_max, int* nmo, int* nmo0, 
                                       int* nocc, int* nocc0, 
                                       T const* A, T2 * B)
{
  int Ka = blockIdx.x;  
  int Kj = blockIdx.y;  
  if(Ka >= nkpts || Kj >= nkpts) return;  
  int na0 = nocc0[Ka]; 
  int nj0 = nmo0[Kj]; 
  int na = nocc[Ka]; 
  int nj = nmo[Kj];
  
  T const* A_(A + ( na0*nmo_tot + nj0 )*nwalk); 
  T2* B_(B + ( (Ka*nkpts + Kj)*nocc_max )*nmo_max*nwalk);

  if(threadIdx.x >= nj) return; 
  if(threadIdx.y >= nwalk) return; 
  
  for(int a=0, a1=0; a<na; a++, a1+=nmo_tot*nwalk) 
    for(int j=threadIdx.x; j<nj; j+=blockDim.x) 
      for(int n=threadIdx.y; n<nwalk; n+=blockDim.y) 
        B_[ n*nocc_max*nmo_max + a*nmo_max + j ] = static_cast<T2>(A_[ a1 + j*nwalk + n ]);
}

template<typename T, typename T2>
__global__ void kernel_KaKjw_to_KKwaj( int nwalk, int nkpts, int nmo_max, int nmo_tot, 
                                       int nocc_max, int* nmo, int* nmo0,
                                       int* nocc, int* nocc0, 
                                       thrust::complex<T> const* A, thrust::complex<T2> * B)
{
  int Ka = blockIdx.x;
  int Kj = blockIdx.y;
  if(Ka >= nkpts || Kj >= nkpts) return;
  int na0 = nocc0[Ka];
  int nj0 = nmo0[Kj];
  int na = nocc[Ka];
  int nj = nmo[Kj];

  thrust::complex<T> const* A_(A + ( na0*nmo_tot + nj0 )*nwalk);
  thrust::complex<T2>* B_(B + ( (Ka*nkpts + Kj)*nocc_max )*nmo_max*nwalk);

  if(threadIdx.x >= nj) return;
  if(threadIdx.y >= nwalk) return;

  for(int a=0, a1=0; a<na; a++, a1+=nmo_tot*nwalk)                         
    for(int j=threadIdx.x; j<nj; j+=blockDim.x)
      for(int n=threadIdx.y; n<nwalk; n+=blockDim.y)
        B_[ n*nocc_max*nmo_max + a*nmo_max + j ] = static_cast<thrust::complex<T2>>(A_[ a1 + j*nwalk + n ]);
}


void KaKjw_to_KKwaj( int nwalk, int nkpts, int nmo_max, int nmo_tot, 
                     int nocc_max, int* nmo, int* nmo0, 
                     int* nocc, int* nocc0,
                     double const* A, double * B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk,32);
  dim3 block_dim(xblock_dim,yblock_dim,1);
  dim3 grid_dim(nkpts,nkpts,1);
  kernel_KaKjw_to_KKwaj<<<grid_dim, block_dim>>>(nwalk,nkpts,nmo_max,nmo_tot,nocc_max,
                                                 nmo,nmo0,nocc,nocc0,A,B);
  qmc_cuda::cuda_check(hipGetLastError(),"KaKjw_to_KKwaj");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"KaKjw_to_KKwaj");
}

void KaKjw_to_KKwaj( int nwalk, int nkpts, int nmo_max, int nmo_tot,
                     int nocc_max, int* nmo, int* nmo0,
                     int* nocc, int* nocc0,
                     float const* A, float * B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk,32);
  dim3 block_dim(xblock_dim,yblock_dim,1);
  dim3 grid_dim(nkpts,nkpts,1);
  kernel_KaKjw_to_KKwaj<<<grid_dim, block_dim>>>(nwalk,nkpts,nmo_max,nmo_tot,nocc_max,
                                                 nmo,nmo0,nocc,nocc0,A,B);
  qmc_cuda::cuda_check(hipGetLastError(),"KaKjw_to_KKwaj");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"KaKjw_to_KKwaj");
}

void KaKjw_to_KKwaj( int nwalk, int nkpts, int nmo_max, int nmo_tot,
                     int nocc_max, int* nmo, int* nmo0,
                     int* nocc, int* nocc0,
                     double const* A, float * B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk,32);
  dim3 block_dim(xblock_dim,yblock_dim,1);
  dim3 grid_dim(nkpts,nkpts,1);
  kernel_KaKjw_to_KKwaj<<<grid_dim, block_dim>>>(nwalk,nkpts,nmo_max,nmo_tot,nocc_max,
                                                 nmo,nmo0,nocc,nocc0,A,B);
  qmc_cuda::cuda_check(hipGetLastError(),"KaKjw_to_KKwaj");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"KaKjw_to_KKwaj");
}

void KaKjw_to_KKwaj( int nwalk, int nkpts, int nmo_max, int nmo_tot,
                     int nocc_max, int* nmo, int* nmo0,
                     int* nocc, int* nocc0,
                     std::complex<double> const* A, std::complex<double> * B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk,32);
  dim3 block_dim(xblock_dim,yblock_dim,1);
  dim3 grid_dim(nkpts,nkpts,1);
  kernel_KaKjw_to_KKwaj<<<grid_dim, block_dim>>>(nwalk,nkpts,nmo_max,nmo_tot,nocc_max,
                                                 nmo,nmo0,nocc,nocc0,
                reinterpret_cast<thrust::complex<double> const*>(A),
                reinterpret_cast<thrust::complex<double> *>(B));
  qmc_cuda::cuda_check(hipGetLastError(),"KaKjw_to_KKwaj");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"KaKjw_to_KKwaj");
}

void KaKjw_to_KKwaj( int nwalk, int nkpts, int nmo_max, int nmo_tot,
                     int nocc_max, int* nmo, int* nmo0,
                     int* nocc, int* nocc0,
                     std::complex<float> const* A, std::complex<float> * B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk,32);
  dim3 block_dim(xblock_dim,yblock_dim,1);
  dim3 grid_dim(nkpts,nkpts,1);
  kernel_KaKjw_to_KKwaj<<<grid_dim, block_dim>>>(nwalk,nkpts,nmo_max,nmo_tot,nocc_max,
                                                 nmo,nmo0,nocc,nocc0,
                reinterpret_cast<thrust::complex<float> const*>(A),
                reinterpret_cast<thrust::complex<float> *>(B));
  qmc_cuda::cuda_check(hipGetLastError(),"KaKjw_to_KKwaj");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"KaKjw_to_KKwaj");
}

void KaKjw_to_KKwaj( int nwalk, int nkpts, int nmo_max, int nmo_tot,
                     int nocc_max, int* nmo, int* nmo0,
                     int* nocc, int* nocc0,
                     std::complex<double> const* A, std::complex<float> * B)
{
  int xblock_dim = 16;
  int yblock_dim = std::min(nwalk,32);
  dim3 block_dim(xblock_dim,yblock_dim,1);
  dim3 grid_dim(nkpts,nkpts,1);
  kernel_KaKjw_to_KKwaj<<<grid_dim, block_dim>>>(nwalk,nkpts,nmo_max,nmo_tot,nocc_max,
                                                 nmo,nmo0,nocc,nocc0,
                reinterpret_cast<thrust::complex<double> const*>(A),
                reinterpret_cast<thrust::complex<float> *>(B));
  qmc_cuda::cuda_check(hipGetLastError(),"KaKjw_to_KKwaj");
  qmc_cuda::cuda_check(hipDeviceSynchronize(),"KaKjw_to_KKwaj");
}


}
