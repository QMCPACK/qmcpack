#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov
//    Lawrence Livermore National Laboratory
////////////////////////////////////////////////////////////////////////////////

#include <cassert>
#include <complex>
#include <thrust/complex.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>

namespace kernels
{
template<typename T>
__global__ void kernel_print(T const* p, int n)
{
  printf("D: %d ", n);
  for (int i = 0; i < n; i++)
    printf("%g ", *(p + i));
}

template<>
__global__ void kernel_print(int const* p, int n)
{
  printf("int: %d ", n);
  for (int i = 0; i < n; i++)
    printf("%d ", *(p + i));
}

template<>
__global__ void kernel_print(long const* p, int n)
{
  printf("long: %d ", n);
  for (int i = 0; i < n; i++)
    printf("%ld ", *(p + i));
}

template<>
__global__ void kernel_print(size_t const* p, int n)
{
  printf("ulong: %d ", n);
  for (int i = 0; i < n; i++)
    printf("%lu ", *(p + i));
}

template<typename T>
__global__ void kernel_print(thrust::complex<T> const* p, int n)
{
  printf("Z: %d ", n);
  for (int i = 0; i < n; i++)
    printf("(%g, %g) ", (p + i)->real(), (p + i)->imag());
}

void print(std::string str, double const* p, int n)
{
  std::cout << str << "D n: " << n << " ";
  kernel_print<<<1, 1>>>(p, n);
  hipDeviceSynchronize();
  std::cout << std::endl;
}

void print(std::string str, int const* p, int n)
{
  std::cout << str << "I n: " << n << " ";
  kernel_print<<<1, 1>>>(p, n);
  hipDeviceSynchronize();
  std::cout << std::endl;
}

void print(std::string str, size_t const* p, int n)
{
  std::cout << str << "UL n: " << n << " ";
  kernel_print<<<1, 1>>>(p, n);
  hipDeviceSynchronize();
  std::cout << std::endl;
}

void print(std::string str, long const* p, int n)
{
  std::cout << str << "L n: " << n << " ";
  kernel_print<<<1, 1>>>(p, n);
  hipDeviceSynchronize();
  std::cout << std::endl;
}

void print(std::string str, std::complex<double> const* p, int n)
{
  std::cout << str << " Z n: " << n << " ";
  kernel_print<<<1, 1>>>(reinterpret_cast<thrust::complex<double> const*>(p), n);
  hipDeviceSynchronize();
  std::cout << std::endl;
}

} // namespace kernels
