#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory 
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov 
//    Lawrence Livermore National Laboratory 
////////////////////////////////////////////////////////////////////////////////

#include<cassert>
#include <complex>
#include<hip/hip_runtime.h>
#include <thrust/complex.h>
#include<hip/hip_runtime.h>
#define QMC_CUDA 1
#include "AFQMC/Memory/CUDA/cuda_utilities.hpp"

namespace kernels 
{



template<typename T>
__global__ void kernel_setIdentity(int n, T * A, int lda)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  if( (i < n) && (j < n) ) 
    if(i==j) {
      A[i*lda+i] = T(1.0); 
    } else {  
      A[j*lda+i] = T(0.0);
    }
}

template<typename T>
__global__ void kernel_setIdentity(int n, thrust::complex<T> * A, int lda)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  if( (i < n) && (j < n) )
    if(i==j) {
      A[i*lda+i] = thrust::complex<T>(1.0,0.0);
    } else {
      A[j*lda+i] = thrust::complex<T>(0.0,0.0);
    }
}

void setIdentity(int n, double * A, int lda)
{
  int xblock_dim = 16;
  int xgrid_dim = (n + xblock_dim - 1)/xblock_dim;
  dim3 block_dim(xblock_dim,xblock_dim);
  dim3 grid_dim(xgrid_dim,xgrid_dim); 
  kernel_setIdentity<<<grid_dim, block_dim>>>(n,A,lda);
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void setIdentity(int n, float * A, int lda)
{
  int xblock_dim = 16;
  int xgrid_dim = (n + xblock_dim - 1)/xblock_dim;
  dim3 block_dim(xblock_dim,xblock_dim);
  dim3 grid_dim(xgrid_dim,xgrid_dim);
  kernel_setIdentity<<<grid_dim, block_dim>>>(n,A,lda);
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void setIdentity(int n, std::complex<double> * A, int lda)
{
  int xblock_dim = 16;
  int xgrid_dim = (n + xblock_dim - 1)/xblock_dim;
  dim3 block_dim(xblock_dim,xblock_dim);
  dim3 grid_dim(xgrid_dim,xgrid_dim);
  kernel_setIdentity<<<grid_dim, block_dim>>>(n,reinterpret_cast<thrust::complex<double> *>(A),lda);
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void setIdentity(int n, std::complex<float> * A, int lda)
{
  int xblock_dim = 16;
  int xgrid_dim = (n + xblock_dim - 1)/xblock_dim;
  dim3 block_dim(xblock_dim,xblock_dim);
  dim3 grid_dim(xgrid_dim,xgrid_dim);
  kernel_setIdentity<<<grid_dim, block_dim>>>(n,reinterpret_cast<thrust::complex<float> *>(A),lda);
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

}

