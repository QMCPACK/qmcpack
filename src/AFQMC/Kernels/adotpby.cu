#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory 
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov 
//    Lawrence Livermore National Laboratory 
////////////////////////////////////////////////////////////////////////////////

#include<cassert>
#include <complex>
#include<hip/hip_runtime.h>
#include <thrust/complex.h>
#include<hip/hip_runtime.h>
#define QMC_CUDA 1
#include "AFQMC/Memory/CUDA/cuda_utilities.hpp"

namespace kernels 
{

// Meant to be run with 1 block
/*
template<typename T>
__global__ void kernel_adotpby(int N, T const alpha, T const* x, int const incx, 
                                      T const* y, int const incy, T const beta, T* res) {
   // assert(blockIdx.x==0 and blockIdx.y==0 and blockIdx.z==0)

   __shared__ T tmp[256];
   int t = threadIdx.x;

   tmp[t]=T(0.0);
   int nloop = (N+blockDim.x-1)/blockDim.x;

   for(int i=0, ip=threadIdx.x; i<nloop; i++, ip+=blockDim.x)
    if(ip < N)
    {
      tmp[t] += x[ip*incx]*y[ip*incy]; 
    }
   tmp[t] *= alpha;
   __syncthreads();

   // not optimal but ok for now
   if (threadIdx.x == 0) {
     int imax = (N > blockDim.x)?blockDim.x:N;
     for(int i=1; i<imax; i++)
       tmp[0] += tmp[i];
     *res = tmp[0] + beta *(*res);
   }
   __syncthreads();
}
*/

template<typename T, typename Q> 
__global__ void kernel_adotpby(int N, T const alpha, T const* x, int const incx,
                                      T const* y, int const incy, Q const beta, Q* res) {
   // assert(blockIdx.x==0 and blockIdx.y==0 and blockIdx.z==0)

   __shared__ T tmp[256];
   int t = threadIdx.x;

   tmp[t]=T(0.0);
   int nloop = (N+blockDim.x-1)/blockDim.x;

   for(int i=0, ip=threadIdx.x; i<nloop; i++, ip+=blockDim.x)
    if(ip < N)
    {
      tmp[t] += x[ip*incx]*y[ip*incy];
    }
   tmp[t] *= alpha;
   __syncthreads();

   // not optimal but ok for now
   if (threadIdx.x == 0) {
     int imax = (N > blockDim.x)?blockDim.x:N;
     for(int i=1; i<imax; i++)
       tmp[0] += tmp[i];
     *res = static_cast<Q>(tmp[0]) + beta *(*res);
   }
   __syncthreads();
}

void adotpby(int N, double const alpha, double const* x, int const incx,
                    double const* y, int const incy, 
                    double const beta, double* res) 
{
  kernel_adotpby<<<1,256>>>(N,alpha,x,incx,y,incy,beta,res);
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void adotpby(int N, std::complex<double> const alpha, 
                    std::complex<double> const* x, int const incx,
                    std::complex<double> const* y, int const incy, 
                    std::complex<double> const beta, std::complex<double>* res)
{
  kernel_adotpby<<<1,256>>>(N,
                            static_cast<thrust::complex<double> const >(alpha),
                            reinterpret_cast<thrust::complex<double> const*>(x),incx,
                            reinterpret_cast<thrust::complex<double> const*>(y),incy,
                            static_cast<thrust::complex<double> const>(beta),
                            reinterpret_cast<thrust::complex<double> *>(res));
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void adotpby(int N, float const alpha, float const* x, int const incx,
                    float const* y, int const incy,                   
                    float const beta, float* res)
{
  kernel_adotpby<<<1,256>>>(N,alpha,x,incx,y,incy,beta,res);
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void adotpby(int N, std::complex<float> const alpha, 
                    std::complex<float> const* x, int const incx,
                    std::complex<float> const* y, int const incy,
                    std::complex<float> const beta, std::complex<float>* res)
{
  kernel_adotpby<<<1,256>>>(N,
                            static_cast<thrust::complex<float> const>(alpha),
                            reinterpret_cast<thrust::complex<float> const*>(x),incx,
                            reinterpret_cast<thrust::complex<float> const*>(y),incy,
                            static_cast<thrust::complex<float> const>(beta),
                            reinterpret_cast<thrust::complex<float> *>(res));
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void adotpby(int N, float const alpha, float const* x, int const incx,
                    float const* y, int const incy,
                    double const beta, double* res)
{
  kernel_adotpby<<<1,256>>>(N,alpha,x,incx,y,incy,beta,res);
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void adotpby(int N, std::complex<float> const alpha,
                    std::complex<float> const* x, int const incx,
                    std::complex<float> const* y, int const incy,
                    std::complex<double> const beta, std::complex<double>* res)
{
  kernel_adotpby<<<1,256>>>(N,
                            static_cast<thrust::complex<float> const>(alpha),
                            reinterpret_cast<thrust::complex<float> const*>(x),incx,
                            reinterpret_cast<thrust::complex<float> const*>(y),incy,
                            static_cast<thrust::complex<double> const>(beta),
                            reinterpret_cast<thrust::complex<double> *>(res));
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

}

