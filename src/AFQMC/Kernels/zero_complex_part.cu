#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source
// License.  See LICENSE file in top directory for details.
//
// Copyright (c) 2016 Jeongnim Kim and QMCPACK developers.
//
// File developed by:
//    Lawrence Livermore National Laboratory 
//
// File created by:
// Miguel A. Morales, moralessilva2@llnl.gov 
//    Lawrence Livermore National Laboratory 
////////////////////////////////////////////////////////////////////////////////

#include<cassert>
#include <complex>
#include<hip/hip_runtime.h>
#include <thrust/complex.h>
#include<hip/hip_runtime.h>
#define QMC_CUDA 1
#include "AFQMC/Memory/CUDA/cuda_utilities.hpp"

namespace kernels 
{

template<typename T>
__global__ void kernel_zero_complex_part(int n, thrust::complex<T> * x)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i<n)
    x[i] = thrust::complex<T>(x[i].real(),0.0); 
}

void zero_complex_part(int n, std::complex<double> * x) 
{
  int block_dim = 256;
  int grid_dim = (n + block_dim - 1)/block_dim;
  kernel_zero_complex_part<<<grid_dim, block_dim>>>(n,reinterpret_cast<thrust::complex<double>*>(x));
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void zero_complex_part(int n, std::complex<float> * x)
{
  int block_dim = 256;
  int grid_dim = (n + block_dim - 1)/block_dim;
  kernel_zero_complex_part<<<grid_dim, block_dim>>>(n,reinterpret_cast<thrust::complex<float>*>(x));
  qmc_cuda::cuda_check(hipDeviceSynchronize());
}

void zero_complex_part(int n, double * x)
{
  return;
}

void zero_complex_part(int n, float * x)
{
  return;
}

}
