#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2021 QMCPACK developers.
//
// File developed by: Peter Doak, doakpw@ornl.gov, Oak Ridge National Laboratory
//
// File created by: Peter Doak, doakpw@ornl.gov, Oak Ridge National Laboratory
//////////////////////////////////////////////////////////////////////////////////////


#include <hipblas.h>
#include "cuBLAS_LU.hpp"
#include "hipError_t.h"
#include <stdexcept>
#include <type_traits>
#include <complex>
#include <hip/hip_complex.h>
#include "cuBLAS.hpp"
#include <thrust/system/cuda/detail/core/util.h>

namespace qmcplusplus
{
namespace cuBLAS_LU
{


template<int COLBS>
__global__ void computeLogDet_kernel(const int n,
                                     const hipDoubleComplex* const LU_diags,
                                     const int* const pivots,
                                     hipDoubleComplex* logdets)
{
  const int iw                                   = blockIdx.x;
  const int block_num                            = blockIdx.y;
  const hipDoubleComplex* __restrict__ LU_diag_iw = LU_diags + iw * n;
  const int* __restrict__ pivot_iw               = pivots + iw * n;
  int n_index                                    = threadIdx.x + block_num * COLBS;
  __shared__ hipDoubleComplex logdet_vals[COLBS];
  logdet_vals[threadIdx.x] = {0.0, 0.0};
  if (n_index < n)
  {
    logdet_vals[threadIdx.x].x = norm(2, (double*)(LU_diag_iw + n_index));
    logdet_vals[threadIdx.x].y = atan2(LU_diag_iw[n_index].y, LU_diag_iw[n_index].x);
  }
  // insure that when we reduce logdet_vals all the threads in the block are done.
  __syncthreads();
  {
    hipDoubleComplex block_sum_log_det{0.0, 0.0};
    for (int iv = 0; iv < COLBS; ++iv)
    {
      block_sum_log_det.x += logdet_vals[iv].x;
      block_sum_log_det.y += logdet_vals[iv].y;
    }
    atomicAdd((double*)(logdets + iw), block_sum_log_det.x);
    atomicAdd((double*)(logdets + iw) + 1, block_sum_log_det.y);
  }
}

template<int COLBS>
__global__ void computeLogDet_kernel(const int n,
                                     const double* const LU_diags,
                                     const int* const pivots,
                                     hipDoubleComplex* logdets)
{
  const int iw                          = blockIdx.x;
  const int block_num                   = blockIdx.y;
  const double* __restrict__ LU_diag_iw = LU_diags + iw * n;
  const int* __restrict__ pivots_iw     = pivots + iw * n;
  int n_index                           = threadIdx.x + block_num * COLBS;
  __shared__ hipDoubleComplex logdet_vals[COLBS];
  logdet_vals[threadIdx.x] = {0.0, 0.0};
  if (n_index < n)
  {
    logdet_vals[threadIdx.x].x = log(abs(LU_diag_iw[n_index]));
    logdet_vals[threadIdx.x].y = ((LU_diag_iw[n_index] < 0) != ((pivots_iw[n_index] - 1) == n_index)) * M_PI;
  }
  // insure that when we reduce logdet_vals all the threads in the block are done.
  __syncthreads();
  if (threadIdx.x == 0)
  {
    hipDoubleComplex block_sum_log_det{0.0, 0.0};
    for (int iv = 0; iv < COLBS; ++iv)
    {
      block_sum_log_det.x += logdet_vals[iv].x;
      block_sum_log_det.y += logdet_vals[iv].y;
    }
    atomicAdd((double*)(logdets + iw), block_sum_log_det.x);
    atomicAdd((double*)(logdets + iw) + 1, block_sum_log_det.y);
  }
}

/** Calculates logdets using LU_diags and pivots
 *  \param[in] LU_mat - the LU output from cublasXgetrfBatched
 *  \param[out] LU_diags - the LU_diags from the LU
 *  \param[in] batch_size - no a big deal here.
 */
template<typename T>
hipError_t computeLogDet_batched_impl(hipStream_t& hstream,
                                       const int n,
                                       const T* LU_diags,
                                       const int* pivots,
                                       hipDoubleComplex* logdets,
                                       const int batch_size)
{
  // Perhaps this should throw an exception. I can think of no good reason it should ever happen other than
  // developer error.
  if (batch_size == 0 || n == 0)
    return hipSuccess;

  const int COLBS          = 256;
  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_size, num_col_blocks);
  computeLogDet_kernel<COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, LU_diags, pivots, logdets);

  return hipPeekAtLastError();
}

void computeLogDet_batched(hipStream_t& hstream,
                           const int n,
                           const double* LU_diags,
                           const int* pivots,
                           std::complex<double>* logdets,
                           const int batch_size)
{
  cudaErrorCheck(computeLogDet_batched_impl(hstream, n, LU_diags, pivots, reinterpret_cast<hipDoubleComplex*>(logdets),
                                            batch_size),
                 "failed to calculate log determinant values in computeLogDet_batched_impl");
}

template<typename T, int COLBS>
__global__ void computeLUDiag_kernel(const int n, const int lda, T** mat_lus, T* LU_diag)
{
  const int iw                = blockIdx.x;
  const int block_num         = blockIdx.y;
  const T* __restrict__ lu_iw = mat_lus[iw];
  T* __restrict__ LU_diag_iw  = LU_diag + iw * n;
  int n_index                 = threadIdx.x + block_num * COLBS;
  if (n_index < n)
    *(LU_diag_iw + n_index) = *(lu_iw + n_index * lda + n_index);
}

/** Extracts the LU_diags from the LU in invA.
 *  \param[in] LU_mat - the LU output from cublasXgetrfBatched
 *  \param[out] LU_diags - the LU_diags from the LU
 *  \param[in] batch_size - no a big deal here.
 */
template<typename T>
hipError_t computeLUDiag_batched_impl(hipStream_t hstream,
                                       const int n,
                                       const int lda,
                                       T** LU_mat,
                                       T* LU_diags,
                                       const int batch_size)
{
  // Perhaps this should throw an exception. I can think of no good reason it should ever happen other than
  // developer error.
  if (batch_size == 0 || n == 0)
    return hipSuccess;

  const int COLBS          = 256;
  const int num_col_blocks = (n + COLBS - 1) / COLBS;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_size, num_col_blocks);
  computeLUDiag_kernel<T, COLBS><<<dimGrid, dimBlock, 0, hstream>>>(n, lda, LU_mat, LU_diags);

  return hipPeekAtLastError();
}

/** Takes the transpose of PsiM using LU factorization calculates the log determinant and invPsiM
 *
 *  \param[inout] Ms -       pointers to pointers to working memory for Ms that are used to return invMs
 *  \param[in]    pivots -   pointer to n * nw ints allocated in device memory for pivots array.
 *  \param[in]    infos -    pointer to nw ints allocated in device memory factorization infos
 *  \param[out]   log_dets - pointer device memory for nw log determinant values to be returned, 
 *                           maybe this is supposed to be just RealType
 *  \param[in]    batch_size - if this changes over run a huge performance hit will be taken as memory allocation syncs device.
 */
void computeInverseAndDetLog_batched(hipblasHandle_t& h_cublas,
                                     hipStream_t& hstream,
                                     const int n,
                                     const int lda,
                                     double* Ms[],
                                     double* Cs[],
                                     double* LU_diags,
                                     int* pivots,
                                     int* infos,
                                     std::complex<double>* log_dets,
                                     const int batch_size)
{
  //LU is returned in Ms
  cublasErrorCheck(cuBLAS::getrf_batched(h_cublas, n, Ms, lda, pivots, infos, batch_size),
                   "cuBLAS::getrf_batched failed in computeInverseAndDetLog_batched");
  cudaErrorCheck(computeLUDiag_batched_impl(hstream, n, lda, Ms, LU_diags, batch_size),
                 "failed to extract LU diag values at cuomputeLUDiag_batched_impl");
  cudaErrorCheck(computeLogDet_batched_impl(hstream, n, LU_diags, pivots, reinterpret_cast<hipDoubleComplex*>(log_dets),
                                            batch_size),
                 "failed to calculate log determinant values in computeLogDet_batched_impl");
  cublasErrorCheck(cuBLAS::getri_batched(h_cublas, n, Ms, lda, pivots, Cs, lda, infos, batch_size),
                   "cuBLAS::getri_batched failed in computeInverseAndDetLog_batched");
}

void computeGetrf_batched(hipblasHandle_t& h_cublas,
                          const int n,
                          const int lda,
                          double* Ms[],
                          int* pivots,
                          int* infos,
                          const int batch_size)
{
  cublasErrorCheck(cuBLAS::getrf_batched(h_cublas, n, Ms, lda, pivots, infos, batch_size),
                   "cuBLAS::getrf_batched failed in computeInverseAndDetLog_batched");
}

void computeGetri_batched(hipblasHandle_t& h_cublas,
                          const int n,
                          const int lda,
                          double* Ms[],
                          double* Cs[],
                          int* pivots,
                          int* infos,
                          const int batch_size)
{
  cublasErrorCheck(cuBLAS::getri_batched(h_cublas, n, Ms, lda, pivots, Cs, lda, infos, batch_size),
                   "cuBLAS::getri_batched failed in computeInverseAndDetLog_batched");
}


void computeLUDiag_batched(hipStream_t& hstream,
                           const int n,
                           const int lda,
                           double** Ms,
                           double* LU_diags,
                           const int batch_size)
{
  cudaErrorCheck(computeLUDiag_batched_impl(hstream, n, lda, Ms, LU_diags, batch_size),
                 "failed to extract LU diag values at cuomputeLUDiag_batched_impl");
}

template<typename T, typename COMPLT, int COLBS>
__global__ void peekinvM_kernel(T** M, T** invM, int* pivots, int* infos, COMPLT* log_dets)
{
  const int iw        = blockIdx.x;
  const int block_num = blockIdx.y;
  const T* invM_iw    = invM[iw];
  const T* M_iw       = M[iw];
  COMPLT* log_dets_iw         = log_dets + iw;
}

template<typename T, typename COMPLT>
hipError_t peekinvM_batched_impl(hipStream_t hstream,
                                  T** M,
                                  T** invM,
                                  int* pivots,
                                  int* infos,
                                  COMPLT* log_dets,
                                  const int batch_size)
{
  const int COLBS = 256;
  dim3 dimBlock(COLBS);
  dim3 dimGrid(batch_size, 1);
  peekinvM_kernel<T, COMPLT, COLBS>
    <<<dimGrid, dimBlock, 0, hstream>>>(M, invM, pivots, infos, log_dets);
  return hipPeekAtLastError();
}

template hipError_t peekinvM_batched_impl<double, hipDoubleComplex>(hipStream_t hstream,
                                  double** M,
                                  double** invM,
                                  int* pivots,
                                  int* infos,
                                  hipDoubleComplex* log_dets,
                                  const int batch_size);
  
void peekinvM_batched(hipStream_t& hstream,
                      double** Ms,
                      double** invMs,
                      int* pivots,
                      int* infos,
                      std::complex<double>* log_dets,
                      const int batch_size)
{
  cudaErrorCheck(peekinvM_batched_impl(hstream, Ms, invMs, pivots, infos, reinterpret_cast<TypesMapper<std::complex<double>>*>(log_dets), batch_size),
                 "failed to extract LU diag values at cuomputeLUDiag_batched_impl");
}


} // namespace cuBLAS_LU
} // namespace qmcplusplus
