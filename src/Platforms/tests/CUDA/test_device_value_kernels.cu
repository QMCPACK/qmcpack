#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////
// This file is distributed under the University of Illinois/NCSA Open Source License.
// See LICENSE file in top directory for details.
//
// Copyright (c) 2021 QMCPACK developers.
//
// File developed by: Peter Doak, doakpw@ornl.gov, Oak Ridge National Laboratory
//
// File created by: Peter Doak, doakpw@ornl.gov, Oak Ridge National Laboratory
//////////////////////////////////////////////////////////////////////////////////////

#include "test_device_value_kernels.hpp"
#include "CUDA/CUDAallocator.hpp"

namespace qmcplusplus
{
namespace testing
{

/** checking an on device pointer's value against a passed value
 */
template<typename T>
__global__ void checkValue_kernel(T* device_value_ptr, const T value, bool* result)
{
  if (*device_value_ptr == value)
    *result = true;
  else
    *result = false;
}

/** checking an on device pointer's value against a passed value
 */
template<typename T>
hipError_t checkValueCUDA(hipStream_t hstream, T* device_value_ptr, T value, bool& result)
{
  CUDAAllocator<bool> bool_allocator;
  CUDAHostAllocator<bool> host_bool_allocator;
  bool* device_result = bool_allocator.allocate(1);
  bool* host_result   = host_bool_allocator.allocate(1);
  dim3 dim_block(1);
  dim3 dim_grid(1);
  checkValue_kernel<<<dim_grid, dim_block, 0, hstream>>>(device_value_ptr, value, device_result);
  cudaCheck(hipStreamSynchronize(hstream));
  hipError_t kernel_error = hipPeekAtLastError();
  cudaCheck(hipMemcpyAsync(host_result, device_result, sizeof(bool), hipMemcpyDeviceToHost, hstream));
  cudaCheck(hipStreamSynchronize(hstream));
  result = *host_result;
  bool_allocator.deallocate(device_result, 1);
  host_bool_allocator.deallocate(host_result, 1);
  return kernel_error;
}

__global__ void checkDualStruct_kernel(DualStruct* device_struct_ptr, const DualStruct dual_struct, bool* result)
{
  if (device_struct_ptr->index == dual_struct.index && device_struct_ptr->value == dual_struct.value)
    *result = true;
  else
    *result = false;
}

/** check a particular test structure at device pointer against passed  by value struct
 */  
hipError_t checkDualStruct(hipStream_t hstream, DualStruct* device_struct_ptr, DualStruct dual_struct, bool& result)
{
  CUDAAllocator<bool> bool_allocator;
  CUDAHostAllocator<bool> host_bool_allocator;
  bool* device_result = bool_allocator.allocate(1);
  bool* host_result   = host_bool_allocator.allocate(1);
  dim3 dim_block(1);
  dim3 dim_grid(1);
  checkDualStruct_kernel<<<dim_grid, dim_block, 0, hstream>>>(device_struct_ptr, dual_struct, device_result);
  cudaCheck(hipStreamSynchronize(hstream));
  hipError_t kernel_error = hipPeekAtLastError();
  cudaCheck(hipMemcpyAsync(host_result, device_result, sizeof(bool), hipMemcpyDeviceToHost, hstream));
  cudaCheck(hipStreamSynchronize(hstream));
  result = *host_result;
  bool_allocator.deallocate(device_result, 1);
  host_bool_allocator.deallocate(host_result, 1);
  return kernel_error;
}

template hipError_t checkValueCUDA(hipStream_t hstream, double* device_value_ptr, double value, bool& result);

} // namespace testing
} // namespace qmcplusplus
